#include "hip/hip_runtime.h"
/*

Copyright 1996-2006 Roeland Merks

This file is part of Tissue Simulation Toolkit.

Tissue Simulation Toolkit is free software; you can redistribute
it and/or modify it under the terms of the GNU General Public
License as published by the Free Software Foundation; either
version 2 of the License, or (at your option) any later version.

Tissue Simulation Toolkit is distributed in the hope that it will
be useful, but WITHOUT ANY WARRANTY; without even the implied
warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Tissue Simulation Toolkit; if not, write to the Free
Software Foundation, Inc., 51 Franklin St, Fifth Floor, Boston, MA
02110-1301 USA

*/

    //equations for Paci2020
  
    // Software implementation of the Paci2020 model of the action potential 
    // of human induced pluripotent stem cell-derived cardiomyocytes, 
    // used in 10.1016/j.bpj.2020.03.018
    //
    // This software is provided for NON-COMMERCIAL USE ONLY 
    // License included:

/*
ACADEMIC PUBLIC LICENSE (software implementation of the Paci2020 model, v1.0)

Copyright (c) 2021 Michelangelo Paci, Tampere University Foundation sr, All rights reserved

The following license governs the use of the software implementation of the Paci2020 model of the action potential of human induced pluripotent stem cell-derived cardiomyocytes, used in the paper "All-Optical Electrophysiology Refines Populations of In Silico Human iPSC-CMs for Drug Evaluation" (10.1016/j.bpj.2020.03.018), in non-commercial academic environments. In case of need for license extensions, please contact inventions@tuni.fi

------------------------------------------------------------------------------------------------------
Preamble

This license contains the terms and conditions of using the software implementation of the Paci2020 model in non-commercial settings: at academic institutions for teaching and research use, and at not-for-profit research organizations. You will find that this license provides non-commercial users of the software implementation of the Paci2020 model with rights that are similar to the well-known GNU General Public License 2.0, yet it retains the possibility for the software implementation of the Paci2020 model authors to financially support the development by selling commercial licenses. In fact, if you intend to use the software implementation of the Paci2020 model in a "for-profit" environment, where the software implementation of the Paci2020 model simulations are conducted to develop or enhance a product (including commercial or industry-sponsored research at academic institutions), or to use the software implementation of the Paci2020 model in a commercial service offering, then you need to obtain a license extension for the software implementation of the Paci2020 model. In that case, please contact inventions@tuni.fi.

What are the rights given to non-commercial users? Similarly, to GPL 2.0, you have the right to use the software, to distribute copies, to receive source code, to change the software and distribute your modifications or the modified software. Also, similarly to the GPL 2.0, if you distribute verbatim or modified copies of this software, they must be distributed under this license.

By modeling the GPL 2.0, this license guarantees that you’re safe when using the software implementation of the Paci2020 model in your work, for teaching, and research. This license guarantees that the software implementation of the Paci2020 model will remain available free of charge for non-profit use. You can modify the software implementation of the Paci2020 model to your purposes, and you can also share your modifications. Even in case of the authors abandoning the software implementation of the Paci2020 model entirely, this license permits anyone to continue developing it from the last release, and to create further releases under this license.

The precise terms and conditions for using, copying, distribution and modification follow.
------------------------------------------------------------------------------------------------------

Terms and Conditions for Use, Copying, Distribution and Modification
Definitions

•	"Program" means a copy of the software implementation of the Paci2020 model and all the files included in this archive, which are said to be distributed under this Academic Public License.
•	"Work based on the Program" means either the Program or any derivative work under copyright law: that is to say, a work containing the Program or a portion of it, either verbatim or with modifications and/or translated into another language. (Hereinafter, translation is included without limitation in the term "modification".)
•	"Using the Program" means any act of creating executables that contain or directly use libraries that are part of the Program, running any of the tools that are part of the Program, or creating works based on the Program.
•	Each licensee is addressed as "you".

§1. Permission is hereby granted to use the Program free of charge for any non-commercial purpose, including teaching and research at universities, colleges and other educational institutions, non-commercial research at organizations that are either not-for-profit or reinvest all profits in their scientific research, and personal not-for-profit purposes. For using the Program for commercial purposes, including but not restricted to commercial research at academic institutions, industrially sponsored research at academic institutions, consulting activities, and design of commercial hardware or software products or services, you have to contact inventions@tuni.fi for an appropriate license.

§2. You may copy and distribute verbatim copies of the source code of the program via any medium, provided that you add a conspicuous and appropriate copyright notice and a warranty disclaimer to each copy. Retain all notices relating to this license and the lack of any warranty. Forward a copy of this license to all other recipients of the program.

§3. You are entitled to change your copies of the Program or a part thereof and thus create a work based on the Program. You may copy and distribute changes or work in accordance with the provisions of Section 2 provided you also meet all of the following conditions: a) You must ensure that the changed files are provided with noticeable comments stating the author of the change and when this change was made. b) You must ensure that all work that you distribute or publish, that contains or is derived from the Program or parts thereof, as a whole, is licensed under the conditions of this license.

These requirements apply to the changed work as a whole. If identifiable sections of this work do not come from the Program and can be considered separate, this license and its terms do not apply to those sections if you distribute them as separate work. However, if you distribute the same sections as part of a whole that is based on the Program, the distribution of the whole must be done in accordance with the terms of this license as outlined in §2, independently of who wrote it.

The mere merging of another work that is not based on the Program with the Program (or a work based on the Program) does not bring the other work into the scope of this license.

§4. You may copy and distribute the Program (or a work based on it, in accordance with §3, in object code or executable form in accordance with the provisions of above Sections 2 and 3, provided that you also add the complete corresponding machine-readable source code. For an executable program, complete source code means the entire source code for all modules contained therein, as well as all associated interface definition files and scripts, with which the compilation and installation of the executable file is controlled.

§5. Any attempt to copy, modify, sublicense or distribute the Program in any other way than specified in this license is void, and will automatically terminate your rights under this license. However, parties who have received copies or rights from you under this license will not lose their license as long as these parties fully comply with the terms.

§6. You do not have to accept this license because you have not signed it. However, if you want to change or distribute the Program (or a work based on the Program), you automatically consent to this license and all its terms for copying, distributing or changing the program or the works based upon it.

§7. Each time you redistribute the Program (or any work based on the Program), the recipient automatically acquires a license from the initial licensor to copy, distribute or modify the Program in accordance with these terms and conditions. You may not impose any further restrictions on the recipient's exercise of the rights granted here. You are not responsible for ensuring that this license is enforced by third parties.

§8. If, as a result of a court decision or violation of a patent right, or for any other reason (not limited to patent issues), conditions are imposed that conflict with the terms of this license, you will not be released from the terms of this license. If you cannot distribute the Program because you would have to meet obligations under this license and other obligations at the same time, you may not distribute the Program at all.

§9. If the distribution and/or use of the Program in certain countries is restricted either by patents or by copyrighted interfaces, the original copyright holder who puts the Program under this license may add an explicit geographic distribution restriction that excludes these countries. In this case, this license contains the restriction as if it was written in the body of this license.

NO WARRANTY

§10. SINCE THE PROGRAM IS LICENSED FOR FREE, THERE IS NO GUARANTEE FOR THE PROGRAM. THE COPYRIGHT HOLDERS AND/OR OTHER PARTIES PROVIDE THE PROGRAM "AS IS" WITHOUT ANY EXPRESSED OR IMPLIED GUARANTEE, INCLUDING BUT NOT LIMITED TO THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE. THE ENTIRE RISK TO THE QUALITY AND PERFORMANCE OF THE PROGRAM IS YOURS. IF THE PROGRAM TURNS OUT TO BE FAULTY, YOU ARE RESPONSIBLE FOR THE COSTS FOR ALL NECESSARY MAINTENANCE, REPAIR OR CORRECTION WORK.

§11. UNDER NO CIRCUMSTANCES WILL A COPYRIGHT HOLDER OR ANY OTHER PARTY WHO CAN MODIFY AND/OR REDISTRIBUTE THE PROGRAM AS PERMITTED ABOVE BE LIABLE FOR DAMAGE, INCLUDING GENERAL, SPECIAL, ACCIDENTAL OR OTHER DAMAGE. THE DISCLAIMER ALSO INCLUDES CONSEQUENTIAL DAMAGES THAT RESULT FROM USING THE PROGRAM ALONE OR IN CONJUNCTION WITH OTHER PROGRAMS, INCLUDING BUT NOT LIMITED TO THE LOSS OR CORRUPTION OF DATA.

IF YOU DO NOT AGREE WITH THIS LICENSE TERMS, DO NOT USE, COPY, CHANGE OR DISTRIBUTE THE PROGRAM (OR A WORK BASED ON THE PROGRAM).

This license was
- initially written by Andras Varga (public domain) for OMNeT++ https://omnetpp.org/intro/license, 
- then adapted by the openCARP project https://opencarp.org/download/license,
- now adapted for the software implementation of the Paci2020 model.
The adaptation is licensed under CC0 1.0 (Public Domain Dedication).
*/
  
  
    //-------------------------------------------------------------------------------
    // State variables
    //-------------------------------------------------------------------------------
  
    // 0: Vm (volt) (in Membrane)
    // 1: Ca_SR (millimolar) (in calcium_dynamics)
    // 2: Cai (millimolar) (in calcium_dynamics)
    // NOT USED 3: g (dimensionless) (in calcium_dynamics)
    // 4: d (dimensionless) (in i_CaL_d_gate)
    // 5: f1 (dimensionless) (in i_CaL_f1_gate)
    // 6: f2 (dimensionless) (in i_CaL_f2_gate)
    // 7: fCa (dimensionless) (in i_CaL_fCa_gate)
    // 8: Xr1 (dimensionless) (in i_Kr_Xr1_gate)
    // 9: Xr2 (dimensionless) (in i_Kr_Xr2_gate)
    // 10: Xs (dimensionless) (in i_Ks_Xs_gate)
    // 11: h (dimensionless) (in i_Na_h_gate)
    // 12: j (dimensionless) (in i_Na_j_gate)
    // 13: m (dimensionless) (in i_Na_m_gate)
    // 14: Xf (dimensionless) (in i_f_Xf_gate)
    // 15: q (dimensionless) (in i_to_q_gate)
    // 16: r (dimensionless) (in i_to_r_gate)
    // 17: Nai (millimolar) (in sodium_dynamics)
    // 18: m_L (dimensionless) (in i_NaL_m_gate)
    // 19: h_L (dimensionless) (in i_NaL_h_gate)
    // 20: RyRa (dimensionless) (in calcium_dynamics)
    // 21: RyRo (dimensionless) (in calcium_dynamics)
    // 22: RyRc (dimensionless) (in calcium_dynamics)


/*ODE solver is an adaptation from Press, W. H. (2007). 
Numerical recipes : the art of scientific computing (3rd ed.). 
/New York, N.Y., [etc.]: Cambridge University Press.

*/




#include <stdio.h>
#include <unistd.h>
#include <math.h>
#include <cstdlib>
#include <fstream>
#include <sstream>
#include <chrono>

#include "crash.hpp"
#include "parameter.hpp"
#include "ca.hpp"
#include "pde.hpp"
#include "conrec.hpp"
#include "graph.hpp"
#include <hipsparse.h>
#include <hip/hip_runtime_api.h>
#define ARRAY_SIZE 2

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* STATIC DATA MEMBER INITIALISATION */
const int PDE::nx[9] = {0, 1, 1, 1, 0,-1,-1,-1, 0 };
const int PDE::ny[9] = {0, 1, 0,-1,-1,-1, 0, 1, 1 };

extern Parameter par;





/** PRIVATE **/

PDE::PDE(const int l, const int sx, const int sy) {
  PDEvars=0;
  thetime=0;
  PDEsteps=0;
  sizex=sx;
  sizey=sy;
  layers=l;
  bpm = par.beats_per_minute;
  pacing_interval = 1/(bpm/60);
  PDEvars = new PDEFIELD_TYPE[layers*sizex*sizey];
  alt_PDEvars = new PDEFIELD_TYPE[layers*sizex*sizey];
  min_stepsize = par.min_stepsize;
  btype = 1;
  dx2 = par.dx*par.dx;
  dt = par.dt;
  usePDEorAltPDE = true;
}


PDE::PDE(void) {

  PDEvars=0;
  alt_PDEvars=0;
  sizex=0; sizey=0; layers=0;
  thetime=0;
  if (par.useopencl){this->SetupOpenCL();}
}

// destructor (virtual)
PDE::~PDE(void) {
  if (PDEvars) {
    hipFree(PDEvars);
  }
  if (alt_PDEvars) {
    hipFree(alt_PDEvars);
  }
  free(couplingcoefficient);
  hipFree(PDEvars);
  hipFree(alt_PDEvars);
  hipFree(couplingcoefficient);
  hipFree(upperH);
  hipFree(diagH);
  hipFree(lowerH);
  hipFree(BH);
  hipFree(XH);
  hipFree(upperV);
  hipFree(diagV);
  hipFree(lowerV);
  hipFree(BV);
}

/*
PDEFIELD_TYPE ***PDE::AllocatePDEvars(const int layers, const int sx, const int sy) { //Omschrijven naar [xcoordinaat][ycoordinaat][layercoordinaat] (eerst compileren zo)
  PDEFIELD_TYPE ***mem;
  sizex=sx; sizey=sy;
  mem=(PDEFIELD_TYPE ***)malloc(layers*sizeof(PDEFIELD_TYPE **));
  if (mem==NULL) {
    MemoryWarning();
  }
  mem[0]=(PDEFIELD_TYPE **)malloc(layers*sizex*sizeof(PDEFIELD_TYPE *));
  if (mem[0]==NULL) { 
    MemoryWarning();
  }
  for (int i=1;i<layers;i++) {
    mem[i]=mem[i-1]+sizex*sizey;
  }
  mem[0][0]=(PDEFIELD_TYPE *)malloc(layers*sizex*sizey*sizeof(PDEFIELD_TYPE));
  if (mem[0][0]==NULL) {
    MemoryWarning();
  }
  for (int i=1;i<layers*sizex;i++) {
    mem[0][i]=mem[0][i-1]+sizey;
  }
  
  //Clear PDE plane 
  for (int i=0;i<layers*sizex*sizey;i++) {
    mem[0][0][i]=0.; 
  }
  return mem;
}


void PDE::AllocateTridiagonalvars(int sx, int sy){
  
  sizex=sx; sizey=sy;


  //Allocate lowerH
  lowerH=(PDEFIELD_TYPE **)malloc(sizey*sizeof(PDEFIELD_TYPE*));
  if (lowerH==NULL)
    MemoryWarning();

  lowerH[0]=(PDEFIELD_TYPE *)malloc(sizey*sizex*sizeof(PDEFIELD_TYPE));
  if (lowerH[0]==NULL)  
    MemoryWarning();

    {for (int i=1;i<sizey;i++) 
    lowerH[i]=lowerH[i-1]+sizex;}
  // clear matrix 

  {for (int i=0;i<sizey*sizex;i++) 
    lowerH[0][i]=0; }

  //Allocate upperH
  upperH=(PDEFIELD_TYPE **)malloc(sizey*sizeof(PDEFIELD_TYPE*));
  if (upperH==NULL)
    MemoryWarning();

  upperH[0]=(PDEFIELD_TYPE *)malloc(sizey*sizex*sizeof(PDEFIELD_TYPE));
  if (upperH[0]==NULL)  
    MemoryWarning();

    {for (int i=1;i<sizey;i++) 
    upperH[i]=upperH[i-1]+sizex;}
  // clear matrix 

  {for (int i=0;i<sizey*sizex;i++) 
    upperH[0][i]=0; }


  //Allocate diagH
  diagH=(PDEFIELD_TYPE **)malloc(sizey*sizeof(PDEFIELD_TYPE*));
  if (diagH==NULL)
    MemoryWarning();

  diagH[0]=(PDEFIELD_TYPE *)malloc(sizey*sizex*sizeof(PDEFIELD_TYPE));
  if (diagH[0]==NULL)  
    MemoryWarning();

    {for (int i=1;i<sizey;i++) 
    diagH[i]=diagH[i-1]+sizex;}
  // clear matrix 

  {for (int i=0;i<sizey*sizex;i++) 
    diagH[0][i]=0; }


  //Allocate BH
  BH=(PDEFIELD_TYPE **)malloc(sizey*sizeof(PDEFIELD_TYPE*));
  if (BH==NULL)
    MemoryWarning();

  BH[0]=(PDEFIELD_TYPE *)malloc(sizey*sizex*sizeof(PDEFIELD_TYPE));
  if (BH[0]==NULL)  
    MemoryWarning();

    {for (int i=1;i<sizey;i++) 
    BH[i]=BH[i-1]+sizex;}

  // clear matrix 
  {for (int i=0;i<sizey*sizex;i++) 
    BH[0][i]=0; }

    //Allocate XH
  XH=(PDEFIELD_TYPE **)malloc(sizey*sizeof(PDEFIELD_TYPE*));
  if (BH==NULL)
    MemoryWarning();

  XH[0]=(PDEFIELD_TYPE *)malloc(sizey*sizex*sizeof(PDEFIELD_TYPE));
  if (XH[0]==NULL)  
    MemoryWarning();

    {for (int i=1;i<sizey;i++) 
    XH[i]=XH[i-1]+sizex;}

  // clear matrix
  {for (int i=0;i<sizey*sizex;i++) 
    XH[0][i]=0; }


  //Allocate lowerV
  lowerV=(PDEFIELD_TYPE **)malloc(sizex*sizeof(PDEFIELD_TYPE*));
  if (lowerV==NULL)
    MemoryWarning();

  lowerV[0]=(PDEFIELD_TYPE *)malloc(sizex*sizey*sizeof(PDEFIELD_TYPE));
  if (lowerV[0]==NULL)  
    MemoryWarning();

    {for (int i=1;i<sizex;i++) 
    lowerV[i]=lowerV[i-1]+sizey;}
  // clear matrix

  {for (int i=0;i<sizex*sizey;i++) 
    lowerV[0][i]=0; }


  //Allocate upperV
  upperV=(PDEFIELD_TYPE **)malloc(sizex*sizeof(PDEFIELD_TYPE*));
  if (upperV==NULL)
    MemoryWarning();

  upperV[0]=(PDEFIELD_TYPE *)malloc(sizex*sizey*sizeof(PDEFIELD_TYPE));
  if (upperV[0]==NULL)  
    MemoryWarning();

    {for (int i=1;i<sizex;i++) 
    upperV[i]=upperV[i-1]+sizey;}
  // clear matrix 

  {for (int i=0;i<sizex*sizey;i++) 
    upperV[0][i]=0; }


  //Allocate diagV
  diagV=(PDEFIELD_TYPE **)malloc(sizex*sizeof(PDEFIELD_TYPE*));
  if (diagV==NULL)
    MemoryWarning();

  diagV[0]=(PDEFIELD_TYPE *)malloc(sizex*sizey*sizeof(PDEFIELD_TYPE));
  if (diagV[0]==NULL)  
    MemoryWarning();

    {for (int i=1;i<sizex;i++) 
    diagV[i]=diagV[i-1]+sizey;}
  // clear matrix

  {for (int i=0;i<sizex*sizey;i++) 
    diagV[0][i]=0; }


  //Allocate BV
  BV=(PDEFIELD_TYPE **)malloc(sizex*sizeof(PDEFIELD_TYPE*));
  if (BV==NULL)
    MemoryWarning();

  BV[0]=(PDEFIELD_TYPE *)malloc(sizex*sizey*sizeof(PDEFIELD_TYPE));
  if (BV[0]==NULL)  
    MemoryWarning();

    {for (int i=1;i<sizex;i++) 
    BV[i]=BV[i-1]+sizey;}
  // clear matrix

  {for (int i=0;i<sizex*sizey;i++) 
    BV[0][i]=0; }

}*/

void PDE::InitializePDEvars(CellularPotts *cpm){
  PDEFIELD_TYPE PDEinit[ARRAY_SIZE];
  bool* mask = cpm->getMask()[0];
  /* For Paci2018
  PDEinit[0] = -0.070;
  PDEinit[1] = 0.32;
  PDEinit[2] = 0.0002;
  PDEinit[3] = 0;
  PDEinit[4] = 0;
  PDEinit[5] = 1;
  PDEinit[6] = 1;
  PDEinit[7] = 1;
  PDEinit[8] = 0;
  PDEinit[9] = 1;
  PDEinit[10] = 0;
  PDEinit[11] = 0.75;
  PDEinit[12] = 0.75; 
  PDEinit[13] = 0;
  PDEinit[14] = 0.1;
  PDEinit[15] = 1;
  PDEinit[16] = 0;
  PDEinit[17] = 9.2;
  PDEinit[18] = 0;
  PDEinit[19] = 0.75;
  PDEinit[20] = 0.3;
  PDEinit[21] = 0.9;
  PDEinit[22] = 0.1;*/

  PDEinit[0] = -1.2275879383;
  PDEinit[1] = -0.6109462976;
  for (int layer = 0; layer<layers; layer++){
    for (int i = layer*sizex*sizey; i<(layer+1)*sizex*sizey; i++){
      PDEvars[i] = PDEinit[layer];
      //if (layer == 0 && (i%(sizey+1) == 130))
      //  PDEvars[i] = 100;
    }
  }

  for (int i = 0; i < sizex*sizey; i++)
    if(!mask[i])
          PDEvars[i] = -10;
  //PDEvars[246183] = 10000;

}




void PDE::Plot(Graphics *g,const int l) {
  // l=layer: default layer is 0
  for (int x=0;x<sizex;x++) {
    for (int y=0;y<sizey;y++) {
      // Make the pixel four times as large
      // to fit with the CPM plane
      g->Point(MapColour(PDEvars[l*sizex*sizey+x*sizey+y]),x,y);
      g->Point(MapColour(PDEvars[l*sizex*sizey+x*sizey+y]),x+1,y);
      g->Point(MapColour(PDEvars[l*sizex*sizey+x*sizey+y]),x,y+1);
      g->Point(MapColour(PDEvars[l*sizex*sizey+x*sizey+y]),x+1,y+1);
    }
  }
}

// Plot the value of the PDE only in the medium of the CPM
void PDE::Plot(Graphics *g, CellularPotts *cpm, const int l) {
  // suspend=true suspends calling of DrawScene
  for (int x=0;x<sizex;x++) {
    for (int y=0;y<sizey;y++) { 
      if (cpm->Sigma(x,y)==0) {
	// Make the pixel four times as large
	// to fit with the CPM plane
        g->Point(MapColour(PDEvars[l*sizex*sizey+x*sizey+y]),x,y);
        g->Point(MapColour(PDEvars[l*sizex*sizey+x*sizey+y]),x+1,y);
        g->Point(MapColour(PDEvars[l*sizex*sizey+x*sizey+y]),x,y+1);
        g->Point(MapColour(PDEvars[l*sizex*sizey+x*sizey+y]),x+1,y+1);
      }
    }
  }
}

void PDE::ContourPlot(Graphics *g, int l, int colour) {
  // calls "conrec" routine by Paul Bourke, as downloaded from
  // http://astronomy.swin.edu.au/~pbourke/projection/conrec

  // number of contouring levels
  int nc = 10;

  // A one dimensional array z(0:nc-1) that saves as a list of the contour levels in increasing order.   
  double *z=(double *)malloc(nc*sizeof(double));
  double min=Min(l), max=Max(l);
  double step=(max-min)/nc;
  if (min == 0 && max == 0) return;

  for (int i=0;i<nc;i++) {
    z[i]=(i+1)*step;
  }
  double *x=(double *)malloc(sizex*sizeof(double));
  for (int i=0;i<sizex;i++) {
    x[i]=i;
  }
  double *y=(double *)malloc(sizey*sizeof(double));
  for (int i=0;i<sizey;i++) {
    y[i]=i;
  }

  conrec(&PDEvars[l*sizex*sizey],0,sizex-1,0,sizey-1,x,y,nc,z,g,colour);
  
  free(x);
  free(y);
  free(z);
}



int MapColour3(double val, int l) {
	int step=0;
	if (l==2){
	step = (240)/par.max_Act;
  return (int)(256-val*step-1);}
	else if (l==3 && val>1){
	step = (256)/2;
  return (int)(500+val*step);}
	else
	return 0;
}


void PDE::PlotInCells (Graphics *g, CellularPotts *cpm, const int l) {
  for (int x=0;x<sizex;x++) {
    for (int y=0;y<sizey;y++) {
      if( cpm->Sigma(x,y)>0) {
        if (par.lambda_Act>0) {
	   g->Rectangle(MapColour3(cpm->actPixels[{x,y}],l), x, y);
        } else {
	  g->Rectangle(255, x, y);
        }
        if (par.lambda_matrix>0) {
          if (cpm->matrix[x][y]>0) {
            g->Rectangle(256, x, y);
          }
        }
      } else if (cpm->Sigma(x,y)==-2) {
        g->Rectangle(10, x, y);
      }
      if (cpm->Sigma(x,y)==-3) {
	g->Rectangle(256, x, y);
      }
    }
  }
}



void PDE::SetupOpenCL(){
  extern CLManager clm;

  program = clm.make_program(par.opencl_core_path, OPENCL_PDE_TYPE);

  //Secretion and diffusion variables
  PDEFIELD_TYPE dt = (PDEFIELD_TYPE) par.dt;
  PDEFIELD_TYPE dx2 = (PDEFIELD_TYPE) par.dx*par.dx;
  PDEFIELD_TYPE decay_rate = (PDEFIELD_TYPE) * par.decay_rate;
  PDEFIELD_TYPE secr_rate = (PDEFIELD_TYPE) * par.secr_rate;
  
  
  int btype = 1;
  if (par.periodic_boundaries) btype=2;

  //Allocate memory on the GPU
  clm.cpm = cl::Buffer(clm.context, CL_MEM_READ_WRITE, sizeof(int)*sizex*sizey); 
  clm.numberofedges = cl::Buffer(clm.context, CL_MEM_READ_WRITE, sizeof(int)*sizex*sizey); 
  clm.couplingcoefficient = cl::Buffer(clm.context, CL_MEM_READ_WRITE, sizeof(PDEFIELD_TYPE)*sizex*sizey); 
  clm.pdeA = cl::Buffer(clm.context, CL_MEM_READ_WRITE, sizeof(PDEFIELD_TYPE)*sizex*sizey*layers);
  clm.pdeB = cl::Buffer(clm.context, CL_MEM_READ_WRITE, sizeof(PDEFIELD_TYPE)*sizex*sizey*layers); 
  clm.diffco = cl::Buffer(clm.context, CL_MEM_READ_WRITE, sizeof(PDEFIELD_TYPE)*layers);


  //Making kernel and setting arguments
  kernel_ODEstep = cl::Kernel(program,"ODEstep");      

  kernel_ODEstep.setArg(0, clm.cpm);
  kernel_ODEstep.setArg(1, clm.pdeA);
  kernel_ODEstep.setArg(2, clm.pdeB);
  kernel_ODEstep.setArg(3, sizeof(int), &sizex);
  kernel_ODEstep.setArg(4, sizeof(int), &sizey);
  kernel_ODEstep.setArg(5, sizeof(int), &layers);
  kernel_ODEstep.setArg(6, sizeof(PDEFIELD_TYPE), &decay_rate);
  kernel_ODEstep.setArg(7, sizeof(PDEFIELD_TYPE), &dt);
  kernel_ODEstep.setArg(8, sizeof(PDEFIELD_TYPE), &dx2);
  kernel_ODEstep.setArg(9, clm.diffco);
  kernel_ODEstep.setArg(10,sizeof(PDEFIELD_TYPE), &secr_rate);
  kernel_ODEstep.setArg(11, sizeof(int),  &btype);
  kernel_ODEstep.setArg(12, clm.numberofedges);
  kernel_ODEstep.setArg(13, clm.couplingcoefficient);


  PDEFIELD_TYPE diff_coeff[layers];

  for (int index = 0; index < layers; index++){
    diff_coeff[index] = (PDEFIELD_TYPE) par.diff_coeff[index];
  }

  clm.queue.enqueueWriteBuffer(clm.diffco,
    CL_TRUE, 0, sizeof(PDEFIELD_TYPE)*layers, diff_coeff);

  openclsetup = true;
} 


void PDE::ODEstepCL(CellularPotts *cpm, int repeat){
    extern CLManager clm; 
    if (!openclsetup ){this->SetupOpenCL();}
    //A B scheme used to keep arrays on GPU
    int errorcode = 0;

    
    
    //Write the cellSigma array to GPU for secretion
    clm.queue.enqueueWriteBuffer(clm.cpm,
    CL_TRUE, 0, sizeof(int)*sizex*sizey, cpm->getSigma()[0]);
    clm.queue.enqueueWriteBuffer(clm.numberofedges,
    CL_TRUE, 0, sizeof(int)*sizex*sizey, cpm->getNumberofedges()[0]);
    clm.queue.enqueueWriteBuffer(clm.couplingcoefficient,
    CL_TRUE, 0, sizeof(PDEFIELD_TYPE)*sizex*sizey, cpm->getCouplingCoefficient()[0]);
    //Writing pdefield PDEvars is only necessary if modified outside of clm.pdeA)kernel
    if (first_round) {
      clm.queue.enqueueWriteBuffer(clm.pdeA,  CL_TRUE, 0, sizeof(PDEFIELD_TYPE)*sizex*sizey*layers, PDEvars);
      first_round = false;
    }
    //Main loop executing kernel and switching between A and B arrays
    for (int index = 0; index < repeat; index ++){
      for (int innerloop = 0; innerloop < 4; innerloop++){

          using std::chrono::high_resolution_clock;
          using std::chrono::duration_cast;
          using std::chrono::duration;
          using std::chrono::milliseconds;

          auto t1 = high_resolution_clock::now();

        kernel_ODEstep.setArg(14, sizeof(int), &PDEsteps);
        if(innerloop == 0 || innerloop == 1){
          kernel_ODEstep.setArg(1, clm.pdeA);
          kernel_ODEstep.setArg(2, clm.pdeB);
        }
        else{
          kernel_ODEstep.setArg(1, clm.pdeB);
          kernel_ODEstep.setArg(2, clm.pdeA);
        }
        if(innerloop == 0 || innerloop == 2){
          errorcode = clm.queue.enqueueNDRangeKernel(kernel_ODEstep,
                      cl::NullRange, cl::NDRange(sizex*sizey), cl::NullRange);
          errorcode = clm.queue.finish();
        }
        else{
          errorcode = clm.queue.enqueueNDRangeKernel(kernel_ODEstep,
                      cl::NullRange, cl::NDRange(1), cl::NullRange);
          errorcode = clm.queue.finish();
        }
        if (errorcode != 0){
          printf("Error during OpenCL secretion and diffusion");
          exit(0);
        }

        auto t2 = high_resolution_clock::now();
        duration<double, std::milli> ms_double = t2 - t1;
        cout << "For PDEsteps = " << PDEsteps << ", " << ms_double.count() << " ms has elapsed" << endl;
        PDEsteps += 1;

      }
    }
    //Reading from correct array containing the output
    if (clm.pde_AB == 0) {
      clm.queue.enqueueReadBuffer(clm.pdeB,CL_TRUE,0,
                            sizeof(PDEFIELD_TYPE)*sizex*sizey*layers, PDEvars);
    }
    else {
      clm.queue.enqueueReadBuffer(clm.pdeA,CL_TRUE,0,
                            sizeof(PDEFIELD_TYPE)*sizex*sizey*layers, PDEvars);
    }
    if (errorcode != CL_SUCCESS) cout << "error:" << errorcode << endl;
    
}

void PDE::InitializeCuda(CellularPotts *cpm){
  //AllocateTridiagonalvars(sizex, sizey);

  hipMalloc((void**) &d_couplingcoefficient, sizex*sizey*sizeof(PDEFIELD_TYPE));
  hipMalloc((void**) &d_celltype, sizex*sizey*sizeof(int));

  hipMalloc((void**) &d_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE));
  hipMemcpy(d_PDEvars, PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyHostToDevice);
  hipMalloc((void**) &d_alt_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE));
  hipMemcpy(d_alt_PDEvars, alt_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyHostToDevice);


  //Needed for ADI steps
  gpuErrchk(hipMallocManaged(&upperH, sizex*sizey*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&diagH, sizex*sizey*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&lowerH, sizex*sizey*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&BH, sizex*sizey*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&XH, sizex*sizey*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&upperV, sizey*sizex*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&diagV, sizey*sizex*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&lowerV, sizey*sizex*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&BV, sizey*sizex*sizeof(PDEFIELD_TYPE)));
  gpuErrchk(hipMallocManaged(&next_stepsize, sizey*sizex*sizeof(PDEFIELD_TYPE)));

  handleH = 0;
  pbuffersizeH = 0;
  pbufferH = NULL;
  statusH=hipsparseCreate(&handleH);
  hipsparseSgtsvInterleavedBatch_bufferSizeExt(handleH, 0, sizex, lowerH, diagH, upperH, BH, sizey, &pbuffersizeH); //Compute required buffersize for horizontal sweep

  gpuErrchk(hipMalloc( &pbufferH, sizeof(char)* pbuffersizeH));
  

  handleV = 0;
  pbuffersizeV = 0;
  pbufferV = NULL;
  statusV=hipsparseCreate(&handleV);
  hipsparseSgtsvInterleavedBatch_bufferSizeExt(handleV, 0, sizey, lowerV, diagV, upperV, BV, sizex, &pbuffersizeV); //Compute required buffersize for vertical sweep
  gpuErrchk(hipMalloc( &pbufferV, sizeof(char)* pbuffersizeV));

  

}

__global__ void InitializeLastStepsize(PDEFIELD_TYPE min_stepsize, PDEFIELD_TYPE* next_stepsize, int sizex, int sizey){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < sizex*sizey; id += stride){
    next_stepsize[id] = min_stepsize;
  }
}


void PDE::InitializePDEs(CellularPotts *cpm){
  InitializePDEvars(cpm);
  InitializeCuda(cpm);
  InitializeLastStepsize<<<par.number_of_cores, par.threads_per_core>>>(min_stepsize, next_stepsize, sizex, sizey);
  hipDeviceSynchronize();
}




__global__ void InitializeDiagonals(int sizex, int sizey, PDEFIELD_TYPE twooverdt, PDEFIELD_TYPE dx2, PDEFIELD_TYPE* lowerH, PDEFIELD_TYPE* upperH, PDEFIELD_TYPE* diagH, PDEFIELD_TYPE* lowerV, PDEFIELD_TYPE* upperV, PDEFIELD_TYPE* diagV, PDEFIELD_TYPE* couplingcoefficient){
  //This function could in theory be parellelized further, split into 6 (each part only assigning 1 value.), but this is probably slower
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int xloc; //position we currently want to assign to
  int yloc;
  int idcc; //id corresponding to the couplingcoefficient (+sizey to get the value right, +1 to get the value above)
  for (int id = index; id < sizex*sizey; id += stride){
    xloc = id/sizey; //needed to obtain interleaved format
    yloc = id%sizey;
    idcc = xloc*sizey + yloc;
    if(xloc == 0){
      lowerH[id] = 0;
      diagH[id] = couplingcoefficient[idcc+sizey]/dx2 + twooverdt;
      upperH[id] = -couplingcoefficient[idcc+sizey]/dx2;  
    }
    else if(xloc == sizex -1){
      lowerH[id] = -couplingcoefficient[idcc-sizey]/dx2;
      diagH[id] = couplingcoefficient[idcc-sizey]/dx2 + twooverdt;
      upperH[id] = 0;
    }
    else{
      lowerH[id] = -couplingcoefficient[idcc-sizey]/dx2;
      diagH[id] = (couplingcoefficient[idcc+sizey]+couplingcoefficient[idcc-sizey])/dx2 + twooverdt;
      upperH[id] = -couplingcoefficient[idcc+sizey]/dx2;
    }

    xloc = id%sizex; //needed to obtain interleaved format
    yloc = id/sizex;
    idcc = xloc*sizey + yloc;
    if(yloc == 0){
      lowerV[id] = 0;
      diagV[id] = couplingcoefficient[idcc+1]/dx2 + twooverdt;
      upperV[id] = -couplingcoefficient[idcc+1]/dx2;
    }
    else if(yloc == sizey -1){
      lowerV[id] = -couplingcoefficient[idcc-1]/dx2;
      diagV[id] = couplingcoefficient[idcc-1]/dx2 + twooverdt;
      upperV[id] = 0;
    }
    else{
      lowerV[id] = -couplingcoefficient[idcc-1]/dx2;
      diagV[id] = (couplingcoefficient[idcc+1]+couplingcoefficient[idcc-1])/dx2 + twooverdt;
      upperV[id] = -couplingcoefficient[idcc+1]/dx2;
      
    }
  }
}

__global__ void InitializeHorizontalVectors(int sizex, int sizey, PDEFIELD_TYPE twooverdt, PDEFIELD_TYPE dx2, PDEFIELD_TYPE* BH, PDEFIELD_TYPE* couplingcoefficient, PDEFIELD_TYPE* alt_PDEvars){
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int xloc;
  int yloc;
  int idcc; //id corresponding to the couplingcoefficient and alt_PDEvars(+sizey to get the value right, +1 to get the value above)
  for (int id = index; id < sizex*sizey; id += stride){
    xloc = id/sizey; //needed to obtain interleaved format
    yloc = id%sizey;
    idcc = xloc*sizey + yloc;

    if (yloc == 0)
      BH[id] = twooverdt*alt_PDEvars[idcc] + (couplingcoefficient[idcc+1]*(alt_PDEvars[idcc+1] - alt_PDEvars[idcc]))/dx2; 
    else if (yloc == sizey-1)
      BH[id] = twooverdt*alt_PDEvars[idcc] + (couplingcoefficient[idcc-1]*(alt_PDEvars[idcc-1] - alt_PDEvars[idcc]))/dx2;  
    else 
      BH[id] = twooverdt*alt_PDEvars[idcc] + (couplingcoefficient[idcc+1]*(alt_PDEvars[idcc+1] - alt_PDEvars[idcc]) + couplingcoefficient[idcc-1]*(alt_PDEvars[idcc-1] - alt_PDEvars[idcc]))/dx2;
  }
}

__global__ void InitializeVerticalVectors(int sizex, int sizey, PDEFIELD_TYPE twooverdt, PDEFIELD_TYPE dx2, PDEFIELD_TYPE* BV, PDEFIELD_TYPE* couplingcoefficient, PDEFIELD_TYPE* alt_PDEvars){

  //This function could in theory be parellelized further, split into 6 (each part only assigning 1 value.), but this is probably slower
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int xloc;
  int yloc;
  int idcc; //id corresponding to the couplingcoefficient and alt_PDEvars(+sizey to get the value right, +1 to get the value above)
  for (int id = index; id < sizex*sizey; id += stride){
    xloc = id%sizex;
    yloc = id/sizex;
    idcc = xloc*sizey + yloc;

    if (xloc == 0)
      BV[id] = twooverdt*alt_PDEvars[idcc] + (couplingcoefficient[idcc+sizey]*(alt_PDEvars[idcc+sizey] - alt_PDEvars[idcc]))/dx2; 
    else if (xloc == sizex-1)
      BV[id] = twooverdt*alt_PDEvars[idcc] + (couplingcoefficient[idcc-sizey]*(alt_PDEvars[idcc-sizey] - alt_PDEvars[idcc]))/dx2;  
    else 
      BV[id] = twooverdt*alt_PDEvars[idcc] + (couplingcoefficient[idcc+sizey]*(alt_PDEvars[idcc+sizey] - alt_PDEvars[idcc]) + couplingcoefficient[idcc-sizey]*(alt_PDEvars[idcc-sizey] - alt_PDEvars[idcc]))/dx2;
  }

}



__global__ void NewPDEfieldH0(int sizex, int sizey, PDEFIELD_TYPE* BH, PDEFIELD_TYPE* PDEvars){ //Take the values from BH and assign the new values of the first layers of PDEvars
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < sizex*sizey; id += stride)
    PDEvars[id] = BH[id];      
}


__global__ void NewPDEfieldV0(int sizex, int sizey, PDEFIELD_TYPE* BV, PDEFIELD_TYPE* PDEvars){ //Take the values from BV and assign the new values of the first layers of PDEvars
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < sizex*sizey; id += stride){
    PDEvars[sizey*(id%sizex)+id/sizex] = BV[id]; //Conversion is needed because PDEvars iterates over columns first and then rows, while BV does the opposite 
  }
}


__global__ void NewPDEfieldOthers(int sizex, int sizey, int layers, PDEFIELD_TYPE* BH, PDEFIELD_TYPE* PDEvars, PDEFIELD_TYPE* alt_PDEvars){ //copy the other values from alt_PDEvars to PDEvars
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index+sizex*sizey; id < layers*sizex*sizey; id += stride)
    PDEvars[id] = alt_PDEvars[id]; 
}





#if 0
__global__ void RungeKuttaStepOld(PDEFIELD_TYPE dt, PDEFIELD_TYPE thetime, int layers, int sizex, int sizey, PDEFIELD_TYPE* PDEvars, PDEFIELD_TYPE* alt_PDEvars, int* celltype){
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int i;



  
  PDEFIELD_TYPE dydt[ARRAY_SIZE];
  PDEFIELD_TYPE ak2[ARRAY_SIZE];
  PDEFIELD_TYPE ak3[ARRAY_SIZE];
  PDEFIELD_TYPE ak4[ARRAY_SIZE];
  PDEFIELD_TYPE ak5[ARRAY_SIZE];
  PDEFIELD_TYPE ak6[ARRAY_SIZE];
  PDEFIELD_TYPE ytemp[ARRAY_SIZE]; 
  PDEFIELD_TYPE y[ARRAY_SIZE];
  PDEFIELD_TYPE yerr[ARRAY_SIZE];
  PDEFIELD_TYPE yout[ARRAY_SIZE];

  static PDEFIELD_TYPE a2=0.2,a3=0.3,a4=0.6,a5=1.0,a6=0.875,b21=0.2,
  b31=3.0/40.0,b32=9.0/40.0,b41=0.3,b42 = -0.9,b43=1.2,
  b51 = -11.0/54.0, b52=2.5,b53 = -70.0/27.0,b54=35.0/27.0,
  b61=1631.0/55296.0,b62=175.0/512.0,b63=575.0/13824.0,
  b64=44275.0/110592.0,b65=253.0/4096.0,c1=37.0/378.0,
  c3=250.0/621.0,c4=125.0/594.0,c6=512.0/1771.0,
  dc5 = -277.00/14336.0;
  PDEFIELD_TYPE dc1=c1-2825.0/27648.0,dc3=c3-18575.0/48384.0,
    dc4=c4-13525.0/55296.0,dc6=c6-0.25;

  //Declare variables needed for Paci2020 model and assign the constants

  //// Constants
  PDEFIELD_TYPE F = 96485.3415;     // coulomb_per_mole (in model_parameters)
  PDEFIELD_TYPE R = 8.314472;       // joule_per_mole_kelvin (in model_parameters)
  PDEFIELD_TYPE T = 310.0;          // kelvin (in model_parameters) //37°C

  //// Cell geometry
  PDEFIELD_TYPE V_SR = 583.73;        // micrometre_cube (in model_parameters)
  PDEFIELD_TYPE Vc   = 8800.0;        // micrometre_cube (in model_parameters)
  PDEFIELD_TYPE Cm   = 9.87109e-11;   // farad (in model_parameters)

  //// Extracellular concentrations
  PDEFIELD_TYPE Nao = 151.0; // millimolar (in model_parameters)
  PDEFIELD_TYPE Ko  = 5.4;   // millimolar (in model_parameters)
  PDEFIELD_TYPE Cao = 1.8;   // millimolar (in model_parameters)

  //// Intracellular concentrations
  // Naio = 10 mM y[17]
  PDEFIELD_TYPE Ki = 150.0;   // millimolar (in model_parameters)
  // Cai  = 0.0002 mM y[2]
  // caSR = 0.3 mM y[1]  
    
      //// Nernst potential
  PDEFIELD_TYPE E_Na;
  PDEFIELD_TYPE E_Ca;
  PDEFIELD_TYPE E_K;
  PDEFIELD_TYPE PkNa = 0.03;   // dimensionless (in electric_potentials)
  PDEFIELD_TYPE E_Ks;
    
  //// INa adapted from DOI:10.3389/fphys.2018.00080
  PDEFIELD_TYPE g_Na = 3671.2302; //((time<tDrugApplication)*1+(time >= tDrugApplication)*INaFRedMed)*6447.1896;
  PDEFIELD_TYPE i_Na;
    
  PDEFIELD_TYPE m_inf;
  PDEFIELD_TYPE tau_m;
    
  PDEFIELD_TYPE h_inf;
  PDEFIELD_TYPE tau_h;
    
  PDEFIELD_TYPE j_inf;
  PDEFIELD_TYPE tau_j;
    
    
  //// INaL
  PDEFIELD_TYPE myCoefTauM  = 1;
  PDEFIELD_TYPE tauINaL = 200; //ms
  PDEFIELD_TYPE GNaLmax = 17.25;//((time<tDrugApplication)*1+(time >= tDrugApplication)*INaLRedMed)* 2.3*7.5; //(S/F)
  PDEFIELD_TYPE Vh_hLate = 87.61;
  PDEFIELD_TYPE i_NaL;
    
  PDEFIELD_TYPE m_inf_L;
  PDEFIELD_TYPE alpha_m_L;
  PDEFIELD_TYPE beta_m_L;
  PDEFIELD_TYPE tau_m_L;
    
  PDEFIELD_TYPE h_inf_L;
  PDEFIELD_TYPE tau_h_L = 1 * tauINaL;
    
  //// If adapted from DOI:10.3389/fphys.2018.00080
  PDEFIELD_TYPE g_f = 1; //((time<tDrugApplication)*1+(time >= tDrugApplication)*IfRedMed)*22.2763088;
  PDEFIELD_TYPE fNa = 0.37;
  PDEFIELD_TYPE fK = 1 - fNa;
  PDEFIELD_TYPE i_fK;
  PDEFIELD_TYPE i_fNa;
  PDEFIELD_TYPE i_f;
    
  PDEFIELD_TYPE Xf_infinity;
  PDEFIELD_TYPE tau_Xf; 
    
      //// ICaL
  PDEFIELD_TYPE g_CaL = 8.635702e-5;   // metre_cube_per_F_per_s (in i_CaL)
  PDEFIELD_TYPE i_CaL;  
  PDEFIELD_TYPE precision = 0.0001;     
    
  PDEFIELD_TYPE d_infinity;
  PDEFIELD_TYPE alpha_d;
  PDEFIELD_TYPE beta_d;
  PDEFIELD_TYPE gamma_d;
  PDEFIELD_TYPE tau_d;
    
  PDEFIELD_TYPE f1_inf;
  PDEFIELD_TYPE constf1;
    
  PDEFIELD_TYPE tau_f1;
    
  PDEFIELD_TYPE f2_inf;
  PDEFIELD_TYPE constf2 = 1.0;
  PDEFIELD_TYPE tau_f2;
    
  PDEFIELD_TYPE alpha_fCa;
  PDEFIELD_TYPE beta_fCa;
  PDEFIELD_TYPE gamma_fCa;
  PDEFIELD_TYPE fCa_inf;
    
  PDEFIELD_TYPE constfCa;
    
  PDEFIELD_TYPE tau_fCa     = 0.002;   // second (in i_CaL_fCa_gate)
    
  //// Ito
  PDEFIELD_TYPE g_to = 29.9038;   // S_per_F (in i_to)  
  PDEFIELD_TYPE i_to;
    
  PDEFIELD_TYPE q_inf;
  PDEFIELD_TYPE tau_q;
    
    
  PDEFIELD_TYPE r_inf;
  PDEFIELD_TYPE tau_r;
    
  //// IKs
  PDEFIELD_TYPE g_Ks = 2.041;   // S_per_F (in i_Ks)
  PDEFIELD_TYPE i_Ks; // ((time<tDrugApplication)*1+(time >= tDrugApplication)*IKsRedMed)*g_Ks*(y[0]-E_Ks)*pow((float)y[10],2.0)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/y[2]),1.4f)));
    
  PDEFIELD_TYPE Xs_infinity;
  PDEFIELD_TYPE alpha_Xs;
  PDEFIELD_TYPE beta_Xs;
  PDEFIELD_TYPE tau_Xs;
    
  //// IKr
  PDEFIELD_TYPE L0 = 0.025;   // dimensionless (in i_Kr_Xr1_gate)
  PDEFIELD_TYPE Q = 2.3;     // dimensionless (in i_Kr_Xr1_gate)
  PDEFIELD_TYPE g_Kr = 29.8667;   // S_per_F (in i_Kr)
  PDEFIELD_TYPE i_Kr;
    
  PDEFIELD_TYPE V_half;
    
  PDEFIELD_TYPE Xr1_inf;
  PDEFIELD_TYPE alpha_Xr1;
  PDEFIELD_TYPE beta_Xr1;
  PDEFIELD_TYPE tau_Xr1;
    
  PDEFIELD_TYPE Xr2_infinity;
  PDEFIELD_TYPE alpha_Xr2;
  PDEFIELD_TYPE beta_Xr2;
  PDEFIELD_TYPE tau_Xr2;
    
  //// IK1
  PDEFIELD_TYPE alpha_K1;
  PDEFIELD_TYPE beta_K1;
  PDEFIELD_TYPE XK1_inf;
  PDEFIELD_TYPE g_K1 = 28.1492;   // S_per_F (in i_K1)
  PDEFIELD_TYPE i_K1;
    
  //// INaCa
  PDEFIELD_TYPE KmCa = 1.38;   // millimolar (in i_NaCa)
  PDEFIELD_TYPE KmNai = 87.5;   // millimolar (in i_NaCa)
  PDEFIELD_TYPE Ksat = 0.1;    // dimensionless (in i_NaCa)
  PDEFIELD_TYPE gamma = 0.35;   // dimensionless (in i_NaCa)
  PDEFIELD_TYPE alpha = 2.16659;
  PDEFIELD_TYPE kNaCa = 3917.0463; //((time<tDrugApplication)*1+(time >= tDrugApplication)*INaCaRedMed) * 6514.47574;   // A_per_F (in i_NaCa)
  PDEFIELD_TYPE i_NaCa;

  //// INaK
  PDEFIELD_TYPE Km_K = 1.0;    // millimolar (in i_NaK)
  PDEFIELD_TYPE Km_Na = 40.0;   // millimolar (in i_NaK)
  PDEFIELD_TYPE PNaK = 2.74240;// A_per_F (in i_NaK)
  PDEFIELD_TYPE i_NaK;
    
  //// IpCa
  PDEFIELD_TYPE KPCa = 0.0005;   // millimolar (in i_PCa)
  PDEFIELD_TYPE g_PCa = 0.4125;   // A_per_F (in i_PCa)
  PDEFIELD_TYPE i_PCa;
    
  //// Background currents
  PDEFIELD_TYPE g_b_Na = 1.14;         // S_per_F (in i_b_Na)
  PDEFIELD_TYPE i_b_Na;
    
  PDEFIELD_TYPE g_b_Ca = 0.8727264;    // S_per_F (in i_b_Ca)
  PDEFIELD_TYPE i_b_Ca;

  PDEFIELD_TYPE i_up;
  PDEFIELD_TYPE i_leak;
    
  //// Sarcoplasmic reticulum
  PDEFIELD_TYPE VmaxUp = 0.82205;
  PDEFIELD_TYPE Kup	= 4.40435e-4;
    
  PDEFIELD_TYPE V_leak = 4.48209e-4;
    
  // RyR
  PDEFIELD_TYPE g_irel_max = 55.808061;
  PDEFIELD_TYPE RyRa1 = 0.05169;
  PDEFIELD_TYPE RyRa2 = 0.050001;
  PDEFIELD_TYPE RyRahalf = 0.02632;
  PDEFIELD_TYPE RyRohalf = 0.00944;
  PDEFIELD_TYPE RyRchalf = 0.00167;
    
  PDEFIELD_TYPE RyRSRCass;
  PDEFIELD_TYPE i_rel;
    
  PDEFIELD_TYPE RyRainfss;
  PDEFIELD_TYPE RyRtauadapt = 1; //s
    
  PDEFIELD_TYPE RyRoinfss;
  PDEFIELD_TYPE RyRtauact;
    
  PDEFIELD_TYPE RyRcinfss;
  PDEFIELD_TYPE RyRtauinact;

  //// Ca2+ buffering
  PDEFIELD_TYPE Buf_C = 0.25;   // millimolar (in calcium_dynamics)
  PDEFIELD_TYPE Buf_SR = 10.0;   // millimolar (in calcium_dynamics)
  PDEFIELD_TYPE Kbuf_C = 0.001;   // millimolar (in calcium_dynamics)
  PDEFIELD_TYPE Kbuf_SR = 0.3;   // millimolar (in calcium_dynamics)
  PDEFIELD_TYPE Cai_bufc;
  PDEFIELD_TYPE Ca_SR_bufSR;
    
      //// Stimulation
    //  PDEFIELD_TYPE i_stim_Amplitude 		= 5.5e-10;//7.5e-10;   // ampere (in stim_mode)
    //  PDEFIELD_TYPE i_stim_End 				= 1000.0;   // second (in stim_mode)
    //  PDEFIELD_TYPE i_stim_PulseDuration	= 0.005;   // second (in stim_mode)
    //  PDEFIELD_TYPE i_stim_Start 			= 0.0;   // second (in stim_mode)
    //  PDEFIELD_TYPE i_stim_frequency        = 60.0;   // per_second (in stim_mode)
      //PDEFIELD_TYPE stim_flag 				= stimFlag;   // dimensionless (in stim_mode)
    //  PDEFIELD_TYPE i_stim_Period 			= 60.0/i_stim_frequency;
    
      //if stim_flag~=0 && stim_flag~=1
      //error('Paci2020: wrong pacing! stimFlag can be only 0 (spontaneous) or 1 (paced)');
      //end
    
      /*
      if ((time >= i_stim_Start) && (time <= i_stim_End) && (time-i_stim_Start-floor((time-i_stim_Start)/i_stim_Period)*i_stim_Period <= i_stim_PulseDuration))
          i_stim = stim_flag*i_stim_Amplitude/Cm;
      else
          i_stim = 0.0;
      */
  PDEFIELD_TYPE i_stim = 0;
  
  for (int id = index; id < sizex*sizey; id += stride){
    if (celltype[id] < 1)
      for (int l = 0; l < layers; l++) //fill with current PDE values
        alt_PDEvars[l*sizex*sizey + id]= PDEvars[l*sizex*sizey + id];
    else{    
      for (int l = 0; l < layers; l++) //fill with current PDE values
        y[l] = PDEvars[l*sizex*sizey + id];

      //-----FIRST STEP ------------------------------------------------------------------------------------------------------------------------------------------------------------------  
      
        //// Nernst potential
      E_Na = R*T/F*log(Nao/y[17]);
      E_Ca = 0.5*R*T/F*log(Cao/y[2]);
      E_K  = R*T/F*log(Ko/Ki);
      E_Ks = R*T/F*log((Ko+PkNa*Nao)/(Ki+PkNa*y[17]));
    
      //// INa adapted from DOI:10.3389/fphys.2018.00080
      i_Na        =  g_Na*pow((float)y[13],3.0f)*y[11]*y[12]*(y[0] - E_Na);
    
      m_inf       = 1 / (1 + exp((y[0]*1000 + 39)/-11.2));
      tau_m       = (0.00001 + 0.00013*exp(-pow((float)((y[0]*1000 + 48)/15),2.0f)) + 0.000045 / (1 + exp((y[0]*1000 + 42)/-5)));
      dydt[13]   = (m_inf-y[13])/tau_m;
    
      h_inf       = 1 / (1 + exp((y[0]*1000 + 66.5)/6.8));
      tau_h       = (0.00007 + 0.034 / (1 + exp((y[0]*1000 + 41)/5.5) + exp(-(y[0]*1000 + 41)/14)) + 0.0002 / (1 + exp(-(y[0]*1000 + 79)/14)));
      dydt[11]   = (h_inf-y[11])/tau_h;
    
      j_inf       = h_inf;
      tau_j       = 10*(0.0007 + 0.15 / (1 + exp((y[0]*1000 + 41)/5.5) + exp(-(y[0]*1000 + 41)/14)) + 0.002 / (1 + exp(-(y[0]*1000 + 79)/14)));
      dydt[12]   = (j_inf-y[12])/tau_j;
    
    
      //// INaL
      tauINaL     = 200; //ms
      GNaLmax     = 17.25;//((time<tDrugApplication)*1+(time >= tDrugApplication)*INaLRedMed)* 2.3*7.5; //(S/F)
      Vh_hLate    = 87.61;
      i_NaL       = GNaLmax* pow((float)y[18],3.0f)*y[19]*(y[0]-E_Na);
    
      m_inf_L     = 1/(1+exp(-(y[0]*1000+42.85)/(5.264)));
      alpha_m_L   = 1/(1+exp((-60-y[0]*1000)/5));
      beta_m_L    = 0.1/(1+exp((y[0]*1000+35)/5))+0.1/(1+exp((y[0]*1000-50)/200));
      tau_m_L     = 1 * myCoefTauM*alpha_m_L*beta_m_L;
      dydt[18]   = (m_inf_L-y[18])/tau_m_L*1000;
    
      h_inf_L     = 1/(1+exp((y[0]*1000+Vh_hLate)/(7.488)));
      tau_h_L     = 1 * tauINaL;
      dydt[19]   = (h_inf_L-y[19])/tau_h_L*1000;
    
      //// If adapted from DOI:10.3389/fphys.2018.00080
      i_fK        = fK*g_f*y[14]*(y[0] - E_K);
      i_fNa       = fNa*g_f*y[14]*(y[0] - E_Na);
      i_f         = i_fK + i_fNa;
    
      Xf_infinity = 1.0/(1.0 + exp((y[0]*1000 + 69)/8));
      tau_Xf      = 5600 / (1 + exp((y[0]*1000 + 65)/7) + exp(-(y[0]*1000 + 65)/19));
      dydt[14]   = 1000*(Xf_infinity-y[14])/tau_Xf;
      
    
      //// ICaL
      //Prevent division by 0
      if(y[0]< precision && y[0] > -precision) //hopital
        i_CaL =  g_CaL*4.0*y[0]*pow((float)F,2.0f)/(R*T) *y[4]*y[5]*y[6]*y[7] / (2.0*F/(R*T)) * (y[2] - 0.341*Cao);
      else
        i_CaL = g_CaL*4.0*y[0]*pow((float)F,2.0f)/(R*T)*(y[2]*exp(2.0*y[0]*F/(R*T))-0.341*Cao)/(exp(2.0*y[0]*F/(R*T))-1.0)*y[4]*y[5]*y[6]*y[7]; //((time<tDrugApplication)*1+(time >= tDrugApplication)*ICaLRedMed)*g_CaL*4.0*y[0]*pow(F,2.0)/(R*T)*(y[2]*exp(2.0*y[0]*F/(R*T))-0.341*Cao)/(exp(2.0*y[0]*F/(R*T))-1.0)*y[4]*y[5]*y[6]*y[7];
    
      d_infinity  = 1.0/(1.0+exp(-(y[0]*1000.0+9.1)/7.0));
      alpha_d     = 0.25+1.4/(1.0+exp((-y[0]*1000.0-35.0)/13.0));
      beta_d      = 1.4/(1.0+exp((y[0]*1000.0+5.0)/5.0));
      gamma_d     = 1.0/(1.0+exp((-y[0]*1000.0+50.0)/20.0));
      tau_d       = (alpha_d*beta_d+gamma_d)*1.0/1000.0;
      dydt[4]    = (d_infinity-y[4])/tau_d;
    
      f1_inf      = 1.0/(1.0+exp((y[0]*1000.0+26.0)/3.0));
      if (f1_inf-y[5] > 0.0)
          constf1 = 1.0+1433.0*(y[2]-50.0*1.0e-6);
      else
          constf1 = 1.0;
    
      tau_f1      = (20.0+1102.5*exp(-pow((float)((y[0]*1000.0+27.0)/15.0),2.0f))+200.0/(1.0+exp((13.0-y[0]*1000.0)/10.0))+180.0/(1.0+exp((30.0+y[0]*1000.0)/10.0)))*constf1/1000.0;
      dydt[5]    = (f1_inf-y[5])/tau_f1;
    
      f2_inf      = 0.33+0.67/(1.0+exp((y[0]*1000.0+32.0)/4.0));
      tau_f2      = (600.0*exp(-pow((float)(y[0]*1000.0+25.0),2.0f)/170.0)+31.0/(1.0+exp((25.0-y[0]*1000.0)/10.0))+16.0/(1.0+exp((30.0+y[0]*1000.0)/10.0)))*constf2/1000.0;
      dydt[6]    = (f2_inf-y[6])/tau_f2;
    
      alpha_fCa   = 1.0/(1.0+pow((float)(y[2]/0.0006),8.0f));
      beta_fCa    = 0.1/(1.0+exp((y[2]-0.0009)/0.0001));
      gamma_fCa   = 0.3/(1.0+exp((y[2]-0.00075)/0.0008));
      fCa_inf     = (alpha_fCa+beta_fCa+gamma_fCa)/1.3156;
    
      if ((y[0] > -0.06) && (fCa_inf > y[7]))
          constfCa = 0.0;
      else
          constfCa = 1.0;
    
      dydt[7]    = constfCa*(fCa_inf-y[7])/tau_fCa;
    
      //// Ito
      i_to        = g_to*(y[0]-E_K)*y[15]*y[16];
    
      q_inf       = 1.0/(1.0+exp((y[0]*1000.0+53.0)/13.0));
      tau_q       = (6.06+39.102/(0.57*exp(-0.08*(y[0]*1000.0+44.0))+0.065*exp(0.1*(y[0]*1000.0+45.93))))/1000.0;
      dydt[15]   = (q_inf-y[15])/tau_q;
    
    
      r_inf       = 1.0/(1.0+exp(-(y[0]*1000.0-22.3)/18.75));
      tau_r       = (2.75352+14.40516/(1.037*exp(0.09*(y[0]*1000.0+30.61))+0.369*exp(-0.12*(y[0]*1000.0+23.84))))/1000.0;
      dydt[16]   = (r_inf-y[16])/tau_r;
    
      //// IKs
      i_Ks        = g_Ks*(y[0]-E_Ks)*pow((float)y[10],2.0f)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/y[2]),1.4f))); // ((time<tDrugApplication)*1+(time >= tDrugApplication)*IKsRedMed)*g_Ks*(y[0]-E_Ks)*pow((float)y[10],2.0)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/y[2]),1.4f)));
    
      Xs_infinity = 1.0/(1.0+exp((-y[0]*1000.0-20.0)/16.0));
      alpha_Xs    = 1100.0/sqrt(1.0+exp((-10.0-y[0]*1000.0)/6.0));
      beta_Xs     = 1.0/(1.0+exp((-60.0+y[0]*1000.0)/20.0));
      tau_Xs      = 1.0*alpha_Xs*beta_Xs/1000.0;
      dydt[10]   = (Xs_infinity-y[10])/tau_Xs;
    
      //// IKr
      i_Kr         = g_Kr*(y[0]-E_K)*y[8]*y[9]*sqrt(Ko/5.4); //((time<tDrugApplication)*1+(time >= tDrugApplication)*IKrRedMed)*g_Kr*(y[0]-E_K)*y[8]*y[9]*sqrt(Ko/5.4);
    
      V_half       = 1000.0*(-R*T/(F*Q)*log(pow((float)(1.0+Cao/2.6),4.0f)/(pow((float)(1.0+Cao/0.58),4.0f)*L0))-0.019);
    
      Xr1_inf      = 1.0/(1.0+exp((V_half-y[0]*1000.0)/4.9));
      alpha_Xr1    = 450.0/(1.0+exp((-45.0-y[0]*1000.0)/10.0));
      beta_Xr1     = 6.0/(1.0+exp((30.0+y[0]*1000.0)/11.5));
      tau_Xr1      = 1.0*alpha_Xr1*beta_Xr1/1000.0;
      dydt[8]     = (Xr1_inf-y[8])/tau_Xr1;
    
      Xr2_infinity = 1.0/(1.0+exp((y[0]*1000.0+88.0)/50.0));
      alpha_Xr2    = 3.0/(1.0+exp((-60.0-y[0]*1000.0)/20.0));
      beta_Xr2     = 1.12/(1.0+exp((-60.0+y[0]*1000.0)/20.0));
      tau_Xr2      = 1.0*alpha_Xr2*beta_Xr2/1000.0;
      dydt[9]    = (Xr2_infinity-y[9])/tau_Xr2;
    
      //// IK1
      alpha_K1    = 3.91/(1.0+exp(0.5942*(y[0]*1000.0-E_K*1000.0-200.0)));
      beta_K1     = (-1.509*exp(0.0002*(y[0]*1000.0-E_K*1000.0+100.0))+exp(0.5886*(y[0]*1000.0-E_K*1000.0-10.0)))/(1.0+exp(0.4547*(y[0]*1000.0-E_K*1000.0)));
      XK1_inf     = alpha_K1/(alpha_K1+beta_K1);
      i_K1        = g_K1*XK1_inf*(y[0]-E_K)*sqrt(Ko/5.4);
    
      //// INaCa
      i_NaCa      = kNaCa*(exp(gamma*y[0]*F/(R*T))*pow((float)y[17],3.0f)*Cao-exp((gamma-1.0)*y[0]*F/(R*T))*pow((float)Nao,3.0f)*y[2]*alpha)/((pow((float)KmNai,3.0f)+pow((float)Nao,3.0f))*(KmCa+Cao)*(1.0+Ksat*exp((gamma-1.0)*y[0]*F/(R*T))));
    
      //// INaK
      i_NaK       = PNaK*Ko/(Ko+Km_K)*y[17]/(y[17]+Km_Na)/(1.0+0.1245*exp(-0.1*y[0]*F/(R*T))+0.0353*exp(-y[0]*F/(R*T)));
    
      //// IpCa
      i_PCa       = g_PCa*y[2]/(y[2]+KPCa);
    
      //// Background currents
      i_b_Na      = g_b_Na*(y[0]-E_Na);
    
      i_b_Ca      = g_b_Ca*(y[0]-E_Ca);
    
      //// Sarcoplasmic reticulum
      i_up        = VmaxUp/(1.0+pow((float)Kup,2.0f)/pow((float)y[2],2.0f));
    
      i_leak      = (y[1]-y[2])*V_leak;
    
      dydt[3]    = 0;
    
      // RyR
    
      RyRSRCass   = (1 - 1/(1 +  exp((y[1]-0.3)/0.1)));
      i_rel       = g_irel_max*RyRSRCass*y[21]*y[22]*(y[1]-y[2]);
    
      RyRainfss   = RyRa1-RyRa2/(1 + exp((1000*y[2]-(RyRahalf))/0.0082));
      dydt[20]   = (RyRainfss- y[20])/RyRtauadapt;
    
      RyRoinfss   = (1 - 1/(1 +  exp((1000*y[2]-(y[20]+ RyRohalf))/0.003)));
      if (RyRoinfss>= y[21])
        RyRtauact = 18.75e-3;       //s
      else
        RyRtauact = 0.1*18.75e-3;   //s
    
      dydt[21]    = (RyRoinfss- y[21])/(RyRtauact);
    
      RyRcinfss   = (1/(1 + exp((1000*y[2]-(y[20]+RyRchalf))/0.001)));
      if (RyRcinfss>= y[22])
        RyRtauinact = 2*87.5e-3;    //s
      else
        RyRtauinact = 87.5e-3;      //s
    
      dydt[22]    = (RyRcinfss- y[22])/(RyRtauinact);
    
    
    
    
      //// Ca2+ buffering
      Cai_bufc    = 1.0/(1.0+Buf_C*Kbuf_C/pow((float)(y[2]+Kbuf_C), 2.0f));
      Ca_SR_bufSR = 1.0/(1.0+Buf_SR*Kbuf_SR/pow((float)(y[1]+Kbuf_SR), 2.0f));
    
      //// Ionic concentrations
      //Nai
      dydt[17]   = -Cm*(i_Na+i_NaL+i_b_Na+3.0*i_NaK+3.0*i_NaCa+i_fNa)/(F*Vc*1.0e-18);
      //Cai
      dydt[2]    = Cai_bufc*(i_leak-i_up+i_rel-(i_CaL+i_b_Ca+i_PCa-2.0*i_NaCa)*Cm/(2.0*Vc*F*1.0e-18));
      //caSR
      dydt[1]    = Ca_SR_bufSR*Vc/V_SR*(i_up-(i_rel+i_leak));
    
      //// Stimulation
    //  PDEFIELD_TYPE i_stim_Amplitude 		= 5.5e-10;//7.5e-10;   // ampere (in stim_mode)
    //  PDEFIELD_TYPE i_stim_End 				= 1000.0;   // second (in stim_mode)
    //  PDEFIELD_TYPE i_stim_PulseDuration	= 0.005;   // second (in stim_mode)
    //  PDEFIELD_TYPE i_stim_Start 			= 0.0;   // second (in stim_mode)
    //  PDEFIELD_TYPE i_stim_frequency        = 60.0;   // per_second (in stim_mode)
      //PDEFIELD_TYPE stim_flag 				= stimFlag;   // dimensionless (in stim_mode)
    //  PDEFIELD_TYPE i_stim_Period 			= 60.0/i_stim_frequency;
    
      //if stim_flag~=0 && stim_flag~=1
      //error('Paci2020: wrong pacing! stimFlag can be only 0 (spontaneous) or 1 (paced)');
      //end
    
      /*
      if ((time >= i_stim_Start) && (time <= i_stim_End) && (time-i_stim_Start-floor((time-i_stim_Start)/i_stim_Period)*i_stim_Period <= i_stim_PulseDuration))
          i_stim = stim_flag*i_stim_Amplitude/Cm;
      else
          i_stim = 0.0;
      */
    
      //// Membrane potential
      dydt[0] = -(i_K1+i_to+i_Kr+i_Ks+i_CaL+i_NaK+i_Na+i_NaL+i_NaCa+i_PCa+i_f+i_b_Na+i_b_Ca-i_stim);


    //-----SECOND STEP ------------------------------------------------------------------------------------------------------------------------------------------------------------------  
    
    for (i=0;i<layers;i++)
      ytemp[i]=y[i]+b21*dt*dydt[i];

      E_Na = R*T/F*log(Nao/ytemp[17]);
      E_Ca = 0.5*R*T/F*log(Cao/ytemp[2]);
      E_K  = R*T/F*log(Ko/Ki);
      E_Ks = R*T/F*log((Ko+PkNa*Nao)/(Ki+PkNa*ytemp[17]));
    
      //// INa adapted from DOI:10.3389/fphys.2018.00080
      i_Na        =  g_Na*pow((float)ytemp[13],3.0f)*ytemp[11]*ytemp[12]*(ytemp[0] - E_Na);
    
      m_inf       = 1 / (1 + exp((ytemp[0]*1000 + 39)/-11.2));
      tau_m       = (0.00001 + 0.00013*exp(-pow((float)((ytemp[0]*1000 + 48)/15),2.0f)) + 0.000045 / (1 + exp((ytemp[0]*1000 + 42)/-5)));
      ak2[13]   = (m_inf-ytemp[13])/tau_m;
    
      h_inf       = 1 / (1 + exp((ytemp[0]*1000 + 66.5)/6.8));
      tau_h       = (0.00007 + 0.034 / (1 + exp((ytemp[0]*1000 + 41)/5.5) + exp(-(ytemp[0]*1000 + 41)/14)) + 0.0002 / (1 + exp(-(ytemp[0]*1000 + 79)/14)));
      ak2[11]   = (h_inf-ytemp[11])/tau_h;
    
      j_inf       = h_inf;
      tau_j       = 10*(0.0007 + 0.15 / (1 + exp((ytemp[0]*1000 + 41)/5.5) + exp(-(ytemp[0]*1000 + 41)/14)) + 0.002 / (1 + exp(-(ytemp[0]*1000 + 79)/14)));
      ak2[12]   = (j_inf-ytemp[12])/tau_j;
    
    
      //// INaL
      i_NaL       = GNaLmax* pow((float)ytemp[18],3.0f)*ytemp[19]*(ytemp[0]-E_Na);
    
      m_inf_L     = 1/(1+exp(-(ytemp[0]*1000+42.85)/(5.264)));
      alpha_m_L   = 1/(1+exp((-60-ytemp[0]*1000)/5));
      beta_m_L    = 0.1/(1+exp((ytemp[0]*1000+35)/5))+0.1/(1+exp((ytemp[0]*1000-50)/200));
      tau_m_L     = 1 * myCoefTauM*alpha_m_L*beta_m_L;
      ak2[18]   = (m_inf_L-ytemp[18])/tau_m_L*1000;
    
      h_inf_L     = 1/(1+exp((ytemp[0]*1000+Vh_hLate)/(7.488)));
      ak2[19]   = (h_inf_L-ytemp[19])/tau_h_L*1000;
    
      //// If adapted from DOI:10.3389/fphys.2018.00080
      i_fK        = fK*g_f*ytemp[14]*(ytemp[0] - E_K);
      i_fNa       = fNa*g_f*ytemp[14]*(ytemp[0] - E_Na);
      i_f         = i_fK + i_fNa;
    
      Xf_infinity = 1.0/(1.0 + exp((ytemp[0]*1000 + 69)/8));
      tau_Xf      = 5600 / (1 + exp((ytemp[0]*1000 + 65)/7) + exp(-(ytemp[0]*1000 + 65)/19));
      ak2[14]   = 1000*(Xf_infinity-ytemp[14])/tau_Xf;
      
    
      //// ICaL
      //Prevent division by 0
      if(ytemp[0]< precision && ytemp[0] > -precision) //hopital
        i_CaL =  g_CaL*4.0*ytemp[0]*pow((float)F,2.0f)/(R*T) *ytemp[4]*ytemp[5]*ytemp[6]*ytemp[7] / (2.0*F/(R*T)) * (ytemp[2] - 0.341*Cao);
      else
        i_CaL = g_CaL*4.0*ytemp[0]*pow((float)F,2.0f)/(R*T)*(ytemp[2]*exp(2.0*ytemp[0]*F/(R*T))-0.341*Cao)/(exp(2.0*ytemp[0]*F/(R*T))-1.0)*ytemp[4]*ytemp[5]*ytemp[6]*ytemp[7]; //((time<tDrugApplication)*1+(time >= tDrugApplication)*ICaLRedMed)*g_CaL*4.0*ytemp[0]*pow(F,2.0)/(R*T)*(ytemp[2]*exp(2.0*ytemp[0]*F/(R*T))-0.341*Cao)/(exp(2.0*ytemp[0]*F/(R*T))-1.0)*ytemp[4]*ytemp[5]*ytemp[6]*ytemp[7];
    
      d_infinity  = 1.0/(1.0+exp(-(ytemp[0]*1000.0+9.1)/7.0));
      alpha_d     = 0.25+1.4/(1.0+exp((-ytemp[0]*1000.0-35.0)/13.0));
      beta_d      = 1.4/(1.0+exp((ytemp[0]*1000.0+5.0)/5.0));
      gamma_d     = 1.0/(1.0+exp((-ytemp[0]*1000.0+50.0)/20.0));
      tau_d       = (alpha_d*beta_d+gamma_d)*1.0/1000.0;
      ak2[4]    = (d_infinity-ytemp[4])/tau_d;
    
      f1_inf      = 1.0/(1.0+exp((ytemp[0]*1000.0+26.0)/3.0));
      if (f1_inf-ytemp[5] > 0.0)
          constf1 = 1.0+1433.0*(ytemp[2]-50.0*1.0e-6);
      else
          constf1 = 1.0;
    
      tau_f1      = (20.0+1102.5*exp(-pow((float)((ytemp[0]*1000.0+27.0)/15.0),2.0f))+200.0/(1.0+exp((13.0-ytemp[0]*1000.0)/10.0))+180.0/(1.0+exp((30.0+ytemp[0]*1000.0)/10.0)))*constf1/1000.0;
      ak2[5]    = (f1_inf-ytemp[5])/tau_f1;
    
      f2_inf      = 0.33+0.67/(1.0+exp((ytemp[0]*1000.0+32.0)/4.0));
      tau_f2      = (600.0*exp(-pow((float)(ytemp[0]*1000.0+25.0),2.0f)/170.0)+31.0/(1.0+exp((25.0-ytemp[0]*1000.0)/10.0))+16.0/(1.0+exp((30.0+ytemp[0]*1000.0)/10.0)))*constf2/1000.0;
      ak2[6]    = (f2_inf-ytemp[6])/tau_f2;
    
      alpha_fCa   = 1.0/(1.0+pow((float)(ytemp[2]/0.0006),8.0f));
      beta_fCa    = 0.1/(1.0+exp((ytemp[2]-0.0009)/0.0001));
      gamma_fCa   = 0.3/(1.0+exp((ytemp[2]-0.00075)/0.0008));
      fCa_inf     = (alpha_fCa+beta_fCa+gamma_fCa)/1.3156;
    
      if ((ytemp[0] > -0.06) && (fCa_inf > ytemp[7]))
          constfCa = 0.0;
      else
          constfCa = 1.0;
    
      ak2[7]    = constfCa*(fCa_inf-ytemp[7])/tau_fCa;
    
      //// Ito
      i_to        = g_to*(ytemp[0]-E_K)*ytemp[15]*ytemp[16];
    
      q_inf       = 1.0/(1.0+exp((ytemp[0]*1000.0+53.0)/13.0));
      tau_q       = (6.06+39.102/(0.57*exp(-0.08*(ytemp[0]*1000.0+44.0))+0.065*exp(0.1*(ytemp[0]*1000.0+45.93))))/1000.0;
      ak2[15]   = (q_inf-ytemp[15])/tau_q;
    
    
      r_inf       = 1.0/(1.0+exp(-(ytemp[0]*1000.0-22.3)/18.75));
      tau_r       = (2.75352+14.40516/(1.037*exp(0.09*(ytemp[0]*1000.0+30.61))+0.369*exp(-0.12*(ytemp[0]*1000.0+23.84))))/1000.0;
      ak2[16]   = (r_inf-ytemp[16])/tau_r;
    
      //// IKs
      i_Ks        = g_Ks*(ytemp[0]-E_Ks)*pow((float)ytemp[10],2.0f)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/ytemp[2]),1.4f))); // ((time<tDrugApplication)*1+(time >= tDrugApplication)*IKsRedMed)*g_Ks*(ytemp[0]-E_Ks)*pow((float)ytemp[10],2.0)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/ytemp[2]),1.4f)));
    
      Xs_infinity = 1.0/(1.0+exp((-ytemp[0]*1000.0-20.0)/16.0));
      alpha_Xs    = 1100.0/sqrt(1.0+exp((-10.0-ytemp[0]*1000.0)/6.0));
      beta_Xs     = 1.0/(1.0+exp((-60.0+ytemp[0]*1000.0)/20.0));
      tau_Xs      = 1.0*alpha_Xs*beta_Xs/1000.0;
      ak2[10]   = (Xs_infinity-ytemp[10])/tau_Xs;
    
      //// IKr
      i_Kr         = g_Kr*(ytemp[0]-E_K)*ytemp[8]*ytemp[9]*sqrt(Ko/5.4); //((time<tDrugApplication)*1+(time >= tDrugApplication)*IKrRedMed)*g_Kr*(ytemp[0]-E_K)*ytemp[8]*ytemp[9]*sqrt(Ko/5.4);
    
      V_half       = 1000.0*(-R*T/(F*Q)*log(pow((float)(1.0+Cao/2.6),4.0f)/(pow((float)(1.0+Cao/0.58),4.0f)*L0))-0.019);
    
      Xr1_inf      = 1.0/(1.0+exp((V_half-ytemp[0]*1000.0)/4.9));
      alpha_Xr1    = 450.0/(1.0+exp((-45.0-ytemp[0]*1000.0)/10.0));
      beta_Xr1     = 6.0/(1.0+exp((30.0+ytemp[0]*1000.0)/11.5));
      tau_Xr1      = 1.0*alpha_Xr1*beta_Xr1/1000.0;
      ak2[8]     = (Xr1_inf-ytemp[8])/tau_Xr1;
    
      Xr2_infinity = 1.0/(1.0+exp((ytemp[0]*1000.0+88.0)/50.0));
      alpha_Xr2    = 3.0/(1.0+exp((-60.0-ytemp[0]*1000.0)/20.0));
      beta_Xr2     = 1.12/(1.0+exp((-60.0+ytemp[0]*1000.0)/20.0));
      tau_Xr2      = 1.0*alpha_Xr2*beta_Xr2/1000.0;
      ak2[9]    = (Xr2_infinity-ytemp[9])/tau_Xr2;
    
      //// IK1
      alpha_K1    = 3.91/(1.0+exp(0.5942*(ytemp[0]*1000.0-E_K*1000.0-200.0)));
      beta_K1     = (-1.509*exp(0.0002*(ytemp[0]*1000.0-E_K*1000.0+100.0))+exp(0.5886*(ytemp[0]*1000.0-E_K*1000.0-10.0)))/(1.0+exp(0.4547*(ytemp[0]*1000.0-E_K*1000.0)));
      XK1_inf     = alpha_K1/(alpha_K1+beta_K1);
      i_K1        = g_K1*XK1_inf*(ytemp[0]-E_K)*sqrt(Ko/5.4);
    
      //// INaCa
      i_NaCa      = kNaCa*(exp(gamma*ytemp[0]*F/(R*T))*pow((float)ytemp[17],3.0f)*Cao-exp((gamma-1.0)*ytemp[0]*F/(R*T))*pow((float)Nao,3.0f)*ytemp[2]*alpha)/((pow((float)KmNai,3.0f)+pow((float)Nao,3.0f))*(KmCa+Cao)*(1.0+Ksat*exp((gamma-1.0)*ytemp[0]*F/(R*T))));
    
      //// INaK
      i_NaK       = PNaK*Ko/(Ko+Km_K)*ytemp[17]/(ytemp[17]+Km_Na)/(1.0+0.1245*exp(-0.1*ytemp[0]*F/(R*T))+0.0353*exp(-ytemp[0]*F/(R*T)));
    
      //// IpCa
      i_PCa       = g_PCa*ytemp[2]/(ytemp[2]+KPCa);
    
      //// Background currents
      i_b_Na      = g_b_Na*(ytemp[0]-E_Na);
    
      i_b_Ca      = g_b_Ca*(ytemp[0]-E_Ca);
    
      //// Sarcoplasmic reticulum
      i_up        = VmaxUp/(1.0+pow((float)Kup,2.0f)/pow((float)ytemp[2],2.0f));
    
      i_leak      = (ytemp[1]-ytemp[2])*V_leak;
    
      ak2[3]    = 0;
    
    
      RyRSRCass   = (1 - 1/(1 +  exp((ytemp[1]-0.3)/0.1)));
      i_rel       = g_irel_max*RyRSRCass*ytemp[21]*ytemp[22]*(ytemp[1]-ytemp[2]);
    
      RyRainfss   = RyRa1-RyRa2/(1 + exp((1000*ytemp[2]-(RyRahalf))/0.0082));
      ak2[20]   = (RyRainfss- ytemp[20])/RyRtauadapt;
    
      RyRoinfss   = (1 - 1/(1 +  exp((1000*ytemp[2]-(ytemp[20]+ RyRohalf))/0.003)));
      if (RyRoinfss>= ytemp[21])
        RyRtauact = 18.75e-3;       //s
      else
        RyRtauact = 0.1*18.75e-3;   //s
    
      ak2[21]    = (RyRoinfss- ytemp[21])/(RyRtauact);
    
      RyRcinfss   = (1/(1 + exp((1000*ytemp[2]-(ytemp[20]+RyRchalf))/0.001)));
      if (RyRcinfss>= ytemp[22])
        RyRtauinact = 2*87.5e-3;    //s
      else
        RyRtauinact = 87.5e-3;      //s
    
      ak2[22]    = (RyRcinfss- ytemp[22])/(RyRtauinact);
    
    
    
    
      //// Ca2+ buffering
      Cai_bufc    = 1.0/(1.0+Buf_C*Kbuf_C/pow((float)(ytemp[2]+Kbuf_C), 2.0f));
      Ca_SR_bufSR = 1.0/(1.0+Buf_SR*Kbuf_SR/pow((float)(ytemp[1]+Kbuf_SR), 2.0f));
    
      //// Ionic concentrations
      //Nai
      ak2[17]   = -Cm*(i_Na+i_NaL+i_b_Na+3.0*i_NaK+3.0*i_NaCa+i_fNa)/(F*Vc*1.0e-18);
      //Cai
      ak2[2]    = Cai_bufc*(i_leak-i_up+i_rel-(i_CaL+i_b_Ca+i_PCa-2.0*i_NaCa)*Cm/(2.0*Vc*F*1.0e-18));
       //caSR
      ak2[1]    = Ca_SR_bufSR*Vc/V_SR*(i_up-(i_rel+i_leak));
    
      //// Stimulation
    //  i_stim_Amplitude 		= 5.5e-10;//7.5e-10;   // ampere (in stim_mode)
    //  i_stim_End 				= 1000.0;   // second (in stim_mode)
    //  i_stim_PulseDuration	= 0.005;   // second (in stim_mode)
    //  i_stim_Start 			= 0.0;   // second (in stim_mode)
    //  i_stim_frequency        = 60.0;   // per_second (in stim_mode)
      //stim_flag 				= stimFlag;   // dimensionless (in stim_mode)
    //  i_stim_Period 			= 60.0/i_stim_frequency;
    
      //if stim_flag~=0 && stim_flag~=1
      //error('Paci2020: wrong pacing! stimFlag can be only 0 (spontaneous) or 1 (paced)');
      //end
    
      /*
      if ((time >= i_stim_Start) && (time <= i_stim_End) && (time-i_stim_Start-floor((time-i_stim_Start)/i_stim_Period)*i_stim_Period <= i_stim_PulseDuration))
          i_stim = stim_flag*i_stim_Amplitude/Cm;
      else
          i_stim = 0.0;
      */
    
      //// Membrane potential
      ak2[0] = -(i_K1+i_to+i_Kr+i_Ks+i_CaL+i_NaK+i_Na+i_NaL+i_NaCa+i_PCa+i_f+i_b_Na+i_b_Ca-i_stim);


    //-----THIRD STEP ------------------------------------------------------------------------------------------------------------------------------------------------------------------  
    for (i=0;i<layers;i++)
      ytemp[i]=y[i]+dt*(b31*dydt[i]+b32*ak2[i]);

      E_Na = R*T/F*log(Nao/ytemp[17]);
      E_Ca = 0.5*R*T/F*log(Cao/ytemp[2]);
      E_K  = R*T/F*log(Ko/Ki);
      E_Ks = R*T/F*log((Ko+PkNa*Nao)/(Ki+PkNa*ytemp[17]));
    
      //// INa adapted from DOI:10.3389/fphys.2018.00080
      i_Na        =  g_Na*pow((float)ytemp[13],3.0f)*ytemp[11]*ytemp[12]*(ytemp[0] - E_Na);
    
      m_inf       = 1 / (1 + exp((ytemp[0]*1000 + 39)/-11.2));
      tau_m       = (0.00001 + 0.00013*exp(-pow((float)((ytemp[0]*1000 + 48)/15),2.0f)) + 0.000045 / (1 + exp((ytemp[0]*1000 + 42)/-5)));
      ak3[13]   = (m_inf-ytemp[13])/tau_m;
    
      h_inf       = 1 / (1 + exp((ytemp[0]*1000 + 66.5)/6.8));
      tau_h       = (0.00007 + 0.034 / (1 + exp((ytemp[0]*1000 + 41)/5.5) + exp(-(ytemp[0]*1000 + 41)/14)) + 0.0002 / (1 + exp(-(ytemp[0]*1000 + 79)/14)));
      ak3[11]   = (h_inf-ytemp[11])/tau_h;
    
      j_inf       = h_inf;
      tau_j       = 10*(0.0007 + 0.15 / (1 + exp((ytemp[0]*1000 + 41)/5.5) + exp(-(ytemp[0]*1000 + 41)/14)) + 0.002 / (1 + exp(-(ytemp[0]*1000 + 79)/14)));
      ak3[12]   = (j_inf-ytemp[12])/tau_j;
    
    
      //// INaL
      i_NaL       = GNaLmax* pow((float)ytemp[18],3.0f)*ytemp[19]*(ytemp[0]-E_Na);
    
      m_inf_L     = 1/(1+exp(-(ytemp[0]*1000+42.85)/(5.264)));
      alpha_m_L   = 1/(1+exp((-60-ytemp[0]*1000)/5));
      beta_m_L    = 0.1/(1+exp((ytemp[0]*1000+35)/5))+0.1/(1+exp((ytemp[0]*1000-50)/200));
      tau_m_L     = 1 * myCoefTauM*alpha_m_L*beta_m_L;
      ak3[18]   = (m_inf_L-ytemp[18])/tau_m_L*1000;
    
      h_inf_L     = 1/(1+exp((ytemp[0]*1000+Vh_hLate)/(7.488)));
      ak3[19]   = (h_inf_L-ytemp[19])/tau_h_L*1000;
    
      //// If adapted from DOI:10.3389/fphys.2018.00080
      i_fK        = fK*g_f*ytemp[14]*(ytemp[0] - E_K);
      i_fNa       = fNa*g_f*ytemp[14]*(ytemp[0] - E_Na);
      i_f         = i_fK + i_fNa;
    
      Xf_infinity = 1.0/(1.0 + exp((ytemp[0]*1000 + 69)/8));
      tau_Xf      = 5600 / (1 + exp((ytemp[0]*1000 + 65)/7) + exp(-(ytemp[0]*1000 + 65)/19));
      ak3[14]   = 1000*(Xf_infinity-ytemp[14])/tau_Xf;
      
    
      //// ICaL  
      //Prevent division by 0
      if(ytemp[0]< precision && ytemp[0] > -precision) //hopital
        i_CaL =  g_CaL*4.0*ytemp[0]*pow((float)F,2.0f)/(R*T) *ytemp[4]*ytemp[5]*ytemp[6]*ytemp[7] / (2.0*F/(R*T)) * (ytemp[2] - 0.341*Cao);
      else
        i_CaL = g_CaL*4.0*ytemp[0]*pow((float)F,2.0f)/(R*T)*(ytemp[2]*exp(2.0*ytemp[0]*F/(R*T))-0.341*Cao)/(exp(2.0*ytemp[0]*F/(R*T))-1.0)*ytemp[4]*ytemp[5]*ytemp[6]*ytemp[7]; //((time<tDrugApplication)*1+(time >= tDrugApplication)*ICaLRedMed)*g_CaL*4.0*ytemp[0]*pow(F,2.0)/(R*T)*(ytemp[2]*exp(2.0*ytemp[0]*F/(R*T))-0.341*Cao)/(exp(2.0*ytemp[0]*F/(R*T))-1.0)*ytemp[4]*ytemp[5]*ytemp[6]*ytemp[7];
    
      d_infinity  = 1.0/(1.0+exp(-(ytemp[0]*1000.0+9.1)/7.0));
      alpha_d     = 0.25+1.4/(1.0+exp((-ytemp[0]*1000.0-35.0)/13.0));
      beta_d      = 1.4/(1.0+exp((ytemp[0]*1000.0+5.0)/5.0));
      gamma_d     = 1.0/(1.0+exp((-ytemp[0]*1000.0+50.0)/20.0));
      tau_d       = (alpha_d*beta_d+gamma_d)*1.0/1000.0;
      ak3[4]    = (d_infinity-ytemp[4])/tau_d;
    
      f1_inf      = 1.0/(1.0+exp((ytemp[0]*1000.0+26.0)/3.0));
      if (f1_inf-ytemp[5] > 0.0)
          constf1 = 1.0+1433.0*(ytemp[2]-50.0*1.0e-6);
      else
          constf1 = 1.0;
    
      tau_f1      = (20.0+1102.5*exp(-pow((float)((ytemp[0]*1000.0+27.0)/15.0),2.0f))+200.0/(1.0+exp((13.0-ytemp[0]*1000.0)/10.0))+180.0/(1.0+exp((30.0+ytemp[0]*1000.0)/10.0)))*constf1/1000.0;
      ak3[5]    = (f1_inf-ytemp[5])/tau_f1;
    
      f2_inf      = 0.33+0.67/(1.0+exp((ytemp[0]*1000.0+32.0)/4.0));
      tau_f2      = (600.0*exp(-pow((float)(ytemp[0]*1000.0+25.0),2.0f)/170.0)+31.0/(1.0+exp((25.0-ytemp[0]*1000.0)/10.0))+16.0/(1.0+exp((30.0+ytemp[0]*1000.0)/10.0)))*constf2/1000.0;
      ak3[6]    = (f2_inf-ytemp[6])/tau_f2;
    
      alpha_fCa   = 1.0/(1.0+pow((float)(ytemp[2]/0.0006),8.0f));
      beta_fCa    = 0.1/(1.0+exp((ytemp[2]-0.0009)/0.0001));
      gamma_fCa   = 0.3/(1.0+exp((ytemp[2]-0.00075)/0.0008));
      fCa_inf     = (alpha_fCa+beta_fCa+gamma_fCa)/1.3156;
    
      if ((ytemp[0] > -0.06) && (fCa_inf > ytemp[7]))
          constfCa = 0.0;
      else
          constfCa = 1.0;

      ak3[7]    = constfCa*(fCa_inf-ytemp[7])/tau_fCa;
    
      //// Ito
      i_to        = g_to*(ytemp[0]-E_K)*ytemp[15]*ytemp[16];
    
      q_inf       = 1.0/(1.0+exp((ytemp[0]*1000.0+53.0)/13.0));
      tau_q       = (6.06+39.102/(0.57*exp(-0.08*(ytemp[0]*1000.0+44.0))+0.065*exp(0.1*(ytemp[0]*1000.0+45.93))))/1000.0;
      ak3[15]   = (q_inf-ytemp[15])/tau_q;
    
    
      r_inf       = 1.0/(1.0+exp(-(ytemp[0]*1000.0-22.3)/18.75));
      tau_r       = (2.75352+14.40516/(1.037*exp(0.09*(ytemp[0]*1000.0+30.61))+0.369*exp(-0.12*(ytemp[0]*1000.0+23.84))))/1000.0;
      ak3[16]   = (r_inf-ytemp[16])/tau_r;
    
      //// IKs
      i_Ks        = g_Ks*(ytemp[0]-E_Ks)*pow((float)ytemp[10],2.0f)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/ytemp[2]),1.4f))); // ((time<tDrugApplication)*1+(time >= tDrugApplication)*IKsRedMed)*g_Ks*(ytemp[0]-E_Ks)*pow((float)ytemp[10],2.0)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/ytemp[2]),1.4f)));
    
      Xs_infinity = 1.0/(1.0+exp((-ytemp[0]*1000.0-20.0)/16.0));
      alpha_Xs    = 1100.0/sqrt(1.0+exp((-10.0-ytemp[0]*1000.0)/6.0));
      beta_Xs     = 1.0/(1.0+exp((-60.0+ytemp[0]*1000.0)/20.0));
      tau_Xs      = 1.0*alpha_Xs*beta_Xs/1000.0;
      ak3[10]   = (Xs_infinity-ytemp[10])/tau_Xs;
    
      //// IKr
      i_Kr         = g_Kr*(ytemp[0]-E_K)*ytemp[8]*ytemp[9]*sqrt(Ko/5.4); //((time<tDrugApplication)*1+(time >= tDrugApplication)*IKrRedMed)*g_Kr*(ytemp[0]-E_K)*ytemp[8]*ytemp[9]*sqrt(Ko/5.4);
    
      V_half       = 1000.0*(-R*T/(F*Q)*log(pow((float)(1.0+Cao/2.6),4.0f)/(pow((float)(1.0+Cao/0.58),4.0f)*L0))-0.019);
    
      Xr1_inf      = 1.0/(1.0+exp((V_half-ytemp[0]*1000.0)/4.9));
      alpha_Xr1    = 450.0/(1.0+exp((-45.0-ytemp[0]*1000.0)/10.0));
      beta_Xr1     = 6.0/(1.0+exp((30.0+ytemp[0]*1000.0)/11.5));
      tau_Xr1      = 1.0*alpha_Xr1*beta_Xr1/1000.0;
      ak3[8]     = (Xr1_inf-ytemp[8])/tau_Xr1;
    
      Xr2_infinity = 1.0/(1.0+exp((ytemp[0]*1000.0+88.0)/50.0));
      alpha_Xr2    = 3.0/(1.0+exp((-60.0-ytemp[0]*1000.0)/20.0));
      beta_Xr2     = 1.12/(1.0+exp((-60.0+ytemp[0]*1000.0)/20.0));
      tau_Xr2      = 1.0*alpha_Xr2*beta_Xr2/1000.0;
      ak3[9]    = (Xr2_infinity-ytemp[9])/tau_Xr2;
    
      //// IK1
      alpha_K1    = 3.91/(1.0+exp(0.5942*(ytemp[0]*1000.0-E_K*1000.0-200.0)));
      beta_K1     = (-1.509*exp(0.0002*(ytemp[0]*1000.0-E_K*1000.0+100.0))+exp(0.5886*(ytemp[0]*1000.0-E_K*1000.0-10.0)))/(1.0+exp(0.4547*(ytemp[0]*1000.0-E_K*1000.0)));
      XK1_inf     = alpha_K1/(alpha_K1+beta_K1);
      i_K1        = g_K1*XK1_inf*(ytemp[0]-E_K)*sqrt(Ko/5.4);
    
      //// INaCa
      i_NaCa      = kNaCa*(exp(gamma*ytemp[0]*F/(R*T))*pow((float)ytemp[17],3.0f)*Cao-exp((gamma-1.0)*ytemp[0]*F/(R*T))*pow((float)Nao,3.0f)*ytemp[2]*alpha)/((pow((float)KmNai,3.0f)+pow((float)Nao,3.0f))*(KmCa+Cao)*(1.0+Ksat*exp((gamma-1.0)*ytemp[0]*F/(R*T))));
    
      //// INaK
      i_NaK       = PNaK*Ko/(Ko+Km_K)*ytemp[17]/(ytemp[17]+Km_Na)/(1.0+0.1245*exp(-0.1*ytemp[0]*F/(R*T))+0.0353*exp(-ytemp[0]*F/(R*T)));
    
      //// IpCa
      i_PCa       = g_PCa*ytemp[2]/(ytemp[2]+KPCa);
    
      //// Background currents
      i_b_Na      = g_b_Na*(ytemp[0]-E_Na);
    
      i_b_Ca      = g_b_Ca*(ytemp[0]-E_Ca);
    
      //// Sarcoplasmic reticulum
      i_up        = VmaxUp/(1.0+pow((float)Kup,2.0f)/pow((float)ytemp[2],2.0f));
    
      i_leak      = (ytemp[1]-ytemp[2])*V_leak;
    
      ak3[3]    = 0;
  
    
      RyRSRCass   = (1 - 1/(1 +  exp((ytemp[1]-0.3)/0.1)));
      i_rel       = g_irel_max*RyRSRCass*ytemp[21]*ytemp[22]*(ytemp[1]-ytemp[2]);
    
      RyRainfss   = RyRa1-RyRa2/(1 + exp((1000*ytemp[2]-(RyRahalf))/0.0082));
      ak3[20]   = (RyRainfss- ytemp[20])/RyRtauadapt;
    
      RyRoinfss   = (1 - 1/(1 +  exp((1000*ytemp[2]-(ytemp[20]+ RyRohalf))/0.003)));
      if (RyRoinfss>= ytemp[21])
        RyRtauact = 18.75e-3;       //s
      else
        RyRtauact = 0.1*18.75e-3;   //s
    
      ak3[21]    = (RyRoinfss- ytemp[21])/(RyRtauact);
    
      RyRcinfss   = (1/(1 + exp((1000*ytemp[2]-(ytemp[20]+RyRchalf))/0.001)));
      if (RyRcinfss>= ytemp[22])
        RyRtauinact = 2*87.5e-3;    //s
      else
        RyRtauinact = 87.5e-3;      //s
    
      ak3[22]    = (RyRcinfss- ytemp[22])/(RyRtauinact);
    
    
    
    
      //// Ca2+ buffering
      Cai_bufc    = 1.0/(1.0+Buf_C*Kbuf_C/pow((float)(ytemp[2]+Kbuf_C), 2.0f));
      Ca_SR_bufSR = 1.0/(1.0+Buf_SR*Kbuf_SR/pow((float)(ytemp[1]+Kbuf_SR), 2.0f));
    
      //// Ionic concentrations
      //Nai
      ak3[17]   = -Cm*(i_Na+i_NaL+i_b_Na+3.0*i_NaK+3.0*i_NaCa+i_fNa)/(F*Vc*1.0e-18);
      //Cai
      ak3[2]    = Cai_bufc*(i_leak-i_up+i_rel-(i_CaL+i_b_Ca+i_PCa-2.0*i_NaCa)*Cm/(2.0*Vc*F*1.0e-18));
       //caSR
      ak3[1]    = Ca_SR_bufSR*Vc/V_SR*(i_up-(i_rel+i_leak));
    
      //// Stimulation
    //  i_stim_Amplitude 		= 5.5e-10;//7.5e-10;   // ampere (in stim_mode)
    //  i_stim_End 				= 1000.0;   // second (in stim_mode)
    //  i_stim_PulseDuration	= 0.005;   // second (in stim_mode)
    //  i_stim_Start 			= 0.0;   // second (in stim_mode)
    //  i_stim_frequency        = 60.0;   // per_second (in stim_mode)
      //stim_flag 				= stimFlag;   // dimensionless (in stim_mode)
    //  i_stim_Period 			= 60.0/i_stim_frequency;
    
      //if stim_flag~=0 && stim_flag~=1
      //error('Paci2020: wrong pacing! stimFlag can be only 0 (spontaneous) or 1 (paced)');
      //end
    
      /*
      if ((time >= i_stim_Start) && (time <= i_stim_End) && (time-i_stim_Start-floor((time-i_stim_Start)/i_stim_Period)*i_stim_Period <= i_stim_PulseDuration))
          i_stim = stim_flag*i_stim_Amplitude/Cm;
      else
          i_stim = 0.0;
      */
    
      //// Membrane potential
      ak3[0] = -(i_K1+i_to+i_Kr+i_Ks+i_CaL+i_NaK+i_Na+i_NaL+i_NaCa+i_PCa+i_f+i_b_Na+i_b_Ca-i_stim);


    //-----FOURTH STEP ------------------------------------------------------------------------------------------------------------------------------------------------------------------  
    for (i=0;i<layers;i++)
      ytemp[i]=y[i]+dt*(b41*dydt[i]+b42*ak2[i]+b43*ak3[i]);
    
      E_Na = R*T/F*log(Nao/ytemp[17]);
      E_Ca = 0.5*R*T/F*log(Cao/ytemp[2]);
      E_K  = R*T/F*log(Ko/Ki);
      E_Ks = R*T/F*log((Ko+PkNa*Nao)/(Ki+PkNa*ytemp[17]));
    
      //// INa adapted from DOI:10.3389/fphys.2018.00080
      i_Na        =  g_Na*pow((float)ytemp[13],3.0f)*ytemp[11]*ytemp[12]*(ytemp[0] - E_Na);
    
      m_inf       = 1 / (1 + exp((ytemp[0]*1000 + 39)/-11.2));
      tau_m       = (0.00001 + 0.00013*exp(-pow((float)((ytemp[0]*1000 + 48)/15),2.0f)) + 0.000045 / (1 + exp((ytemp[0]*1000 + 42)/-5)));
      ak4[13]   = (m_inf-ytemp[13])/tau_m;
    
      h_inf       = 1 / (1 + exp((ytemp[0]*1000 + 66.5)/6.8));
      tau_h       = (0.00007 + 0.034 / (1 + exp((ytemp[0]*1000 + 41)/5.5) + exp(-(ytemp[0]*1000 + 41)/14)) + 0.0002 / (1 + exp(-(ytemp[0]*1000 + 79)/14)));
      ak4[11]   = (h_inf-ytemp[11])/tau_h;
    
      j_inf       = h_inf;
      tau_j       = 10*(0.0007 + 0.15 / (1 + exp((ytemp[0]*1000 + 41)/5.5) + exp(-(ytemp[0]*1000 + 41)/14)) + 0.002 / (1 + exp(-(ytemp[0]*1000 + 79)/14)));
      ak4[12]   = (j_inf-ytemp[12])/tau_j;
    
    
      //// INaL
      i_NaL       = GNaLmax* pow((float)ytemp[18],3.0f)*ytemp[19]*(ytemp[0]-E_Na);
    
      m_inf_L     = 1/(1+exp(-(ytemp[0]*1000+42.85)/(5.264)));
      alpha_m_L   = 1/(1+exp((-60-ytemp[0]*1000)/5));
      beta_m_L    = 0.1/(1+exp((ytemp[0]*1000+35)/5))+0.1/(1+exp((ytemp[0]*1000-50)/200));
      tau_m_L     = 1 * myCoefTauM*alpha_m_L*beta_m_L;
      ak4[18]   = (m_inf_L-ytemp[18])/tau_m_L*1000;
    
      h_inf_L     = 1/(1+exp((ytemp[0]*1000+Vh_hLate)/(7.488)));
      ak4[19]   = (h_inf_L-ytemp[19])/tau_h_L*1000;
    
      //// If adapted from DOI:10.3389/fphys.2018.00080
      i_fK        = fK*g_f*ytemp[14]*(ytemp[0] - E_K);
      i_fNa       = fNa*g_f*ytemp[14]*(ytemp[0] - E_Na);
      i_f         = i_fK + i_fNa;
    
      Xf_infinity = 1.0/(1.0 + exp((ytemp[0]*1000 + 69)/8));
      tau_Xf      = 5600 / (1 + exp((ytemp[0]*1000 + 65)/7) + exp(-(ytemp[0]*1000 + 65)/19));
      ak4[14]   = 1000*(Xf_infinity-ytemp[14])/tau_Xf;
      
    
      //// ICaL
      //Prevent division by 0
      if(ytemp[0]< precision && ytemp[0] > -precision) //hopital
        i_CaL =  g_CaL*4.0*ytemp[0]*pow((float)F,2.0f)/(R*T) *ytemp[4]*ytemp[5]*ytemp[6]*ytemp[7] / (2.0*F/(R*T)) * (ytemp[2] - 0.341*Cao);
      else
        i_CaL = g_CaL*4.0*ytemp[0]*pow((float)F,2.0f)/(R*T)*(ytemp[2]*exp(2.0*ytemp[0]*F/(R*T))-0.341*Cao)/(exp(2.0*ytemp[0]*F/(R*T))-1.0)*ytemp[4]*ytemp[5]*ytemp[6]*ytemp[7]; //((time<tDrugApplication)*1+(time >= tDrugApplication)*ICaLRedMed)*g_CaL*4.0*ytemp[0]*pow(F,2.0)/(R*T)*(ytemp[2]*exp(2.0*ytemp[0]*F/(R*T))-0.341*Cao)/(exp(2.0*ytemp[0]*F/(R*T))-1.0)*ytemp[4]*ytemp[5]*ytemp[6]*ytemp[7];
    
      d_infinity  = 1.0/(1.0+exp(-(ytemp[0]*1000.0+9.1)/7.0));
      alpha_d     = 0.25+1.4/(1.0+exp((-ytemp[0]*1000.0-35.0)/13.0));
      beta_d      = 1.4/(1.0+exp((ytemp[0]*1000.0+5.0)/5.0));
      gamma_d     = 1.0/(1.0+exp((-ytemp[0]*1000.0+50.0)/20.0));
      tau_d       = (alpha_d*beta_d+gamma_d)*1.0/1000.0;
      ak4[4]    = (d_infinity-ytemp[4])/tau_d;
    
      f1_inf      = 1.0/(1.0+exp((ytemp[0]*1000.0+26.0)/3.0));
      if (f1_inf-ytemp[5] > 0.0)
          constf1 = 1.0+1433.0*(ytemp[2]-50.0*1.0e-6);
      else
          constf1 = 1.0;
    
      tau_f1      = (20.0+1102.5*exp(-pow((float)((ytemp[0]*1000.0+27.0)/15.0),2.0f))+200.0/(1.0+exp((13.0-ytemp[0]*1000.0)/10.0))+180.0/(1.0+exp((30.0+ytemp[0]*1000.0)/10.0)))*constf1/1000.0;
      ak4[5]    = (f1_inf-ytemp[5])/tau_f1;
    
      f2_inf      = 0.33+0.67/(1.0+exp((ytemp[0]*1000.0+32.0)/4.0));
      tau_f2      = (600.0*exp(-pow((float)(ytemp[0]*1000.0+25.0),2.0f)/170.0)+31.0/(1.0+exp((25.0-ytemp[0]*1000.0)/10.0))+16.0/(1.0+exp((30.0+ytemp[0]*1000.0)/10.0)))*constf2/1000.0;
      ak4[6]    = (f2_inf-ytemp[6])/tau_f2;
    
      alpha_fCa   = 1.0/(1.0+pow((float)(ytemp[2]/0.0006),8.0f));
      beta_fCa    = 0.1/(1.0+exp((ytemp[2]-0.0009)/0.0001));
      gamma_fCa   = 0.3/(1.0+exp((ytemp[2]-0.00075)/0.0008));
      fCa_inf     = (alpha_fCa+beta_fCa+gamma_fCa)/1.3156;
    
      if ((ytemp[0] > -0.06) && (fCa_inf > ytemp[7]))
          constfCa = 0.0;
      else
          constfCa = 1.0;
    
      ak4[7]    = constfCa*(fCa_inf-ytemp[7])/tau_fCa;
    
      //// Ito
      i_to        = g_to*(ytemp[0]-E_K)*ytemp[15]*ytemp[16];
    
      q_inf       = 1.0/(1.0+exp((ytemp[0]*1000.0+53.0)/13.0));
      tau_q       = (6.06+39.102/(0.57*exp(-0.08*(ytemp[0]*1000.0+44.0))+0.065*exp(0.1*(ytemp[0]*1000.0+45.93))))/1000.0;
      ak4[15]   = (q_inf-ytemp[15])/tau_q;
    
    
      r_inf       = 1.0/(1.0+exp(-(ytemp[0]*1000.0-22.3)/18.75));
      tau_r       = (2.75352+14.40516/(1.037*exp(0.09*(ytemp[0]*1000.0+30.61))+0.369*exp(-0.12*(ytemp[0]*1000.0+23.84))))/1000.0;
      ak4[16]   = (r_inf-ytemp[16])/tau_r;
    
      //// IKs
      i_Ks        = g_Ks*(ytemp[0]-E_Ks)*pow((float)ytemp[10],2.0f)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/ytemp[2]),1.4f))); // ((time<tDrugApplication)*1+(time >= tDrugApplication)*IKsRedMed)*g_Ks*(ytemp[0]-E_Ks)*pow((float)ytemp[10],2.0)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/ytemp[2]),1.4f)));
    
      Xs_infinity = 1.0/(1.0+exp((-ytemp[0]*1000.0-20.0)/16.0));
      alpha_Xs    = 1100.0/sqrt(1.0+exp((-10.0-ytemp[0]*1000.0)/6.0));
      beta_Xs     = 1.0/(1.0+exp((-60.0+ytemp[0]*1000.0)/20.0));
      tau_Xs      = 1.0*alpha_Xs*beta_Xs/1000.0;
      ak4[10]   = (Xs_infinity-ytemp[10])/tau_Xs;
    
      //// IKr
      i_Kr         = g_Kr*(ytemp[0]-E_K)*ytemp[8]*ytemp[9]*sqrt(Ko/5.4); //((time<tDrugApplication)*1+(time >= tDrugApplication)*IKrRedMed)*g_Kr*(ytemp[0]-E_K)*ytemp[8]*ytemp[9]*sqrt(Ko/5.4);
    
      V_half       = 1000.0*(-R*T/(F*Q)*log(pow((float)(1.0+Cao/2.6),4.0f)/(pow((float)(1.0+Cao/0.58),4.0f)*L0))-0.019);
    
      Xr1_inf      = 1.0/(1.0+exp((V_half-ytemp[0]*1000.0)/4.9));
      alpha_Xr1    = 450.0/(1.0+exp((-45.0-ytemp[0]*1000.0)/10.0));
      beta_Xr1     = 6.0/(1.0+exp((30.0+ytemp[0]*1000.0)/11.5));
      tau_Xr1      = 1.0*alpha_Xr1*beta_Xr1/1000.0;
      ak4[8]     = (Xr1_inf-ytemp[8])/tau_Xr1;
    
      Xr2_infinity = 1.0/(1.0+exp((ytemp[0]*1000.0+88.0)/50.0));
      alpha_Xr2    = 3.0/(1.0+exp((-60.0-ytemp[0]*1000.0)/20.0));
      beta_Xr2     = 1.12/(1.0+exp((-60.0+ytemp[0]*1000.0)/20.0));
      tau_Xr2      = 1.0*alpha_Xr2*beta_Xr2/1000.0;
      ak4[9]    = (Xr2_infinity-ytemp[9])/tau_Xr2;
    
      //// IK1
      alpha_K1    = 3.91/(1.0+exp(0.5942*(ytemp[0]*1000.0-E_K*1000.0-200.0)));
      beta_K1     = (-1.509*exp(0.0002*(ytemp[0]*1000.0-E_K*1000.0+100.0))+exp(0.5886*(ytemp[0]*1000.0-E_K*1000.0-10.0)))/(1.0+exp(0.4547*(ytemp[0]*1000.0-E_K*1000.0)));
      XK1_inf     = alpha_K1/(alpha_K1+beta_K1);
      i_K1        = g_K1*XK1_inf*(ytemp[0]-E_K)*sqrt(Ko/5.4);
    
      //// INaCa
      i_NaCa      = kNaCa*(exp(gamma*ytemp[0]*F/(R*T))*pow((float)ytemp[17],3.0f)*Cao-exp((gamma-1.0)*ytemp[0]*F/(R*T))*pow((float)Nao,3.0f)*ytemp[2]*alpha)/((pow((float)KmNai,3.0f)+pow((float)Nao,3.0f))*(KmCa+Cao)*(1.0+Ksat*exp((gamma-1.0)*ytemp[0]*F/(R*T))));
    
      //// INaK
      i_NaK       = PNaK*Ko/(Ko+Km_K)*ytemp[17]/(ytemp[17]+Km_Na)/(1.0+0.1245*exp(-0.1*ytemp[0]*F/(R*T))+0.0353*exp(-ytemp[0]*F/(R*T)));
    
      //// IpCa
      i_PCa       = g_PCa*ytemp[2]/(ytemp[2]+KPCa);
    
      //// Background currents
      i_b_Na      = g_b_Na*(ytemp[0]-E_Na);
    
      i_b_Ca      = g_b_Ca*(ytemp[0]-E_Ca);
    
      //// Sarcoplasmic reticulum
      i_up        = VmaxUp/(1.0+pow((float)Kup,2.0f)/pow((float)ytemp[2],2.0f));
    
      i_leak      = (ytemp[1]-ytemp[2])*V_leak;
    
      ak4[3]    = 0;
    
    
      RyRSRCass   = (1 - 1/(1 +  exp((ytemp[1]-0.3)/0.1)));
      i_rel       = g_irel_max*RyRSRCass*ytemp[21]*ytemp[22]*(ytemp[1]-ytemp[2]);
    
      RyRainfss   = RyRa1-RyRa2/(1 + exp((1000*ytemp[2]-(RyRahalf))/0.0082));
      ak4[20]   = (RyRainfss- ytemp[20])/RyRtauadapt;
    
      RyRoinfss   = (1 - 1/(1 +  exp((1000*ytemp[2]-(ytemp[20]+ RyRohalf))/0.003)));
      if (RyRoinfss>= ytemp[21])
        RyRtauact = 18.75e-3;       //s
      else
        RyRtauact = 0.1*18.75e-3;   //s
    
      ak4[21]    = (RyRoinfss- ytemp[21])/(RyRtauact);
    
      RyRcinfss   = (1/(1 + exp((1000*ytemp[2]-(ytemp[20]+RyRchalf))/0.001)));
      if (RyRcinfss>= ytemp[22])
        RyRtauinact = 2*87.5e-3;    //s
      else
        RyRtauinact = 87.5e-3;      //s
    
      ak4[22]    = (RyRcinfss- ytemp[22])/(RyRtauinact);
    
    
    
    
      //// Ca2+ buffering
      Cai_bufc    = 1.0/(1.0+Buf_C*Kbuf_C/pow((float)(ytemp[2]+Kbuf_C), 2.0f));
      Ca_SR_bufSR = 1.0/(1.0+Buf_SR*Kbuf_SR/pow((float)(ytemp[1]+Kbuf_SR), 2.0f));
    
      //// Ionic concentrations
      //Nai
      ak4[17]   = -Cm*(i_Na+i_NaL+i_b_Na+3.0*i_NaK+3.0*i_NaCa+i_fNa)/(F*Vc*1.0e-18);
      //Cai
      ak4[2]    = Cai_bufc*(i_leak-i_up+i_rel-(i_CaL+i_b_Ca+i_PCa-2.0*i_NaCa)*Cm/(2.0*Vc*F*1.0e-18));
       //caSR
      ak4[1]    = Ca_SR_bufSR*Vc/V_SR*(i_up-(i_rel+i_leak));
    
      //// Stimulation
    //  i_stim_Amplitude 		= 5.5e-10;//7.5e-10;   // ampere (in stim_mode)
    //  i_stim_End 				= 1000.0;   // second (in stim_mode)
    //  i_stim_PulseDuration	= 0.005;   // second (in stim_mode)
    //  i_stim_Start 			= 0.0;   // second (in stim_mode)
    //  i_stim_frequency        = 60.0;   // per_second (in stim_mode)
      //stim_flag 				= stimFlag;   // dimensionless (in stim_mode)
    //  i_stim_Period 			= 60.0/i_stim_frequency;
    
      //if stim_flag~=0 && stim_flag~=1
      //error('Paci2020: wrong pacing! stimFlag can be only 0 (spontaneous) or 1 (paced)');
      //end
    
      /*
      if ((time >= i_stim_Start) && (time <= i_stim_End) && (time-i_stim_Start-floor((time-i_stim_Start)/i_stim_Period)*i_stim_Period <= i_stim_PulseDuration))
          i_stim = stim_flag*i_stim_Amplitude/Cm;
      else
          i_stim = 0.0;
      */
    
      //// Membrane potential
      ak4[0] = -(i_K1+i_to+i_Kr+i_Ks+i_CaL+i_NaK+i_Na+i_NaL+i_NaCa+i_PCa+i_f+i_b_Na+i_b_Ca-i_stim);
    //-----FIFTH STEP ------------------------------------------------------------------------------------------------------------------------------------------------------------------  
    for (i=0;i<layers;i++)
      ytemp[i]=y[i]+dt*(b51*dydt[i]+b52*ak2[i]+b53*ak3[i]+b54*ak4[i]);

      E_Na = R*T/F*log(Nao/ytemp[17]);
      E_Ca = 0.5*R*T/F*log(Cao/ytemp[2]);
      E_K  = R*T/F*log(Ko/Ki);
      E_Ks = R*T/F*log((Ko+PkNa*Nao)/(Ki+PkNa*ytemp[17]));
    
      //// INa adapted from DOI:10.3389/fphys.2018.00080
      i_Na        =  g_Na*pow((float)ytemp[13],3.0f)*ytemp[11]*ytemp[12]*(ytemp[0] - E_Na);
    
      m_inf       = 1 / (1 + exp((ytemp[0]*1000 + 39)/-11.2));
      tau_m       = (0.00001 + 0.00013*exp(-pow((float)((ytemp[0]*1000 + 48)/15),2.0f)) + 0.000045 / (1 + exp((ytemp[0]*1000 + 42)/-5)));
      ak5[13]   = (m_inf-ytemp[13])/tau_m;
    
      h_inf       = 1 / (1 + exp((ytemp[0]*1000 + 66.5)/6.8));
      tau_h       = (0.00007 + 0.034 / (1 + exp((ytemp[0]*1000 + 41)/5.5) + exp(-(ytemp[0]*1000 + 41)/14)) + 0.0002 / (1 + exp(-(ytemp[0]*1000 + 79)/14)));
      ak5[11]   = (h_inf-ytemp[11])/tau_h;
    
      j_inf       = h_inf;
      tau_j       = 10*(0.0007 + 0.15 / (1 + exp((ytemp[0]*1000 + 41)/5.5) + exp(-(ytemp[0]*1000 + 41)/14)) + 0.002 / (1 + exp(-(ytemp[0]*1000 + 79)/14)));
      ak5[12]   = (j_inf-ytemp[12])/tau_j;
    
    
      //// INaL
      i_NaL       = GNaLmax* pow((float)ytemp[18],3.0f)*ytemp[19]*(ytemp[0]-E_Na);
    
      m_inf_L     = 1/(1+exp(-(ytemp[0]*1000+42.85)/(5.264)));
      alpha_m_L   = 1/(1+exp((-60-ytemp[0]*1000)/5));
      beta_m_L    = 0.1/(1+exp((ytemp[0]*1000+35)/5))+0.1/(1+exp((ytemp[0]*1000-50)/200));
      tau_m_L     = 1 * myCoefTauM*alpha_m_L*beta_m_L;
      ak5[18]   = (m_inf_L-ytemp[18])/tau_m_L*1000;
    
      h_inf_L     = 1/(1+exp((ytemp[0]*1000+Vh_hLate)/(7.488)));
      ak5[19]   = (h_inf_L-ytemp[19])/tau_h_L*1000;
    
      //// If adapted from DOI:10.3389/fphys.2018.00080
      i_fK        = fK*g_f*ytemp[14]*(ytemp[0] - E_K);
      i_fNa       = fNa*g_f*ytemp[14]*(ytemp[0] - E_Na);
      i_f         = i_fK + i_fNa;
    
      Xf_infinity = 1.0/(1.0 + exp((ytemp[0]*1000 + 69)/8));
      tau_Xf      = 5600 / (1 + exp((ytemp[0]*1000 + 65)/7) + exp(-(ytemp[0]*1000 + 65)/19));
      ak5[14]   = 1000*(Xf_infinity-ytemp[14])/tau_Xf;
      
    
      //// ICaL 
      //Prevent division by 0
      if(ytemp[0]< precision && ytemp[0] > -precision) //hopital
        i_CaL =  g_CaL*4.0*ytemp[0]*pow((float)F,2.0f)/(R*T) *ytemp[4]*ytemp[5]*ytemp[6]*ytemp[7] / (2.0*F/(R*T)) * (ytemp[2] - 0.341*Cao);
      else
        i_CaL = g_CaL*4.0*ytemp[0]*pow((float)F,2.0f)/(R*T)*(ytemp[2]*exp(2.0*ytemp[0]*F/(R*T))-0.341*Cao)/(exp(2.0*ytemp[0]*F/(R*T))-1.0)*ytemp[4]*ytemp[5]*ytemp[6]*ytemp[7]; //((time<tDrugApplication)*1+(time >= tDrugApplication)*ICaLRedMed)*g_CaL*4.0*ytemp[0]*pow(F,2.0)/(R*T)*(ytemp[2]*exp(2.0*ytemp[0]*F/(R*T))-0.341*Cao)/(exp(2.0*ytemp[0]*F/(R*T))-1.0)*ytemp[4]*ytemp[5]*ytemp[6]*ytemp[7];
    
      d_infinity  = 1.0/(1.0+exp(-(ytemp[0]*1000.0+9.1)/7.0));
      alpha_d     = 0.25+1.4/(1.0+exp((-ytemp[0]*1000.0-35.0)/13.0));
      beta_d      = 1.4/(1.0+exp((ytemp[0]*1000.0+5.0)/5.0));
      gamma_d     = 1.0/(1.0+exp((-ytemp[0]*1000.0+50.0)/20.0));
      tau_d       = (alpha_d*beta_d+gamma_d)*1.0/1000.0;
      ak5[4]    = (d_infinity-ytemp[4])/tau_d;
    
      f1_inf      = 1.0/(1.0+exp((ytemp[0]*1000.0+26.0)/3.0));
      if (f1_inf-ytemp[5] > 0.0)
          constf1 = 1.0+1433.0*(ytemp[2]-50.0*1.0e-6);
      else
          constf1 = 1.0;
    
      tau_f1      = (20.0+1102.5*exp(-pow((float)((ytemp[0]*1000.0+27.0)/15.0),2.0f))+200.0/(1.0+exp((13.0-ytemp[0]*1000.0)/10.0))+180.0/(1.0+exp((30.0+ytemp[0]*1000.0)/10.0)))*constf1/1000.0;
      ak5[5]    = (f1_inf-ytemp[5])/tau_f1;
    
      f2_inf      = 0.33+0.67/(1.0+exp((ytemp[0]*1000.0+32.0)/4.0));
      tau_f2      = (600.0*exp(-pow((float)(ytemp[0]*1000.0+25.0),2.0f)/170.0)+31.0/(1.0+exp((25.0-ytemp[0]*1000.0)/10.0))+16.0/(1.0+exp((30.0+ytemp[0]*1000.0)/10.0)))*constf2/1000.0;
      ak5[6]    = (f2_inf-ytemp[6])/tau_f2;
    
      alpha_fCa   = 1.0/(1.0+pow((float)(ytemp[2]/0.0006),8.0f));
      beta_fCa    = 0.1/(1.0+exp((ytemp[2]-0.0009)/0.0001));
      gamma_fCa   = 0.3/(1.0+exp((ytemp[2]-0.00075)/0.0008));
      fCa_inf     = (alpha_fCa+beta_fCa+gamma_fCa)/1.3156;
    
      if ((ytemp[0] > -0.06) && (fCa_inf > ytemp[7]))
          constfCa = 0.0;
      else
          constfCa = 1.0;
    
      ak5[7]    = constfCa*(fCa_inf-ytemp[7])/tau_fCa;
    
      //// Ito
      i_to        = g_to*(ytemp[0]-E_K)*ytemp[15]*ytemp[16];
    
      q_inf       = 1.0/(1.0+exp((ytemp[0]*1000.0+53.0)/13.0));
      tau_q       = (6.06+39.102/(0.57*exp(-0.08*(ytemp[0]*1000.0+44.0))+0.065*exp(0.1*(ytemp[0]*1000.0+45.93))))/1000.0;
      ak5[15]   = (q_inf-ytemp[15])/tau_q;
    
    
      r_inf       = 1.0/(1.0+exp(-(ytemp[0]*1000.0-22.3)/18.75));
      tau_r       = (2.75352+14.40516/(1.037*exp(0.09*(ytemp[0]*1000.0+30.61))+0.369*exp(-0.12*(ytemp[0]*1000.0+23.84))))/1000.0;
      ak5[16]   = (r_inf-ytemp[16])/tau_r;
    
      //// IKs
      i_Ks        = g_Ks*(ytemp[0]-E_Ks)*pow((float)ytemp[10],2.0f)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/ytemp[2]),1.4f))); // ((time<tDrugApplication)*1+(time >= tDrugApplication)*IKsRedMed)*g_Ks*(ytemp[0]-E_Ks)*pow((float)ytemp[10],2.0)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/ytemp[2]),1.4f)));
    
      Xs_infinity = 1.0/(1.0+exp((-ytemp[0]*1000.0-20.0)/16.0));
      alpha_Xs    = 1100.0/sqrt(1.0+exp((-10.0-ytemp[0]*1000.0)/6.0));
      beta_Xs     = 1.0/(1.0+exp((-60.0+ytemp[0]*1000.0)/20.0));
      tau_Xs      = 1.0*alpha_Xs*beta_Xs/1000.0;
      ak5[10]   = (Xs_infinity-ytemp[10])/tau_Xs;
    
      //// IKr
      i_Kr         = g_Kr*(ytemp[0]-E_K)*ytemp[8]*ytemp[9]*sqrt(Ko/5.4); //((time<tDrugApplication)*1+(time >= tDrugApplication)*IKrRedMed)*g_Kr*(ytemp[0]-E_K)*ytemp[8]*ytemp[9]*sqrt(Ko/5.4);
    
      V_half       = 1000.0*(-R*T/(F*Q)*log(pow((float)(1.0+Cao/2.6),4.0f)/(pow((float)(1.0+Cao/0.58),4.0f)*L0))-0.019);
    
      Xr1_inf      = 1.0/(1.0+exp((V_half-ytemp[0]*1000.0)/4.9));
      alpha_Xr1    = 450.0/(1.0+exp((-45.0-ytemp[0]*1000.0)/10.0));
      beta_Xr1     = 6.0/(1.0+exp((30.0+ytemp[0]*1000.0)/11.5));
      tau_Xr1      = 1.0*alpha_Xr1*beta_Xr1/1000.0;
      ak5[8]     = (Xr1_inf-ytemp[8])/tau_Xr1;
    
      Xr2_infinity = 1.0/(1.0+exp((ytemp[0]*1000.0+88.0)/50.0));
      alpha_Xr2    = 3.0/(1.0+exp((-60.0-ytemp[0]*1000.0)/20.0));
      beta_Xr2     = 1.12/(1.0+exp((-60.0+ytemp[0]*1000.0)/20.0));
      tau_Xr2      = 1.0*alpha_Xr2*beta_Xr2/1000.0;
      ak5[9]    = (Xr2_infinity-ytemp[9])/tau_Xr2;
    
      //// IK1
      alpha_K1    = 3.91/(1.0+exp(0.5942*(ytemp[0]*1000.0-E_K*1000.0-200.0)));
      beta_K1     = (-1.509*exp(0.0002*(ytemp[0]*1000.0-E_K*1000.0+100.0))+exp(0.5886*(ytemp[0]*1000.0-E_K*1000.0-10.0)))/(1.0+exp(0.4547*(ytemp[0]*1000.0-E_K*1000.0)));
      XK1_inf     = alpha_K1/(alpha_K1+beta_K1);
      i_K1        = g_K1*XK1_inf*(ytemp[0]-E_K)*sqrt(Ko/5.4);
    
      //// INaCa
      i_NaCa      = kNaCa*(exp(gamma*ytemp[0]*F/(R*T))*pow((float)ytemp[17],3.0f)*Cao-exp((gamma-1.0)*ytemp[0]*F/(R*T))*pow((float)Nao,3.0f)*ytemp[2]*alpha)/((pow((float)KmNai,3.0f)+pow((float)Nao,3.0f))*(KmCa+Cao)*(1.0+Ksat*exp((gamma-1.0)*ytemp[0]*F/(R*T))));
    
      //// INaK
      i_NaK       = PNaK*Ko/(Ko+Km_K)*ytemp[17]/(ytemp[17]+Km_Na)/(1.0+0.1245*exp(-0.1*ytemp[0]*F/(R*T))+0.0353*exp(-ytemp[0]*F/(R*T)));
    
      //// IpCa
      i_PCa       = g_PCa*ytemp[2]/(ytemp[2]+KPCa);
    
      //// Background currents
      i_b_Na      = g_b_Na*(ytemp[0]-E_Na);
    
      i_b_Ca      = g_b_Ca*(ytemp[0]-E_Ca);
    
      //// Sarcoplasmic reticulum
      i_up        = VmaxUp/(1.0+pow((float)Kup,2.0f)/pow((float)ytemp[2],2.0f));
    
      i_leak      = (ytemp[1]-ytemp[2])*V_leak;
    
      ak5[3]    = 0;
  
    
      RyRSRCass   = (1 - 1/(1 +  exp((ytemp[1]-0.3)/0.1)));
      i_rel       = g_irel_max*RyRSRCass*ytemp[21]*ytemp[22]*(ytemp[1]-ytemp[2]);
    
      RyRainfss   = RyRa1-RyRa2/(1 + exp((1000*ytemp[2]-(RyRahalf))/0.0082));
      ak5[20]   = (RyRainfss- ytemp[20])/RyRtauadapt;
    
      RyRoinfss   = (1 - 1/(1 +  exp((1000*ytemp[2]-(ytemp[20]+ RyRohalf))/0.003)));
      if (RyRoinfss>= ytemp[21])
        RyRtauact = 18.75e-3;       //s
      else
        RyRtauact = 0.1*18.75e-3;   //s
    
      ak5[21]    = (RyRoinfss- ytemp[21])/(RyRtauact);
    
      RyRcinfss   = (1/(1 + exp((1000*ytemp[2]-(ytemp[20]+RyRchalf))/0.001)));
      if (RyRcinfss>= ytemp[22])
        RyRtauinact = 2*87.5e-3;    //s
      else
        RyRtauinact = 87.5e-3;      //s
    
      ak5[22]    = (RyRcinfss- ytemp[22])/(RyRtauinact);
    
    
    
    
      //// Ca2+ buffering
      Cai_bufc    = 1.0/(1.0+Buf_C*Kbuf_C/pow((float)(ytemp[2]+Kbuf_C), 2.0f));
      Ca_SR_bufSR = 1.0/(1.0+Buf_SR*Kbuf_SR/pow((float)(ytemp[1]+Kbuf_SR), 2.0f));
    
      //// Ionic concentrations
      //Nai
      ak5[17]   = -Cm*(i_Na+i_NaL+i_b_Na+3.0*i_NaK+3.0*i_NaCa+i_fNa)/(F*Vc*1.0e-18);
      //Cai
      ak5[2]    = Cai_bufc*(i_leak-i_up+i_rel-(i_CaL+i_b_Ca+i_PCa-2.0*i_NaCa)*Cm/(2.0*Vc*F*1.0e-18));
       //caSR
      ak5[1]    = Ca_SR_bufSR*Vc/V_SR*(i_up-(i_rel+i_leak));
    
      //// Stimulation
    //  i_stim_Amplitude 		= 5.5e-10;//7.5e-10;   // ampere (in stim_mode)
    //  i_stim_End 				= 1000.0;   // second (in stim_mode)
    //  i_stim_PulseDuration	= 0.005;   // second (in stim_mode)
    //  i_stim_Start 			= 0.0;   // second (in stim_mode)
    //  i_stim_frequency        = 60.0;   // per_second (in stim_mode)
      //stim_flag 				= stimFlag;   // dimensionless (in stim_mode)
    //  i_stim_Period 			= 60.0/i_stim_frequency;
    
      //if stim_flag~=0 && stim_flag~=1
      //error('Paci2020: wrong pacing! stimFlag can be only 0 (spontaneous) or 1 (paced)');
      //end
    
      /*
      if ((time >= i_stim_Start) && (time <= i_stim_End) && (time-i_stim_Start-floor((time-i_stim_Start)/i_stim_Period)*i_stim_Period <= i_stim_PulseDuration))
          i_stim = stim_flag*i_stim_Amplitude/Cm;
      else
          i_stim = 0.0;
      */
    
      //// Membrane potential
      ak5[0] = -(i_K1+i_to+i_Kr+i_Ks+i_CaL+i_NaK+i_Na+i_NaL+i_NaCa+i_PCa+i_f+i_b_Na+i_b_Ca-i_stim);      

    //-----SIXTH STEP ------------------------------------------------------------------------------------------------------------------------------------------------------------------      
    for (i=0;i<layers;i++)
      ytemp[i]=y[i]+dt*(b61*dydt[i]+b62*ak2[i]+b63*ak3[i]+b64*ak4[i]+b65*ak5[i]);


      E_Na = R*T/F*log(Nao/ytemp[17]);
      E_Ca = 0.5*R*T/F*log(Cao/ytemp[2]);
      E_K  = R*T/F*log(Ko/Ki);
      E_Ks = R*T/F*log((Ko+PkNa*Nao)/(Ki+PkNa*ytemp[17]));
    
      //// INa adapted from DOI:10.3389/fphys.2018.00080
      i_Na        =  g_Na*pow((float)ytemp[13],3.0f)*ytemp[11]*ytemp[12]*(ytemp[0] - E_Na);
    
      m_inf       = 1 / (1 + exp((ytemp[0]*1000 + 39)/-11.2));
      tau_m       = (0.00001 + 0.00013*exp(-pow((float)((ytemp[0]*1000 + 48)/15),2.0f)) + 0.000045 / (1 + exp((ytemp[0]*1000 + 42)/-5)));
      ak6[13]   = (m_inf-ytemp[13])/tau_m;
    
      h_inf       = 1 / (1 + exp((ytemp[0]*1000 + 66.5)/6.8));
      tau_h       = (0.00007 + 0.034 / (1 + exp((ytemp[0]*1000 + 41)/5.5) + exp(-(ytemp[0]*1000 + 41)/14)) + 0.0002 / (1 + exp(-(ytemp[0]*1000 + 79)/14)));
      ak6[11]   = (h_inf-ytemp[11])/tau_h;
    
      j_inf       = h_inf;
      tau_j       = 10*(0.0007 + 0.15 / (1 + exp((ytemp[0]*1000 + 41)/5.5) + exp(-(ytemp[0]*1000 + 41)/14)) + 0.002 / (1 + exp(-(ytemp[0]*1000 + 79)/14)));
      ak6[12]   = (j_inf-ytemp[12])/tau_j;
    
    
      //// INaL
      i_NaL       = GNaLmax* pow((float)ytemp[18],3.0f)*ytemp[19]*(ytemp[0]-E_Na);
    
      m_inf_L     = 1/(1+exp(-(ytemp[0]*1000+42.85)/(5.264)));
      alpha_m_L   = 1/(1+exp((-60-ytemp[0]*1000)/5));
      beta_m_L    = 0.1/(1+exp((ytemp[0]*1000+35)/5))+0.1/(1+exp((ytemp[0]*1000-50)/200));
      tau_m_L     = 1 * myCoefTauM*alpha_m_L*beta_m_L;
      ak6[18]   = (m_inf_L-ytemp[18])/tau_m_L*1000;
    
      h_inf_L     = 1/(1+exp((ytemp[0]*1000+Vh_hLate)/(7.488)));
      ak6[19]   = (h_inf_L-ytemp[19])/tau_h_L*1000;
    
      //// If adapted from DOI:10.3389/fphys.2018.00080
      i_fK        = fK*g_f*ytemp[14]*(ytemp[0] - E_K);
      i_fNa       = fNa*g_f*ytemp[14]*(ytemp[0] - E_Na);
      i_f         = i_fK + i_fNa;
    
      Xf_infinity = 1.0/(1.0 + exp((ytemp[0]*1000 + 69)/8));
      tau_Xf      = 5600 / (1 + exp((ytemp[0]*1000 + 65)/7) + exp(-(ytemp[0]*1000 + 65)/19));
      ak6[14]   = 1000*(Xf_infinity-ytemp[14])/tau_Xf;
      
    
      //// ICaL
      //Prevent division by 0
      if(ytemp[0]< precision && ytemp[0] > -precision) //hopital
        i_CaL =  g_CaL*4.0*ytemp[0]*pow((float)F,2.0f)/(R*T) *ytemp[4]*ytemp[5]*ytemp[6]*ytemp[7] / (2.0*F/(R*T)) * (ytemp[2] - 0.341*Cao);
      else
        i_CaL = g_CaL*4.0*ytemp[0]*pow((float)F,2.0f)/(R*T)*(ytemp[2]*exp(2.0*ytemp[0]*F/(R*T))-0.341*Cao)/(exp(2.0*ytemp[0]*F/(R*T))-1.0)*ytemp[4]*ytemp[5]*ytemp[6]*ytemp[7]; //((time<tDrugApplication)*1+(time >= tDrugApplication)*ICaLRedMed)*g_CaL*4.0*ytemp[0]*pow(F,2.0)/(R*T)*(ytemp[2]*exp(2.0*ytemp[0]*F/(R*T))-0.341*Cao)/(exp(2.0*ytemp[0]*F/(R*T))-1.0)*ytemp[4]*ytemp[5]*ytemp[6]*ytemp[7];
    
      d_infinity  = 1.0/(1.0+exp(-(ytemp[0]*1000.0+9.1)/7.0));
      alpha_d     = 0.25+1.4/(1.0+exp((-ytemp[0]*1000.0-35.0)/13.0));
      beta_d      = 1.4/(1.0+exp((ytemp[0]*1000.0+5.0)/5.0));
      gamma_d     = 1.0/(1.0+exp((-ytemp[0]*1000.0+50.0)/20.0));
      tau_d       = (alpha_d*beta_d+gamma_d)*1.0/1000.0;
      ak6[4]    = (d_infinity-ytemp[4])/tau_d;
    
      f1_inf      = 1.0/(1.0+exp((ytemp[0]*1000.0+26.0)/3.0));
      if (f1_inf-ytemp[5] > 0.0)
          constf1 = 1.0+1433.0*(ytemp[2]-50.0*1.0e-6);
      else
          constf1 = 1.0;
    
      tau_f1      = (20.0+1102.5*exp(-pow((float)((ytemp[0]*1000.0+27.0)/15.0),2.0f))+200.0/(1.0+exp((13.0-ytemp[0]*1000.0)/10.0))+180.0/(1.0+exp((30.0+ytemp[0]*1000.0)/10.0)))*constf1/1000.0;
      ak6[5]    = (f1_inf-ytemp[5])/tau_f1;
    
      f2_inf      = 0.33+0.67/(1.0+exp((ytemp[0]*1000.0+32.0)/4.0));
      tau_f2      = (600.0*exp(-pow((float)(ytemp[0]*1000.0+25.0),2.0f)/170.0)+31.0/(1.0+exp((25.0-ytemp[0]*1000.0)/10.0))+16.0/(1.0+exp((30.0+ytemp[0]*1000.0)/10.0)))*constf2/1000.0;
      ak6[6]    = (f2_inf-ytemp[6])/tau_f2;
    
      alpha_fCa   = 1.0/(1.0+pow((float)(ytemp[2]/0.0006),8.0f));
      beta_fCa    = 0.1/(1.0+exp((ytemp[2]-0.0009)/0.0001));
      gamma_fCa   = 0.3/(1.0+exp((ytemp[2]-0.00075)/0.0008));
      fCa_inf     = (alpha_fCa+beta_fCa+gamma_fCa)/1.3156;
    
      if ((ytemp[0] > -0.06) && (fCa_inf > ytemp[7]))
          constfCa = 0.0;
      else
          constfCa = 1.0;
    
      ak6[7]    = constfCa*(fCa_inf-ytemp[7])/tau_fCa;
    
      //// Ito
      i_to        = g_to*(ytemp[0]-E_K)*ytemp[15]*ytemp[16];
    
      q_inf       = 1.0/(1.0+exp((ytemp[0]*1000.0+53.0)/13.0));
      tau_q       = (6.06+39.102/(0.57*exp(-0.08*(ytemp[0]*1000.0+44.0))+0.065*exp(0.1*(ytemp[0]*1000.0+45.93))))/1000.0;
      ak6[15]   = (q_inf-ytemp[15])/tau_q;
    
    
      r_inf       = 1.0/(1.0+exp(-(ytemp[0]*1000.0-22.3)/18.75));
      tau_r       = (2.75352+14.40516/(1.037*exp(0.09*(ytemp[0]*1000.0+30.61))+0.369*exp(-0.12*(ytemp[0]*1000.0+23.84))))/1000.0;
      ak6[16]   = (r_inf-ytemp[16])/tau_r;
    
      //// IKs
      i_Ks        = g_Ks*(ytemp[0]-E_Ks)*pow((float)ytemp[10],2.0f)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/ytemp[2]),1.4f))); // ((time<tDrugApplication)*1+(time >= tDrugApplication)*IKsRedMed)*g_Ks*(ytemp[0]-E_Ks)*pow((float)ytemp[10],2.0)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/ytemp[2]),1.4f)));
    
      Xs_infinity = 1.0/(1.0+exp((-ytemp[0]*1000.0-20.0)/16.0));
      alpha_Xs    = 1100.0/sqrt(1.0+exp((-10.0-ytemp[0]*1000.0)/6.0));
      beta_Xs     = 1.0/(1.0+exp((-60.0+ytemp[0]*1000.0)/20.0));
      tau_Xs      = 1.0*alpha_Xs*beta_Xs/1000.0;
      ak6[10]   = (Xs_infinity-ytemp[10])/tau_Xs;
    
      //// IKr
      i_Kr         = g_Kr*(ytemp[0]-E_K)*ytemp[8]*ytemp[9]*sqrt(Ko/5.4); //((time<tDrugApplication)*1+(time >= tDrugApplication)*IKrRedMed)*g_Kr*(ytemp[0]-E_K)*ytemp[8]*ytemp[9]*sqrt(Ko/5.4);
    
      V_half       = 1000.0*(-R*T/(F*Q)*log(pow((float)(1.0+Cao/2.6),4.0f)/(pow((float)(1.0+Cao/0.58),4.0f)*L0))-0.019);
    
      Xr1_inf      = 1.0/(1.0+exp((V_half-ytemp[0]*1000.0)/4.9));
      alpha_Xr1    = 450.0/(1.0+exp((-45.0-ytemp[0]*1000.0)/10.0));
      beta_Xr1     = 6.0/(1.0+exp((30.0+ytemp[0]*1000.0)/11.5));
      tau_Xr1      = 1.0*alpha_Xr1*beta_Xr1/1000.0;
      ak6[8]     = (Xr1_inf-ytemp[8])/tau_Xr1;
    
      Xr2_infinity = 1.0/(1.0+exp((ytemp[0]*1000.0+88.0)/50.0));
      alpha_Xr2    = 3.0/(1.0+exp((-60.0-ytemp[0]*1000.0)/20.0));
      beta_Xr2     = 1.12/(1.0+exp((-60.0+ytemp[0]*1000.0)/20.0));
      tau_Xr2      = 1.0*alpha_Xr2*beta_Xr2/1000.0;
      ak6[9]    = (Xr2_infinity-ytemp[9])/tau_Xr2;
    
      //// IK1
      alpha_K1    = 3.91/(1.0+exp(0.5942*(ytemp[0]*1000.0-E_K*1000.0-200.0)));
      beta_K1     = (-1.509*exp(0.0002*(ytemp[0]*1000.0-E_K*1000.0+100.0))+exp(0.5886*(ytemp[0]*1000.0-E_K*1000.0-10.0)))/(1.0+exp(0.4547*(ytemp[0]*1000.0-E_K*1000.0)));
      XK1_inf     = alpha_K1/(alpha_K1+beta_K1);
      i_K1        = g_K1*XK1_inf*(ytemp[0]-E_K)*sqrt(Ko/5.4);
    
      //// INaCa
      i_NaCa      = kNaCa*(exp(gamma*ytemp[0]*F/(R*T))*pow((float)ytemp[17],3.0f)*Cao-exp((gamma-1.0)*ytemp[0]*F/(R*T))*pow((float)Nao,3.0f)*ytemp[2]*alpha)/((pow((float)KmNai,3.0f)+pow((float)Nao,3.0f))*(KmCa+Cao)*(1.0+Ksat*exp((gamma-1.0)*ytemp[0]*F/(R*T))));
    
      //// INaK
      i_NaK       = PNaK*Ko/(Ko+Km_K)*ytemp[17]/(ytemp[17]+Km_Na)/(1.0+0.1245*exp(-0.1*ytemp[0]*F/(R*T))+0.0353*exp(-ytemp[0]*F/(R*T)));
    
      //// IpCa
      i_PCa       = g_PCa*ytemp[2]/(ytemp[2]+KPCa);
    
      //// Background currents
      i_b_Na      = g_b_Na*(ytemp[0]-E_Na);
    
      i_b_Ca      = g_b_Ca*(ytemp[0]-E_Ca);
    
      //// Sarcoplasmic reticulum
      i_up        = VmaxUp/(1.0+pow((float)Kup,2.0f)/pow((float)ytemp[2],2.0f));
    
      i_leak      = (ytemp[1]-ytemp[2])*V_leak;
    
      ak6[3]    = 0;
    
    
      RyRSRCass   = (1 - 1/(1 +  exp((ytemp[1]-0.3)/0.1)));
      i_rel       = g_irel_max*RyRSRCass*ytemp[21]*ytemp[22]*(ytemp[1]-ytemp[2]);
    
      RyRainfss   = RyRa1-RyRa2/(1 + exp((1000*ytemp[2]-(RyRahalf))/0.0082));
      ak6[20]   = (RyRainfss- ytemp[20])/RyRtauadapt;
    
      RyRoinfss   = (1 - 1/(1 +  exp((1000*ytemp[2]-(ytemp[20]+ RyRohalf))/0.003)));
      if (RyRoinfss>= ytemp[21])
        RyRtauact = 18.75e-3;       //s
      else
        RyRtauact = 0.1*18.75e-3;   //s
    
      ak6[21]    = (RyRoinfss- ytemp[21])/(RyRtauact);
    
      RyRcinfss   = (1/(1 + exp((1000*ytemp[2]-(ytemp[20]+RyRchalf))/0.001)));
      if (RyRcinfss>= ytemp[22])
        RyRtauinact = 2*87.5e-3;    //s
      else
        RyRtauinact = 87.5e-3;      //s
    
      ak6[22]    = (RyRcinfss- ytemp[22])/(RyRtauinact);
    
    
    
    
      //// Ca2+ buffering
      Cai_bufc    = 1.0/(1.0+Buf_C*Kbuf_C/pow((float)(ytemp[2]+Kbuf_C), 2.0f));
      Ca_SR_bufSR = 1.0/(1.0+Buf_SR*Kbuf_SR/pow((float)(ytemp[1]+Kbuf_SR), 2.0f));
    
      //// Ionic concentrations
      //Nai
      ak6[17]   = -Cm*(i_Na+i_NaL+i_b_Na+3.0*i_NaK+3.0*i_NaCa+i_fNa)/(F*Vc*1.0e-18);
      //Cai
      ak6[2]    = Cai_bufc*(i_leak-i_up+i_rel-(i_CaL+i_b_Ca+i_PCa-2.0*i_NaCa)*Cm/(2.0*Vc*F*1.0e-18));
       //caSR
      ak6[1]    = Ca_SR_bufSR*Vc/V_SR*(i_up-(i_rel+i_leak));
    
      //// Stimulation
      //  i_stim_Amplitude 		= 5.5e-10;//7.5e-10;   // ampere (in stim_mode)
      //  i_stim_End 				= 1000.0;   // second (in stim_mode)
      //  i_stim_PulseDuration	= 0.005;   // second (in stim_mode)
      //  i_stim_Start 			= 0.0;   // second (in stim_mode)
      //  i_stim_frequency        = 60.0;   // per_second (in stim_mode)
      //stim_flag 				= stimFlag;   // dimensionless (in stim_mode)
      //  i_stim_Period 			= 60.0/i_stim_frequency;
    
      //if stim_flag~=0 && stim_flag~=1_mode)
      //  i_stim_Period 			= 60.0/i_stim_frequency;
    
      //if stim_flag~=0 && stim_flag~=1
      //error('Paci2020: wrong pacing! stimFlag can be only 0 (spontaneous) or 1 (paced)');
      //end
    
      /*
      if ((time >= i_stim_Start) && (time <= i_stim_End) && (time-i_stim_Start-floor((time-i_stim_Start)/i_stim_Period)*i_stim_Period <= i_stim_PulseDuration))
          i_stim = stim_flag*i_stim_Amplitude/Cm;
      else
          i_stim = 0.0;
      */
    
      //// Membrane potential
      ak6[0] = -(i_K1+i_to+i_Kr+i_Ks+i_CaL+i_NaK+i_Na+i_NaL+i_NaCa+i_PCa+i_f+i_b_Na+i_b_Ca-i_stim);
  
      //-----WRITE NEW VALUES ------------------------------------------------------------------------------------------------------------------------------------------------------------------      
      for (i=0;i<layers;i++){ //Accumulate increments with proper weights.
        yout[i]=y[i]+dt*(c1*dydt[i]+c3*ak3[i]+c4*ak4[i]+c6*ak6[i]);
        yerr[i]=dt*(dc1*dydt[i]+dc3*ak3[i]+dc4*ak4[i]+dc5*ak5[i]+dc6*ak6[i]);
      }

      #pragma unroll
      for (i=0;i<layers;i++) //Accumulate increments with proper weights.
        alt_PDEvars[i*sizex*sizey + id]=y[i]+(c1*dydt[i]+c3*ak3[i]+c4*ak4[i]+c6*ak6[i])*dt;
    }  
  
  }
}
#endif

__device__ void derivsPaci(PDEFIELD_TYPE current_time, PDEFIELD_TYPE* y, PDEFIELD_TYPE* dydt, bool celltype2, PDEFIELD_TYPE pacing_interval, PDEFIELD_TYPE pacing_duration, PDEFIELD_TYPE pacing_strength){

  //Declare variables needed for Paci2020 model and assign the constants

  //// Constants
  PDEFIELD_TYPE F = 96485.3415;     // coulomb_per_mole (in model_parameters)
  PDEFIELD_TYPE R = 8.314472;       // joule_per_mole_kelvin (in model_parameters)
  PDEFIELD_TYPE T = 310.0;          // kelvin (in model_parameters) //37°C

  //// Cell geometry
  PDEFIELD_TYPE V_SR = 583.73;        // micrometre_cube (in model_parameters)
  PDEFIELD_TYPE Vc   = 8800.0;        // micrometre_cube (in model_parameters)
  PDEFIELD_TYPE Cm   = 9.87109e-11;   // farad (in model_parameters)

  //// Extracellular concentrations
  PDEFIELD_TYPE Nao = 151.0; // millimolar (in model_parameters)
  PDEFIELD_TYPE Ko  = 5.4;   // millimolar (in model_parameters)
  PDEFIELD_TYPE Cao = 1.8;   // millimolar (in model_parameters)

  //// Intracellular concentrations
  // Naio = 10 mM y[17]
  PDEFIELD_TYPE Ki = 150.0;   // millimolar (in model_parameters)
  // Cai  = 0.0002 mM y[2]
  // caSR = 0.3 mM y[1]  
    
      //// Nernst potential
  PDEFIELD_TYPE E_Na;
  PDEFIELD_TYPE E_Ca;
  PDEFIELD_TYPE E_K;
  PDEFIELD_TYPE PkNa = 0.03;   // dimensionless (in electric_potentials)
  PDEFIELD_TYPE E_Ks;
    
  //// INa adapted from DOI:10.3389/fphys.2018.00080
  PDEFIELD_TYPE g_Na = 3671.2302; //((time<tDrugApplication)*1+(time >= tDrugApplication)*INaFRedMed)*6447.1896;
  PDEFIELD_TYPE i_Na;
    
  PDEFIELD_TYPE m_inf;
  PDEFIELD_TYPE tau_m;
    
  PDEFIELD_TYPE h_inf;
  PDEFIELD_TYPE tau_h;
    
  PDEFIELD_TYPE j_inf;
  PDEFIELD_TYPE tau_j;
    
    
  //// INaL
  PDEFIELD_TYPE myCoefTauM  = 1;
  PDEFIELD_TYPE tauINaL = 200; //ms
  PDEFIELD_TYPE GNaLmax = 17.25;//((time<tDrugApplication)*1+(time >= tDrugApplication)*INaLRedMed)* 2.3*7.5; //(S/F)
  PDEFIELD_TYPE Vh_hLate = 87.61;
  PDEFIELD_TYPE i_NaL;
    
  PDEFIELD_TYPE m_inf_L;
  PDEFIELD_TYPE alpha_m_L;
  PDEFIELD_TYPE beta_m_L;
  PDEFIELD_TYPE tau_m_L;
    
  PDEFIELD_TYPE h_inf_L;
  PDEFIELD_TYPE tau_h_L = 1 * tauINaL;
    
  //// If adapted from DOI:10.3389/fphys.2018.00080
  PDEFIELD_TYPE g_f = 1; //((time<tDrugApplication)*1+(time >= tDrugApplication)*IfRedMed)*22.2763088;
  PDEFIELD_TYPE fNa = 0.37;
  PDEFIELD_TYPE fK = 1 - fNa;
  PDEFIELD_TYPE i_fK;
  PDEFIELD_TYPE i_fNa;
  PDEFIELD_TYPE i_f;
    
  PDEFIELD_TYPE Xf_infinity;
  PDEFIELD_TYPE tau_Xf; 
    
      //// ICaL
  PDEFIELD_TYPE g_CaL = 8.635702e-5;   // metre_cube_per_F_per_s (in i_CaL)
  PDEFIELD_TYPE i_CaL;  
  PDEFIELD_TYPE precision = 0.0001;     
    
  PDEFIELD_TYPE d_infinity;
  PDEFIELD_TYPE alpha_d;
  PDEFIELD_TYPE beta_d;
  PDEFIELD_TYPE gamma_d;
  PDEFIELD_TYPE tau_d;
    
  PDEFIELD_TYPE f1_inf;
  PDEFIELD_TYPE constf1;
    
  PDEFIELD_TYPE tau_f1;
    
  PDEFIELD_TYPE f2_inf;
  PDEFIELD_TYPE constf2 = 1.0;
  PDEFIELD_TYPE tau_f2;
    
  PDEFIELD_TYPE alpha_fCa;
  PDEFIELD_TYPE beta_fCa;
  PDEFIELD_TYPE gamma_fCa;
  PDEFIELD_TYPE fCa_inf;
    
  PDEFIELD_TYPE constfCa;
    
  PDEFIELD_TYPE tau_fCa     = 0.002;   // second (in i_CaL_fCa_gate)
    
  //// Ito
  PDEFIELD_TYPE g_to = 29.9038;   // S_per_F (in i_to)  
  PDEFIELD_TYPE i_to;
    
  PDEFIELD_TYPE q_inf;
  PDEFIELD_TYPE tau_q;
    
    
  PDEFIELD_TYPE r_inf;
  PDEFIELD_TYPE tau_r;
    
  //// IKs
  PDEFIELD_TYPE g_Ks = 2.041;   // S_per_F (in i_Ks)
  PDEFIELD_TYPE i_Ks; // ((time<tDrugApplication)*1+(time >= tDrugApplication)*IKsRedMed)*g_Ks*(y[0]-E_Ks)*pow((float)y[10],2.0)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/y[2]),1.4f)));
    
  PDEFIELD_TYPE Xs_infinity;
  PDEFIELD_TYPE alpha_Xs;
  PDEFIELD_TYPE beta_Xs;
  PDEFIELD_TYPE tau_Xs;
    
  //// IKr
  PDEFIELD_TYPE L0 = 0.025;   // dimensionless (in i_Kr_Xr1_gate)
  PDEFIELD_TYPE Q = 2.3;     // dimensionless (in i_Kr_Xr1_gate)
  PDEFIELD_TYPE g_Kr = 29.8667;   // S_per_F (in i_Kr)
  PDEFIELD_TYPE i_Kr;
    
  PDEFIELD_TYPE V_half;
    
  PDEFIELD_TYPE Xr1_inf;
  PDEFIELD_TYPE alpha_Xr1;
  PDEFIELD_TYPE beta_Xr1;
  PDEFIELD_TYPE tau_Xr1;
    
  PDEFIELD_TYPE Xr2_infinity;
  PDEFIELD_TYPE alpha_Xr2;
  PDEFIELD_TYPE beta_Xr2;
  PDEFIELD_TYPE tau_Xr2;
    
  //// IK1
  PDEFIELD_TYPE alpha_K1;
  PDEFIELD_TYPE beta_K1;
  PDEFIELD_TYPE XK1_inf;
  PDEFIELD_TYPE g_K1 = 28.1492;   // S_per_F (in i_K1)
  PDEFIELD_TYPE i_K1;
    
  //// INaCa
  PDEFIELD_TYPE KmCa = 1.38;   // millimolar (in i_NaCa)
  PDEFIELD_TYPE KmNai = 87.5;   // millimolar (in i_NaCa)
  PDEFIELD_TYPE Ksat = 0.1;    // dimensionless (in i_NaCa)
  PDEFIELD_TYPE gamma = 0.35;   // dimensionless (in i_NaCa)
  PDEFIELD_TYPE alpha = 2.16659;
  PDEFIELD_TYPE kNaCa = 3917.0463; //((time<tDrugApplication)*1+(time >= tDrugApplication)*INaCaRedMed) * 6514.47574;   // A_per_F (in i_NaCa)
  PDEFIELD_TYPE i_NaCa;

  //// INaK
  PDEFIELD_TYPE Km_K = 1.0;    // millimolar (in i_NaK)
  PDEFIELD_TYPE Km_Na = 40.0;   // millimolar (in i_NaK)
  PDEFIELD_TYPE PNaK = 2.74240;// A_per_F (in i_NaK)
  PDEFIELD_TYPE i_NaK;
    
  //// IpCa
  PDEFIELD_TYPE KPCa = 0.0005;   // millimolar (in i_PCa)
  PDEFIELD_TYPE g_PCa = 0.4125;   // A_per_F (in i_PCa)
  PDEFIELD_TYPE i_PCa;
    
  //// Background currents
  PDEFIELD_TYPE g_b_Na = 1.14;         // S_per_F (in i_b_Na)
  PDEFIELD_TYPE i_b_Na;
    
  PDEFIELD_TYPE g_b_Ca = 0.8727264;    // S_per_F (in i_b_Ca)
  PDEFIELD_TYPE i_b_Ca;

  PDEFIELD_TYPE i_up;
  PDEFIELD_TYPE i_leak;
    
  //// Sarcoplasmic reticulum
  PDEFIELD_TYPE VmaxUp = 0.82205;
  PDEFIELD_TYPE Kup	= 4.40435e-4;
    
  PDEFIELD_TYPE V_leak = 4.48209e-4;
    
  // RyR
  PDEFIELD_TYPE g_irel_max = 55.808061;
  PDEFIELD_TYPE RyRa1 = 0.05169;
  PDEFIELD_TYPE RyRa2 = 0.050001;
  PDEFIELD_TYPE RyRahalf = 0.02632;
  PDEFIELD_TYPE RyRohalf = 0.00944;
  PDEFIELD_TYPE RyRchalf = 0.00167;
    
  PDEFIELD_TYPE RyRSRCass;
  PDEFIELD_TYPE i_rel;
    
  PDEFIELD_TYPE RyRainfss;
  PDEFIELD_TYPE RyRtauadapt = 1; //s
    
  PDEFIELD_TYPE RyRoinfss;
  PDEFIELD_TYPE RyRtauact;
    
  PDEFIELD_TYPE RyRcinfss;
  PDEFIELD_TYPE RyRtauinact;

  //// Ca2+ buffering
  PDEFIELD_TYPE Buf_C = 0.25;   // millimolar (in calcium_dynamics)
  PDEFIELD_TYPE Buf_SR = 10.0;   // millimolar (in calcium_dynamics)
  PDEFIELD_TYPE Kbuf_C = 0.001;   // millimolar (in calcium_dynamics)
  PDEFIELD_TYPE Kbuf_SR = 0.3;   // millimolar (in calcium_dynamics)
  PDEFIELD_TYPE Cai_bufc;
  PDEFIELD_TYPE Ca_SR_bufSR;
    
      //// Stimulation
    //  PDEFIELD_TYPE i_stim_Amplitude 		= 5.5e-10;//7.5e-10;   // ampere (in stim_mode)
    //  PDEFIELD_TYPE i_stim_End 				= 1000.0;   // second (in stim_mode)
    //  PDEFIELD_TYPE i_stim_PulseDuration	= 0.005;   // second (in stim_mode)
    //  PDEFIELD_TYPE i_stim_Start 			= 0.0;   // second (in stim_mode)
    //  PDEFIELD_TYPE i_stim_frequency        = 60.0;   // per_second (in stim_mode)
      //PDEFIELD_TYPE stim_flag 				= stimFlag;   // dimensionless (in stim_mode)
    //  PDEFIELD_TYPE i_stim_Period 			= 60.0/i_stim_frequency;
    
      //if stim_flag~=0 && stim_flag~=1
      //error('Paci2020: wrong pacing! stimFlag can be only 0 (spontaneous) or 1 (paced)');
      //end
    
      /*
      if ((time >= i_stim_Start) && (time <= i_stim_End) && (time-i_stim_Start-floor((time-i_stim_Start)/i_stim_Period)*i_stim_Period <= i_stim_PulseDuration))
          i_stim = stim_flag*i_stim_Amplitude/Cm;
      else
          i_stim = 0.0;
      */
  PDEFIELD_TYPE i_stim = 0;
  

  //-----FIRST STEP ------------------------------------------------------------------------------------------------------------------------------------------------------------------  
   
  //// Nernst potential
  E_Na = R*T/F*log(Nao/y[17]);
  E_Ca = 0.5*R*T/F*log(Cao/y[2]);
  E_K  = R*T/F*log(Ko/Ki);
  E_Ks = R*T/F*log((Ko+PkNa*Nao)/(Ki+PkNa*y[17]));
    
  //// INa adapted from DOI:10.3389/fphys.2018.00080
  i_Na        =  g_Na*pow((float)y[13],3.0f)*y[11]*y[12]*(y[0] - E_Na);
    
  m_inf       = 1 / (1 + exp((y[0]*1000 + 39)/-11.2));
  tau_m       = (0.00001 + 0.00013*exp(-pow((float)((y[0]*1000 + 48)/15),2.0f)) + 0.000045 / (1 + exp((y[0]*1000 + 42)/-5)));
  dydt[13]   = (m_inf-y[13])/tau_m;
    
  h_inf       = 1 / (1 + exp((y[0]*1000 + 66.5)/6.8));
  tau_h       = (0.00007 + 0.034 / (1 + exp((y[0]*1000 + 41)/5.5) + exp(-(y[0]*1000 + 41)/14)) + 0.0002 / (1 + exp(-(y[0]*1000 + 79)/14)));
  dydt[11]   = (h_inf-y[11])/tau_h;
    
  j_inf       = h_inf;
  tau_j       = 10*(0.0007 + 0.15 / (1 + exp((y[0]*1000 + 41)/5.5) + exp(-(y[0]*1000 + 41)/14)) + 0.002 / (1 + exp(-(y[0]*1000 + 79)/14)));
  dydt[12]   = (j_inf-y[12])/tau_j;
    
    
  //// INaL
  tauINaL     = 200; //ms
  GNaLmax     = 17.25;//((time<tDrugApplication)*1+(time >= tDrugApplication)*INaLRedMed)* 2.3*7.5; //(S/F)
  Vh_hLate    = 87.61;
  i_NaL       = GNaLmax* pow((float)y[18],3.0f)*y[19]*(y[0]-E_Na);
    
  m_inf_L     = 1/(1+exp(-(y[0]*1000+42.85)/(5.264)));
  alpha_m_L   = 1/(1+exp((-60-y[0]*1000)/5));
  beta_m_L    = 0.1/(1+exp((y[0]*1000+35)/5))+0.1/(1+exp((y[0]*1000-50)/200));
  tau_m_L     = 1 * myCoefTauM*alpha_m_L*beta_m_L;
  dydt[18]   = (m_inf_L-y[18])/tau_m_L*1000;
    
  h_inf_L     = 1/(1+exp((y[0]*1000+Vh_hLate)/(7.488)));
  tau_h_L     = 1 * tauINaL;
  dydt[19]   = (h_inf_L-y[19])/tau_h_L*1000;
    
  //// If adapted from DOI:10.3389/fphys.2018.00080
  i_fK        = fK*g_f*y[14]*(y[0] - E_K);
  i_fNa       = fNa*g_f*y[14]*(y[0] - E_Na);
  i_f         = i_fK + i_fNa;
    
  Xf_infinity = 1.0/(1.0 + exp((y[0]*1000 + 69)/8));
  tau_Xf      = 5600 / (1 + exp((y[0]*1000 + 65)/7) + exp(-(y[0]*1000 + 65)/19));
  dydt[14]   = 1000*(Xf_infinity-y[14])/tau_Xf;
      
    
  //// ICaL
  //Prevent division by 0
  if(y[0]< precision && y[0] > -precision) //hopital
    i_CaL =  g_CaL*4.0*y[0]*pow((float)F,2.0f)/(R*T) *y[4]*y[5]*y[6]*y[7] / (2.0*F/(R*T)) * (y[2] - 0.341*Cao);
  else
    i_CaL = g_CaL*4.0*y[0]*pow((float)F,2.0f)/(R*T)*(y[2]*exp(2.0*y[0]*F/(R*T))-0.341*Cao)/(exp(2.0*y[0]*F/(R*T))-1.0)*y[4]*y[5]*y[6]*y[7]; //((time<tDrugApplication)*1+(time >= tDrugApplication)*ICaLRedMed)*g_CaL*4.0*y[0]*pow(F,2.0)/(R*T)*(y[2]*exp(2.0*y[0]*F/(R*T))-0.341*Cao)/(exp(2.0*y[0]*F/(R*T))-1.0)*y[4]*y[5]*y[6]*y[7];
    
  d_infinity  = 1.0/(1.0+exp(-(y[0]*1000.0+9.1)/7.0));
  alpha_d     = 0.25+1.4/(1.0+exp((-y[0]*1000.0-35.0)/13.0));
  beta_d      = 1.4/(1.0+exp((y[0]*1000.0+5.0)/5.0));
  gamma_d     = 1.0/(1.0+exp((-y[0]*1000.0+50.0)/20.0));
  tau_d       = (alpha_d*beta_d+gamma_d)*1.0/1000.0;
  dydt[4]    = (d_infinity-y[4])/tau_d;
    
  f1_inf      = 1.0/(1.0+exp((y[0]*1000.0+26.0)/3.0));
  if (f1_inf-y[5] > 0.0)
      constf1 = 1.0+1433.0*(y[2]-50.0*1.0e-6);
  else
      constf1 = 1.0;
  
  tau_f1      = (20.0+1102.5*exp(-pow((float)((y[0]*1000.0+27.0)/15.0),2.0f))+200.0/(1.0+exp((13.0-y[0]*1000.0)/10.0))+180.0/(1.0+exp((30.0+y[0]*1000.0)/10.0)))*constf1/1000.0;
  dydt[5]    = (f1_inf-y[5])/tau_f1;
   
  f2_inf      = 0.33+0.67/(1.0+exp((y[0]*1000.0+32.0)/4.0));
  tau_f2      = (600.0*exp(-pow((float)(y[0]*1000.0+25.0),2.0f)/170.0)+31.0/(1.0+exp((25.0-y[0]*1000.0)/10.0))+16.0/(1.0+exp((30.0+y[0]*1000.0)/10.0)))*constf2/1000.0;
  dydt[6]    = (f2_inf-y[6])/tau_f2;
   
  alpha_fCa   = 1.0/(1.0+pow((float)(y[2]/0.0006),8.0f));
  beta_fCa    = 0.1/(1.0+exp((y[2]-0.0009)/0.0001));
  gamma_fCa   = 0.3/(1.0+exp((y[2]-0.00075)/0.0008));
  fCa_inf     = (alpha_fCa+beta_fCa+gamma_fCa)/1.3156;
    
  if ((y[0] > -0.06) && (fCa_inf > y[7]))
    constfCa = 0.0;
  else
    constfCa = 1.0;
    
  dydt[7]    = constfCa*(fCa_inf-y[7])/tau_fCa;
    
  //// Ito
  i_to        = g_to*(y[0]-E_K)*y[15]*y[16];
    
  q_inf       = 1.0/(1.0+exp((y[0]*1000.0+53.0)/13.0));
  tau_q       = (6.06+39.102/(0.57*exp(-0.08*(y[0]*1000.0+44.0))+0.065*exp(0.1*(y[0]*1000.0+45.93))))/1000.0;
  dydt[15]   = (q_inf-y[15])/tau_q;
    
    
  r_inf       = 1.0/(1.0+exp(-(y[0]*1000.0-22.3)/18.75));
  tau_r       = (2.75352+14.40516/(1.037*exp(0.09*(y[0]*1000.0+30.61))+0.369*exp(-0.12*(y[0]*1000.0+23.84))))/1000.0;
  dydt[16]   = (r_inf-y[16])/tau_r;
    
  //// IKs
  i_Ks        = g_Ks*(y[0]-E_Ks)*pow((float)y[10],2.0f)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/y[2]),1.4f))); // ((time<tDrugApplication)*1+(time >= tDrugApplication)*IKsRedMed)*g_Ks*(y[0]-E_Ks)*pow((float)y[10],2.0)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/y[2]),1.4f)));
    
  Xs_infinity = 1.0/(1.0+exp((-y[0]*1000.0-20.0)/16.0));
  alpha_Xs    = 1100.0/sqrt(1.0+exp((-10.0-y[0]*1000.0)/6.0));
  beta_Xs     = 1.0/(1.0+exp((-60.0+y[0]*1000.0)/20.0));
  tau_Xs      = 1.0*alpha_Xs*beta_Xs/1000.0;
  dydt[10]   = (Xs_infinity-y[10])/tau_Xs;
    
  //// IKr
  i_Kr         = g_Kr*(y[0]-E_K)*y[8]*y[9]*sqrt(Ko/5.4); //((time<tDrugApplication)*1+(time >= tDrugApplication)*IKrRedMed)*g_Kr*(y[0]-E_K)*y[8]*y[9]*sqrt(Ko/5.4);
    
  V_half       = 1000.0*(-R*T/(F*Q)*log(pow((float)(1.0+Cao/2.6),4.0f)/(pow((float)(1.0+Cao/0.58),4.0f)*L0))-0.019);
    
  Xr1_inf      = 1.0/(1.0+exp((V_half-y[0]*1000.0)/4.9));
  alpha_Xr1    = 450.0/(1.0+exp((-45.0-y[0]*1000.0)/10.0));
  beta_Xr1     = 6.0/(1.0+exp((30.0+y[0]*1000.0)/11.5));
  tau_Xr1      = 1.0*alpha_Xr1*beta_Xr1/1000.0;
  dydt[8]     = (Xr1_inf-y[8])/tau_Xr1;
    
  Xr2_infinity = 1.0/(1.0+exp((y[0]*1000.0+88.0)/50.0));
  alpha_Xr2    = 3.0/(1.0+exp((-60.0-y[0]*1000.0)/20.0));
  beta_Xr2     = 1.12/(1.0+exp((-60.0+y[0]*1000.0)/20.0));
  tau_Xr2      = 1.0*alpha_Xr2*beta_Xr2/1000.0;
  dydt[9]    = (Xr2_infinity-y[9])/tau_Xr2;
    
  //// IK1
  alpha_K1    = 3.91/(1.0+exp(0.5942*(y[0]*1000.0-E_K*1000.0-200.0)));
  beta_K1     = (-1.509*exp(0.0002*(y[0]*1000.0-E_K*1000.0+100.0))+exp(0.5886*(y[0]*1000.0-E_K*1000.0-10.0)))/(1.0+exp(0.4547*(y[0]*1000.0-E_K*1000.0)));
  XK1_inf     = alpha_K1/(alpha_K1+beta_K1);
  i_K1        = g_K1*XK1_inf*(y[0]-E_K)*sqrt(Ko/5.4);
    
  //// INaCa
  i_NaCa      = kNaCa*(exp(gamma*y[0]*F/(R*T))*pow((float)y[17],3.0f)*Cao-exp((gamma-1.0)*y[0]*F/(R*T))*pow((float)Nao,3.0f)*y[2]*alpha)/((pow((float)KmNai,3.0f)+pow((float)Nao,3.0f))*(KmCa+Cao)*(1.0+Ksat*exp((gamma-1.0)*y[0]*F/(R*T))));
    
  //// INaK
  i_NaK       = PNaK*Ko/(Ko+Km_K)*y[17]/(y[17]+Km_Na)/(1.0+0.1245*exp(-0.1*y[0]*F/(R*T))+0.0353*exp(-y[0]*F/(R*T)));
    
  //// IpCa
  i_PCa       = g_PCa*y[2]/(y[2]+KPCa);
    
  //// Background currents
  i_b_Na      = g_b_Na*(y[0]-E_Na);
    
  i_b_Ca      = g_b_Ca*(y[0]-E_Ca);
   
  //// Sarcoplasmic reticulum
  i_up        = VmaxUp/(1.0+pow((float)Kup,2.0f)/pow((float)y[2],2.0f));
    
  i_leak      = (y[1]-y[2])*V_leak;
    
  dydt[3]    = 0;
    
  // RyR
    
  RyRSRCass   = (1 - 1/(1 +  exp((y[1]-0.3)/0.1)));
  i_rel       = g_irel_max*RyRSRCass*y[21]*y[22]*(y[1]-y[2]);
    
  RyRainfss   = RyRa1-RyRa2/(1 + exp((1000*y[2]-(RyRahalf))/0.0082));
  dydt[20]   = (RyRainfss- y[20])/RyRtauadapt;
    
  RyRoinfss   = (1 - 1/(1 +  exp((1000*y[2]-(y[20]+ RyRohalf))/0.003)));
  if (RyRoinfss>= y[21])
    RyRtauact = 18.75e-3;       //s
  else
    RyRtauact = 0.1*18.75e-3;   //s
    
  dydt[21]    = (RyRoinfss- y[21])/(RyRtauact);
    
  RyRcinfss   = (1/(1 + exp((1000*y[2]-(y[20]+RyRchalf))/0.001)));
  if (RyRcinfss>= y[22])
    RyRtauinact = 2*87.5e-3;    //s
  else
    RyRtauinact = 87.5e-3;      //s
    
  dydt[22]    = (RyRcinfss- y[22])/(RyRtauinact);
    
    
    
    
  //// Ca2+ buffering
  Cai_bufc    = 1.0/(1.0+Buf_C*Kbuf_C/pow((float)(y[2]+Kbuf_C), 2.0f));
  Ca_SR_bufSR = 1.0/(1.0+Buf_SR*Kbuf_SR/pow((float)(y[1]+Kbuf_SR), 2.0f));
    
  //// Ionic concentrations
  //Nai
  dydt[17]   = -Cm*(i_Na+i_NaL+i_b_Na+3.0*i_NaK+3.0*i_NaCa+i_fNa)/(F*Vc*1.0e-18);
  //Cai
  dydt[2]    = Cai_bufc*(i_leak-i_up+i_rel-(i_CaL+i_b_Ca+i_PCa-2.0*i_NaCa)*Cm/(2.0*Vc*F*1.0e-18));
  //caSR
  dydt[1]    = Ca_SR_bufSR*Vc/V_SR*(i_up-(i_rel+i_leak));
    
  //// Stimulation
  //  PDEFIELD_TYPE i_stim_Amplitude 		= 5.5e-10;//7.5e-10;   // ampere (in stim_mode)
  //  PDEFIELD_TYPE i_stim_End 				= 1000.0;   // second (in stim_mode)
  //  PDEFIELD_TYPE i_stim_PulseDuration	= 0.005;   // second (in stim_mode)
  //  PDEFIELD_TYPE i_stim_Start 			= 0.0;   // second (in stim_mode)
  //  PDEFIELD_TYPE i_stim_frequency        = 60.0;   // per_second (in stim_mode)
  //PDEFIELD_TYPE stim_flag 				= stimFlag;   // dimensionless (in stim_mode)
  //  PDEFIELD_TYPE i_stim_Period 			= 60.0/i_stim_frequency;
    
  //if stim_flag~=0 && stim_flag~=1
  //error('Paci2020: wrong pacing! stimFlag can be only 0 (spontaneous) or 1 (paced)');
  //end
    
  /*
  if ((time >= i_stim_Start) && (time <= i_stim_End) && (time-i_stim_Start-floor((time-i_stim_Start)/i_stim_Period)*i_stim_Period <= i_stim_PulseDuration))
    i_stim = stim_flag*i_stim_Amplitude/Cm;
  else
    i_stim = 0.0;
  */
    
  //// Membrane potential
  dydt[0] = -(i_K1+i_to+i_Kr+i_Ks+i_CaL+i_NaK+i_Na+i_NaL+i_NaCa+i_PCa+i_f+i_b_Na+i_b_Ca-i_stim);
  //printf("dydt[0] = %.9f\n", dydt[0]);

  //Modification of activation by Martijn de Jong, 02-03-2022
  if (celltype2){
    if (fmod(current_time,pacing_interval) <= pacing_duration/10)
      dydt[0] += fmod(current_time,pacing_interval)/(pacing_duration/10)*pacing_strength/Cm;
    else if (fmod(current_time,pacing_interval) > pacing_duration/10 && fmod(current_time,pacing_interval) <= 9*pacing_duration/10)
      dydt[0] += pacing_strength/Cm;
    else if (fmod(current_time,pacing_interval) > 9*pacing_duration/10 && fmod(current_time,pacing_interval) <= pacing_duration)
        dydt[0] += (1-(fmod(current_time,pacing_duration/10)-9*pacing_duration/10)/(pacing_duration/10))*pacing_strength/Cm;
  }  
  
}


__device__ void derivsFitzHughNagumo(PDEFIELD_TYPE current_time, PDEFIELD_TYPE* y, PDEFIELD_TYPE* dydt, bool celltype2, PDEFIELD_TYPE pacing_interval, PDEFIELD_TYPE pacing_duration, PDEFIELD_TYPE pacing_strength, int id, PDEFIELD_TYPE interval_beats, PDEFIELD_TYPE pulse_duration, PDEFIELD_TYPE pulse_strength,  PDEFIELD_TYPE a, PDEFIELD_TYPE b, PDEFIELD_TYPE tau){
  PDEFIELD_TYPE RIext = 0;
  if (fmod(current_time, interval_beats) < pulse_duration && celltype2)
    RIext = pulse_strength;

  dydt[0] = y[0] - pow(y[0],3)/3 - y[1] + RIext;
  dydt[1] = y[0]/tau + a/tau - b*y[1] / tau; 
}


__device__ void RungeKuttaStep(PDEFIELD_TYPE* y, PDEFIELD_TYPE *dydt, int layers, PDEFIELD_TYPE thetime, PDEFIELD_TYPE stepsize, PDEFIELD_TYPE* yout, PDEFIELD_TYPE *yerr, bool celltype2, PDEFIELD_TYPE pacing_interval, PDEFIELD_TYPE pacing_duration, PDEFIELD_TYPE pacing_strength, int id){/*
  //Given values for n variables y[1..n] and their derivatives dydx[1..n] known at x, use
  //the fifth-order Dormand-Prince Runge-Kutta method to advance the solution over an interval h
  //and return the incremented variables as yout[1..n]. Also return an estimate of the local
  //truncation error in yout using the embedded fourth-order method. The user supplies the routine
  //derivs(t,y,dydt,celltype2,pacing_interval,pacing_duration,,acing_strength), which returns derivatives dydt at t.
  int i;
  static PDEFIELD_TYPE 
  c2=0.2,c3=0.3,c4=0.8,c5=8.0/9.0,a21=0.2,a31=3.0/40.0,
  a32=9.0/40.0,a41=44.0/45.0,a42=-56.0/15.0,a43=32.0/9.0,a51=19372.0/6561.0,
  a52=-25360.0/2187.0,a53=64448.0/6561.0,a54=-212.0/729.0,a61=9017.0/3168.0,
  a62=-355.0/33.0,a63=46732.0/5247.0,a64=49.0/176.0,a65=-5103.0/18656.0,
  a71=35.0/384.0,a73=500.0/1113.0,a74=125.0/192.0,a75=-2187.0/6784.0,
  a76=11.0/84.0,e1=71.0/57600.0,e3=-71.0/16695.0,e4=71.0/1920.0,
  e5=-17253.0/339200.0,e6=22.0/525.0,e7=-1.0/40.0;


  PDEFIELD_TYPE k2[ARRAY_SIZE];
  PDEFIELD_TYPE k3[ARRAY_SIZE];
  PDEFIELD_TYPE k4[ARRAY_SIZE];
  PDEFIELD_TYPE k5[ARRAY_SIZE];
  PDEFIELD_TYPE k6[ARRAY_SIZE];
  PDEFIELD_TYPE ytemp[ARRAY_SIZE];
  PDEFIELD_TYPE dydtnew[ARRAY_SIZE];
  for (i=0;i<layers;i++) //First step.
    ytemp[i]=y[i]+a21*stepsize*dydt[i];

  derivsFitzHughNagumo(thetime+c2*stepsize,ytemp,k2,celltype2,pacing_interval,pacing_duration,pacing_strength,id);// Second step.
  for (i=0;i<layers;i++)
    ytemp[i]=y[i]+stepsize*(a31*dydt[i]+a32*k2[i]);
  derivsFitzHughNagumo(thetime+c3*stepsize,ytemp,k3,celltype2,pacing_interval,pacing_duration,pacing_strength,id); //Third step.
  for (i=0;i<layers;i++)
    ytemp[i]=y[i]+stepsize*(a41*dydt[i]+a42*k2[i]+a43*k3[i]);
  derivsFitzHughNagumo(thetime+c4*stepsize,ytemp,k4,celltype2,pacing_interval,pacing_duration,pacing_strength,id); //Fourth step.
  for (i=0;i<layers;i++)
    ytemp[i]=y[i]+stepsize*(a51*dydt[i]+a52*k2[i]+a53*k3[i]+a54*k4[i]);
  derivsFitzHughNagumo(thetime+c5*stepsize,ytemp,k5,celltype2,pacing_interval,pacing_duration,pacing_strength,id); //Fifth step.
  for (i=0;i<layers;i++)
    ytemp[i]=y[i]+stepsize*(a61*dydt[i]+a62*k2[i]+a63*k3[i]+a64*k4[i]+a65*k5[i]);
  PDEFIELD_TYPE timeplusdt = thetime+stepsize;
  derivsFitzHughNagumo(timeplusdt,ytemp,k6,celltype2,pacing_interval,pacing_duration,pacing_strength,id); //Sixth step.
  for (i=0;i<layers;i++) //Accumulate increments with proper weights.
    yout[i]=y[i]+stepsize*(a71*dydt[i]+a73*k3[i]+a74*k4[i]+a75*k5[i]+a76*k6[i]);
  derivsFitzHughNagumo(timeplusdt,yout,dydtnew,celltype2,pacing_interval,pacing_duration,pacing_strength,id);
  for (i=0;i<layers;i++) //Estimate error as difference between fourth- and fifth-order methods.
    yerr[i]=stepsize*(e1*dydt[i]+e3*k3[i]+e4*k4[i]+e5*k5[i]+e6*k6[i]+e7*dydtnew[i]);
*/}

__device__ void StepsizeControl(PDEFIELD_TYPE* y, PDEFIELD_TYPE* dydt, int layers, PDEFIELD_TYPE *thetime, PDEFIELD_TYPE stepsize_try, PDEFIELD_TYPE eps, PDEFIELD_TYPE* yscal, PDEFIELD_TYPE* stepsize_did, PDEFIELD_TYPE* stepsize_next, PDEFIELD_TYPE dt, PDEFIELD_TYPE stepsize_min, bool overshot, bool celltype2, PDEFIELD_TYPE pacing_interval, PDEFIELD_TYPE pacing_duration, PDEFIELD_TYPE pacing_strength, int id){
  /* Fifth-order Runge-Kutta step with monitoring of local truncation error to ensure accuracy and
  adjust stepsize. Input are the dependent variable vector y[1..n] and its derivative dydx[1..n]
  at the starting value of the independent variable x. Also input are the stepsize to be attempted
  htry, the required accuracy eps, and the vector yscal[1..n] against which the error is
  scaled. On output, y and x are replaced by their new values, hdid is the stepsize that was
  actually accomplished, and hnext is the estimated next stepsize. derivs is the user-supplied
  routine that computes the right-hand side derivatives. */
  int i;
  PDEFIELD_TYPE err,stepsize; //stepsize_temp;
  PDEFIELD_TYPE yerr[ARRAY_SIZE];
  PDEFIELD_TYPE ytemp[ARRAY_SIZE];
  PDEFIELD_TYPE scale,scaling_factor,maxy;;

  const PDEFIELD_TYPE alpha = 0.2;
  const PDEFIELD_TYPE Safety = 0.9;
  const PDEFIELD_TYPE minscale = 0.2;
  const PDEFIELD_TYPE maxscale = 10;
  const PDEFIELD_TYPE rtol = 1e-3;
  const PDEFIELD_TYPE atol = 1e-6;

  stepsize=stepsize_try; // Set stepsize to the initial trial value.
  //stepsize=0.000001;
  for(;;){
    RungeKuttaStep(y,dydt,layers,*thetime,stepsize,ytemp,yerr,celltype2,pacing_interval,pacing_duration,pacing_strength,id); // Take a step.  
    if (id == 2100){
      printf("At index %i, we attempt a step of %.10f nanoseconds and the time is %.9f\n", id, stepsize*1e9, *thetime);
    }

    err=0; //Evaluate accuracy.
    for (i=0;i<layers;i++){
      //compute the total euclidean scaled error
      maxy = fabs(y[i]);
      if (fabs(y[i]) < fabs(ytemp[i]))
        maxy = fabs(ytemp[i]);
      scaling_factor = atol+rtol*maxy;
      err += pow(yerr[i]/scaling_factor,2);
    }
    err = sqrt(err/layers); 
    err /= eps; // Scale relative to required tolerance.
    err = 0;
    if (err <= 1.0){  
      break; //Step succeeded. Compute size of next step.
    }

    scale=fmax(Safety*pow(err,-alpha),minscale);
    stepsize *= scale;
    

  }
  scale=Safety*pow(err,-alpha);
  if (scale<minscale) scale = minscale;
  if (scale>maxscale) scale = maxscale;
  *stepsize_next = stepsize*scale;
  *thetime += (*stepsize_did=stepsize);
  for (i=0;i<layers;i++) {
    y[i]=ytemp[i];
  }  
}

__global__ void ODEstepRKA(PDEFIELD_TYPE dt, PDEFIELD_TYPE thetime, int layers, int sizex, int sizey, PDEFIELD_TYPE* PDEvars, PDEFIELD_TYPE* alt_PDEvars, int* celltype, PDEFIELD_TYPE* next_stepsize, PDEFIELD_TYPE stepsize_min, PDEFIELD_TYPE eps, PDEFIELD_TYPE pacing_interval, PDEFIELD_TYPE pacing_duration, PDEFIELD_TYPE pacing_strength){
  /* Ordinary Differential Equation step Runge Kutta Adaptive
  Fifth-order Runge-Kutta step with monitoring of local truncation error to ensure accuracy and
  adjust stepsize. Input are the dependent variable vector y[1..n] and its derivative dydx[1..n]
  at the starting value of the independent variable x. Also input are the stepsize to be attempted
  htry, the required accuracy eps, and the vector yscal[1..n] against which the error is
  scaled. On output, y and x are replaced by their new values, hdid is the stepsize that was
  actually accomplished, and hnext is the estimated next stepsize. derivs is the user-supplied
  routine that computes the right-hand side derivatives. */
  

  PDEFIELD_TYPE begin_time,stepsize_next,stepsize_did,stepsize, end_time;
  PDEFIELD_TYPE yscal[ARRAY_SIZE];
  PDEFIELD_TYPE y[ARRAY_SIZE];
  PDEFIELD_TYPE dydt[ARRAY_SIZE];
  PDEFIELD_TYPE current_time;
  PDEFIELD_TYPE MaxTimeError = 1e-9;
  PDEFIELD_TYPE stepsize_overshot;
  bool overshot = false;
  bool celltype2 = false;
  int i;

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < sizex*sizey; id += stride){
    if (celltype[id] < 1){
      for (i = 0; i < layers; i++) //fill with current PDE values
        alt_PDEvars[i*sizex*sizey + id]= PDEvars[i*sizex*sizey + id];
    }
    
    else{
      celltype2 = false; 
      if (celltype[id] == 2)
        celltype2 = true;
      begin_time = thetime;
      current_time = thetime;
      end_time = thetime + dt;
      stepsize=next_stepsize[id];
      for (i=0;i<layers;i++) 
        y[i]=PDEvars[i*sizex*sizey + id];
      while(fabs(current_time - begin_time - dt)>MaxTimeError){

        overshot = false;
        //derivsFitzHughNagumo(current_time,y,dydt,celltype2, pacing_interval,pacing_duration,pacing_strength,id);

        if (stepsize+current_time > end_time){
          stepsize_overshot = stepsize; 
          stepsize=end_time - current_time;// If stepsize can overshoot, decrease.
          overshot = true;
        }
        StepsizeControl(y,dydt,layers,&current_time,stepsize,eps,yscal,&stepsize_did,&stepsize_next, dt, stepsize_min, overshot, celltype2, pacing_interval,pacing_duration,pacing_strength,id);
        if (fabs(current_time - begin_time - dt)<MaxTimeError) { //Are we done?
          for (i=0;i<layers;i++) {
            alt_PDEvars[i*sizex*sizey + id]=y[i];
          }
          if(overshot && (stepsize_overshot < dt)) 
            next_stepsize[id] = stepsize_overshot;
          else
            next_stepsize[id] = stepsize_next;
        }
        else 
          stepsize = stepsize_next;
      }
    }
  }
}

__global__ void ODEstepFE(PDEFIELD_TYPE dt, PDEFIELD_TYPE thetime, int layers, int sizex, int sizey, PDEFIELD_TYPE* PDEvars, PDEFIELD_TYPE* alt_PDEvars, int* celltype, PDEFIELD_TYPE* next_stepsize, PDEFIELD_TYPE stepsize_min, PDEFIELD_TYPE eps, PDEFIELD_TYPE pacing_interval, PDEFIELD_TYPE pacing_duration, PDEFIELD_TYPE pacing_strength, PDEFIELD_TYPE interval_beats, PDEFIELD_TYPE pulse_duration, PDEFIELD_TYPE pulse_strength,  PDEFIELD_TYPE a, PDEFIELD_TYPE b, PDEFIELD_TYPE tau){
  /* Ordinary Differential Equation step Runge Kutta Adaptive
  Fifth-order Runge-Kutta step with monitoring of local truncation error to ensure accuracy and
  adjust stepsize. Input are the dependent variable vector y[1..n] and its derivative dydx[1..n]
  at the starting value of the independent variable x. Also input are the stepsize to be attempted
  htry, the required accuracy eps, and the vector yscal[1..n] against which the error is
  scaled. On output, y and x are replaced by their new values, hdid is the stepsize that was
  actually accomplished, and hnext is the estimated next stepsize. derivs is the user-supplied
  routine that computes the right-hand side derivatives. */
  



  PDEFIELD_TYPE dtt = dt;
  PDEFIELD_TYPE begin_time,stepsize_next,stepsize_did,stepsize, end_time;
  PDEFIELD_TYPE yscal[ARRAY_SIZE];
  PDEFIELD_TYPE y[ARRAY_SIZE];
  PDEFIELD_TYPE dydt[ARRAY_SIZE];
  PDEFIELD_TYPE current_time;
  PDEFIELD_TYPE MaxTimeError = 1e-5;
  PDEFIELD_TYPE stepsize_overshot;
  bool overshot = false;
  bool celltype2 = false;
  int i;

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < sizex*sizey; id += stride){
    if (celltype[id] < 1){
      for (i = 0; i < layers; i++) //fill with current PDE values
        alt_PDEvars[i*sizex*sizey + id]= PDEvars[i*sizex*sizey + id];
    }
    
    else{
      celltype2 = false; 
      if (celltype[id] == 2)
        celltype2 = true;
      begin_time = thetime;
      current_time = thetime;
      end_time = thetime + dt;
      stepsize=next_stepsize[id];
      for (i=0;i<layers;i++) 
        y[i]=PDEvars[i*sizex*sizey + id];
      while(current_time - begin_time - dt<0){


        overshot = false;
        derivsFitzHughNagumo(current_time,y,dydt,celltype2, pacing_interval,pacing_duration,pacing_strength, id, interval_beats, pulse_duration, pulse_strength,  a, b, tau);
        if (id == 23885)
          printf("dydt[0] = %.10f, dydt[1] = %.10f, y[0] = %.10f, y[1] = %.10f \n", dydt[0], dydt[1], y[0], y[1]);

        current_time += dtt;
        if (fabs(current_time - begin_time - dt) < MaxTimeError) { //Are we done?
          for (i=0;i<layers;i++) {
            alt_PDEvars[i*sizex*sizey + id]=y[i]+dydt[i]*dtt; 
          }
        }
        else{   
          for (i=0;i<layers;i++) {
            y[i]=y[i]+dydt[i]*dtt;  
          }
        }
      }
    }
  }
}


__global__ void ForwardEulerStep(PDEFIELD_TYPE dt, PDEFIELD_TYPE thetime, int layers, int sizex, int sizey, PDEFIELD_TYPE* PDEvars, PDEFIELD_TYPE* alt_PDEvars, int* celltype){
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int i;

  


  
  PDEFIELD_TYPE dydt[ARRAY_SIZE];
  PDEFIELD_TYPE y[ARRAY_SIZE];

  //Declare variables needed for Paci2020 model and assign the constants

  //// Constants
  PDEFIELD_TYPE F = 96485.3415;     // coulomb_per_mole (in model_parameters)
  PDEFIELD_TYPE R = 8.314472;       // joule_per_mole_kelvin (in model_parameters)
  PDEFIELD_TYPE T = 310.0;          // kelvin (in model_parameters) //37°C

  //// Cell geometry
  PDEFIELD_TYPE V_SR = 583.73;        // micrometre_cube (in model_parameters)
  PDEFIELD_TYPE Vc   = 8800.0;        // micrometre_cube (in model_parameters)
  PDEFIELD_TYPE Cm   = 9.87109e-11;   // farad (in model_parameters)

  //// Extracellular concentrations
  PDEFIELD_TYPE Nao = 151.0; // millimolar (in model_parameters)
  PDEFIELD_TYPE Ko  = 5.4;   // millimolar (in model_parameters)
  PDEFIELD_TYPE Cao = 1.8;   // millimolar (in model_parameters)

  //// Intracellular concentrations
  // Naio = 10 mM y[17]
  PDEFIELD_TYPE Ki = 150.0;   // millimolar (in model_parameters)
  // Cai  = 0.0002 mM y[2]
  // caSR = 0.3 mM y[1]  
    
      //// Nernst potential
  PDEFIELD_TYPE E_Na;
  PDEFIELD_TYPE E_Ca;
  PDEFIELD_TYPE E_K;
  PDEFIELD_TYPE PkNa = 0.03;   // dimensionless (in electric_potentials)
  PDEFIELD_TYPE E_Ks;
    
  //// INa adapted from DOI:10.3389/fphys.2018.00080
  PDEFIELD_TYPE g_Na = 3671.2302; //((time<tDrugApplication)*1+(time >= tDrugApplication)*INaFRedMed)*6447.1896;
  PDEFIELD_TYPE i_Na;
    
  PDEFIELD_TYPE m_inf;
  PDEFIELD_TYPE tau_m;
    
  PDEFIELD_TYPE h_inf;
  PDEFIELD_TYPE tau_h;
    
  PDEFIELD_TYPE j_inf;
  PDEFIELD_TYPE tau_j;
    
    
  //// INaL
  PDEFIELD_TYPE myCoefTauM  = 1;
  PDEFIELD_TYPE tauINaL = 200; //ms
  PDEFIELD_TYPE GNaLmax = 17.25;//((time<tDrugApplication)*1+(time >= tDrugApplication)*INaLRedMed)* 2.3*7.5; //(S/F)
  PDEFIELD_TYPE Vh_hLate = 87.61;
  PDEFIELD_TYPE i_NaL;
    
  PDEFIELD_TYPE m_inf_L;
  PDEFIELD_TYPE alpha_m_L;
  PDEFIELD_TYPE beta_m_L;
  PDEFIELD_TYPE tau_m_L;
    
  PDEFIELD_TYPE h_inf_L;
  PDEFIELD_TYPE tau_h_L = 1 * tauINaL;
    
  //// If adapted from DOI:10.3389/fphys.2018.00080
  PDEFIELD_TYPE g_f = 1; //((time<tDrugApplication)*1+(time >= tDrugApplication)*IfRedMed)*22.2763088;
  PDEFIELD_TYPE fNa = 0.37;
  PDEFIELD_TYPE fK = 1 - fNa;
  PDEFIELD_TYPE i_fK;
  PDEFIELD_TYPE i_fNa;
  PDEFIELD_TYPE i_f;
    
  PDEFIELD_TYPE Xf_infinity;
  PDEFIELD_TYPE tau_Xf; 
    
      //// ICaL
  PDEFIELD_TYPE g_CaL = 8.635702e-5;   // metre_cube_per_F_per_s (in i_CaL)
  PDEFIELD_TYPE i_CaL;  
  PDEFIELD_TYPE precision = 0.0001;     
    
  PDEFIELD_TYPE d_infinity;
  PDEFIELD_TYPE alpha_d;
  PDEFIELD_TYPE beta_d;
  PDEFIELD_TYPE gamma_d;
  PDEFIELD_TYPE tau_d;
    
  PDEFIELD_TYPE f1_inf;
  PDEFIELD_TYPE constf1;
    
  PDEFIELD_TYPE tau_f1;
    
  PDEFIELD_TYPE f2_inf;
  PDEFIELD_TYPE constf2 = 1.0;
  PDEFIELD_TYPE tau_f2;
    
  PDEFIELD_TYPE alpha_fCa;
  PDEFIELD_TYPE beta_fCa;
  PDEFIELD_TYPE gamma_fCa;
  PDEFIELD_TYPE fCa_inf;
    
  PDEFIELD_TYPE constfCa;
    
  PDEFIELD_TYPE tau_fCa     = 0.002;   // second (in i_CaL_fCa_gate)
    
  //// Ito
  PDEFIELD_TYPE g_to = 29.9038;   // S_per_F (in i_to)  
  PDEFIELD_TYPE i_to;
    
  PDEFIELD_TYPE q_inf;
  PDEFIELD_TYPE tau_q;
    
    
  PDEFIELD_TYPE r_inf;
  PDEFIELD_TYPE tau_r;
    
  //// IKs
  PDEFIELD_TYPE g_Ks = 2.041;   // S_per_F (in i_Ks)
  PDEFIELD_TYPE i_Ks; // ((time<tDrugApplication)*1+(time >= tDrugApplication)*IKsRedMed)*g_Ks*(y[0]-E_Ks)*pow((float)y[10],2.0)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/y[2]),1.4f)));
    
  PDEFIELD_TYPE Xs_infinity;
  PDEFIELD_TYPE alpha_Xs;
  PDEFIELD_TYPE beta_Xs;
  PDEFIELD_TYPE tau_Xs;
    
  //// IKr
  PDEFIELD_TYPE L0 = 0.025;   // dimensionless (in i_Kr_Xr1_gate)
  PDEFIELD_TYPE Q = 2.3;     // dimensionless (in i_Kr_Xr1_gate)
  PDEFIELD_TYPE g_Kr = 29.8667;   // S_per_F (in i_Kr)
  PDEFIELD_TYPE i_Kr;
    
  PDEFIELD_TYPE V_half;
    
  PDEFIELD_TYPE Xr1_inf;
  PDEFIELD_TYPE alpha_Xr1;
  PDEFIELD_TYPE beta_Xr1;
  PDEFIELD_TYPE tau_Xr1;
    
  PDEFIELD_TYPE Xr2_infinity;
  PDEFIELD_TYPE alpha_Xr2;
  PDEFIELD_TYPE beta_Xr2;
  PDEFIELD_TYPE tau_Xr2;
    
  //// IK1
  PDEFIELD_TYPE alpha_K1;
  PDEFIELD_TYPE beta_K1;
  PDEFIELD_TYPE XK1_inf;
  PDEFIELD_TYPE g_K1 = 28.1492;   // S_per_F (in i_K1)
  PDEFIELD_TYPE i_K1;
    
  //// INaCa
  PDEFIELD_TYPE KmCa = 1.38;   // millimolar (in i_NaCa)
  PDEFIELD_TYPE KmNai = 87.5;   // millimolar (in i_NaCa)
  PDEFIELD_TYPE Ksat = 0.1;    // dimensionless (in i_NaCa)
  PDEFIELD_TYPE gamma = 0.35;   // dimensionless (in i_NaCa)
  PDEFIELD_TYPE alpha = 2.16659;
  PDEFIELD_TYPE kNaCa = 3917.0463; //((time<tDrugApplication)*1+(time >= tDrugApplication)*INaCaRedMed) * 6514.47574;   // A_per_F (in i_NaCa)
  PDEFIELD_TYPE i_NaCa;

  //// INaK
  PDEFIELD_TYPE Km_K = 1.0;    // millimolar (in i_NaK)
  PDEFIELD_TYPE Km_Na = 40.0;   // millimolar (in i_NaK)
  PDEFIELD_TYPE PNaK = 2.74240;// A_per_F (in i_NaK)
  PDEFIELD_TYPE i_NaK;
    
  //// IpCa
  PDEFIELD_TYPE KPCa = 0.0005;   // millimolar (in i_PCa)
  PDEFIELD_TYPE g_PCa = 0.4125;   // A_per_F (in i_PCa)
  PDEFIELD_TYPE i_PCa;
    
  //// Background currents
  PDEFIELD_TYPE g_b_Na = 1.14;         // S_per_F (in i_b_Na)
  PDEFIELD_TYPE i_b_Na;
    
  PDEFIELD_TYPE g_b_Ca = 0.8727264;    // S_per_F (in i_b_Ca)
  PDEFIELD_TYPE i_b_Ca;

  PDEFIELD_TYPE i_up;
  PDEFIELD_TYPE i_leak;
    
  //// Sarcoplasmic reticulum
  PDEFIELD_TYPE VmaxUp = 0.82205;
  PDEFIELD_TYPE Kup	= 4.40435e-4;
    
  PDEFIELD_TYPE V_leak = 4.48209e-4;
    
  // RyR
  PDEFIELD_TYPE g_irel_max = 55.808061;
  PDEFIELD_TYPE RyRa1 = 0.05169;
  PDEFIELD_TYPE RyRa2 = 0.050001;
  PDEFIELD_TYPE RyRahalf = 0.02632;
  PDEFIELD_TYPE RyRohalf = 0.00944;
  PDEFIELD_TYPE RyRchalf = 0.00167;
    
  PDEFIELD_TYPE RyRSRCass;
  PDEFIELD_TYPE i_rel;
    
  PDEFIELD_TYPE RyRainfss;
  PDEFIELD_TYPE RyRtauadapt = 1; //s
    
  PDEFIELD_TYPE RyRoinfss;
  PDEFIELD_TYPE RyRtauact;
    
  PDEFIELD_TYPE RyRcinfss;
  PDEFIELD_TYPE RyRtauinact;

  //// Ca2+ buffering
  PDEFIELD_TYPE Buf_C = 0.25;   // millimolar (in calcium_dynamics)
  PDEFIELD_TYPE Buf_SR = 10.0;   // millimolar (in calcium_dynamics)
  PDEFIELD_TYPE Kbuf_C = 0.001;   // millimolar (in calcium_dynamics)
  PDEFIELD_TYPE Kbuf_SR = 0.3;   // millimolar (in calcium_dynamics)
  PDEFIELD_TYPE Cai_bufc;
  PDEFIELD_TYPE Ca_SR_bufSR;
    
      //// Stimulation
    //  PDEFIELD_TYPE i_stim_Amplitude 		= 5.5e-10;//7.5e-10;   // ampere (in stim_mode)
    //  PDEFIELD_TYPE i_stim_End 				= 1000.0;   // second (in stim_mode)
    //  PDEFIELD_TYPE i_stim_PulseDuration	= 0.005;   // second (in stim_mode)
    //  PDEFIELD_TYPE i_stim_Start 			= 0.0;   // second (in stim_mode)
    //  PDEFIELD_TYPE i_stim_frequency        = 60.0;   // per_second (in stim_mode)
      //PDEFIELD_TYPE stim_flag 				= stimFlag;   // dimensionless (in stim_mode)
    //  PDEFIELD_TYPE i_stim_Period 			= 60.0/i_stim_frequency;
    
      //if stim_flag~=0 && stim_flag~=1
      //error('Paci2020: wrong pacing! stimFlag can be only 0 (spontaneous) or 1 (paced)');
      //end
    
      /*
      if ((time >= i_stim_Start) && (time <= i_stim_End) && (time-i_stim_Start-floor((time-i_stim_Start)/i_stim_Period)*i_stim_Period <= i_stim_PulseDuration))
          i_stim = stim_flag*i_stim_Amplitude/Cm;
      else
          i_stim = 0.0;
      */
  PDEFIELD_TYPE i_stim = 0;
  
  for (int id = index; id < sizex*sizey; id += stride){
    if (celltype[id] < 1){
      for (int l = 0; l < layers; l++) //fill with current PDE values
        alt_PDEvars[l*sizex*sizey + id]= PDEvars[l*sizex*sizey + id];
    }
    else{   
      for (int l = 0; l < layers; l++) //fill with current PDE values
        y[l] = PDEvars[l*sizex*sizey + id];
      //-----FIRST STEP ------------------------------------------------------------------------------------------------------------------------------------------------------------------  
      
        //// Nernst potential
      E_Na = R*T/F*log(Nao/y[17]);
      E_Ca = 0.5*R*T/F*log(Cao/y[2]);
      E_K  = R*T/F*log(Ko/Ki);
      E_Ks = R*T/F*log((Ko+PkNa*Nao)/(Ki+PkNa*y[17]));
    
      //// INa adapted from DOI:10.3389/fphys.2018.00080
      i_Na        =  g_Na*pow((float)y[13],3.0f)*y[11]*y[12]*(y[0] - E_Na);
    
      m_inf       = 1 / (1 + exp((y[0]*1000 + 39)/-11.2));
      tau_m       = (0.00001 + 0.00013*exp(-pow((float)((y[0]*1000 + 48)/15),2.0f)) + 0.000045 / (1 + exp((y[0]*1000 + 42)/-5)));
      dydt[13]   = (m_inf-y[13])/tau_m;
    
      h_inf       = 1 / (1 + exp((y[0]*1000 + 66.5)/6.8));
      tau_h       = (0.00007 + 0.034 / (1 + exp((y[0]*1000 + 41)/5.5) + exp(-(y[0]*1000 + 41)/14)) + 0.0002 / (1 + exp(-(y[0]*1000 + 79)/14)));
      dydt[11]   = (h_inf-y[11])/tau_h;
    
      j_inf       = h_inf;
      tau_j       = 10*(0.0007 + 0.15 / (1 + exp((y[0]*1000 + 41)/5.5) + exp(-(y[0]*1000 + 41)/14)) + 0.002 / (1 + exp(-(y[0]*1000 + 79)/14)));
      dydt[12]   = (j_inf-y[12])/tau_j;
    
    
      //// INaL
      tauINaL     = 200; //ms
      GNaLmax     = 17.25;//((time<tDrugApplication)*1+(time >= tDrugApplication)*INaLRedMed)* 2.3*7.5; //(S/F)
      Vh_hLate    = 87.61;
      i_NaL       = GNaLmax* pow((float)y[18],3.0f)*y[19]*(y[0]-E_Na);
    
      m_inf_L     = 1/(1+exp(-(y[0]*1000+42.85)/(5.264)));
      alpha_m_L   = 1/(1+exp((-60-y[0]*1000)/5));
      beta_m_L    = 0.1/(1+exp((y[0]*1000+35)/5))+0.1/(1+exp((y[0]*1000-50)/200));
      tau_m_L     = 1 * myCoefTauM*alpha_m_L*beta_m_L;
      dydt[18]   = (m_inf_L-y[18])/tau_m_L*1000;
    
      h_inf_L     = 1/(1+exp((y[0]*1000+Vh_hLate)/(7.488)));
      tau_h_L     = 1 * tauINaL;
      dydt[19]   = (h_inf_L-y[19])/tau_h_L*1000;
    
      //// If adapted from DOI:10.3389/fphys.2018.00080
      i_fK        = fK*g_f*y[14]*(y[0] - E_K);
      i_fNa       = fNa*g_f*y[14]*(y[0] - E_Na);
      i_f         = i_fK + i_fNa;
    
      Xf_infinity = 1.0/(1.0 + exp((y[0]*1000 + 69)/8));
      tau_Xf      = 5600 / (1 + exp((y[0]*1000 + 65)/7) + exp(-(y[0]*1000 + 65)/19));
      dydt[14]   = 1000*(Xf_infinity-y[14])/tau_Xf;
      
    
      //// ICaL
      //Prevent division by 0
      if(y[0]< precision && y[0] > -precision) //hopital
        i_CaL =  g_CaL*4.0*y[0]*pow((float)F,2.0f)/(R*T) *y[4]*y[5]*y[6]*y[7] / (2.0*F/(R*T)) * (y[2] - 0.341*Cao);
      else
        i_CaL = g_CaL*4.0*y[0]*pow((float)F,2.0f)/(R*T)*(y[2]*exp(2.0*y[0]*F/(R*T))-0.341*Cao)/(exp(2.0*y[0]*F/(R*T))-1.0)*y[4]*y[5]*y[6]*y[7]; //((time<tDrugApplication)*1+(time >= tDrugApplication)*ICaLRedMed)*g_CaL*4.0*y[0]*pow(F,2.0)/(R*T)*(y[2]*exp(2.0*y[0]*F/(R*T))-0.341*Cao)/(exp(2.0*y[0]*F/(R*T))-1.0)*y[4]*y[5]*y[6]*y[7];
    
      d_infinity  = 1.0/(1.0+exp(-(y[0]*1000.0+9.1)/7.0));
      alpha_d     = 0.25+1.4/(1.0+exp((-y[0]*1000.0-35.0)/13.0));
      beta_d      = 1.4/(1.0+exp((y[0]*1000.0+5.0)/5.0));
      gamma_d     = 1.0/(1.0+exp((-y[0]*1000.0+50.0)/20.0));
      tau_d       = (alpha_d*beta_d+gamma_d)*1.0/1000.0;
      dydt[4]    = (d_infinity-y[4])/tau_d;
    
      f1_inf      = 1.0/(1.0+exp((y[0]*1000.0+26.0)/3.0));
      if (f1_inf-y[5] > 0.0)
          constf1 = 1.0+1433.0*(y[2]-50.0*1.0e-6);
      else
          constf1 = 1.0;
    
      tau_f1      = (20.0+1102.5*exp(-pow((float)((y[0]*1000.0+27.0)/15.0),2.0f))+200.0/(1.0+exp((13.0-y[0]*1000.0)/10.0))+180.0/(1.0+exp((30.0+y[0]*1000.0)/10.0)))*constf1/1000.0;
      dydt[5]    = (f1_inf-y[5])/tau_f1;
    
      f2_inf      = 0.33+0.67/(1.0+exp((y[0]*1000.0+32.0)/4.0));
      tau_f2      = (600.0*exp(-pow((float)(y[0]*1000.0+25.0),2.0f)/170.0)+31.0/(1.0+exp((25.0-y[0]*1000.0)/10.0))+16.0/(1.0+exp((30.0+y[0]*1000.0)/10.0)))*constf2/1000.0;
      dydt[6]    = (f2_inf-y[6])/tau_f2;
    
      alpha_fCa   = 1.0/(1.0+pow((float)(y[2]/0.0006),8.0f));
      beta_fCa    = 0.1/(1.0+exp((y[2]-0.0009)/0.0001));
      gamma_fCa   = 0.3/(1.0+exp((y[2]-0.00075)/0.0008));
      fCa_inf     = (alpha_fCa+beta_fCa+gamma_fCa)/1.3156;
    
      if ((y[0] > -0.06) && (fCa_inf > y[7]))
          constfCa = 0.0;
      else
          constfCa = 1.0;
    
      dydt[7]    = constfCa*(fCa_inf-y[7])/tau_fCa;
    
      //// Ito
      i_to        = g_to*(y[0]-E_K)*y[15]*y[16];
    
      q_inf       = 1.0/(1.0+exp((y[0]*1000.0+53.0)/13.0));
      tau_q       = (6.06+39.102/(0.57*exp(-0.08*(y[0]*1000.0+44.0))+0.065*exp(0.1*(y[0]*1000.0+45.93))))/1000.0;
      dydt[15]   = (q_inf-y[15])/tau_q;
    
    
      r_inf       = 1.0/(1.0+exp(-(y[0]*1000.0-22.3)/18.75));
      tau_r       = (2.75352+14.40516/(1.037*exp(0.09*(y[0]*1000.0+30.61))+0.369*exp(-0.12*(y[0]*1000.0+23.84))))/1000.0;
      dydt[16]   = (r_inf-y[16])/tau_r;
    
      //// IKs
      i_Ks        = g_Ks*(y[0]-E_Ks)*pow((float)y[10],2.0f)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/y[2]),1.4f))); // ((time<tDrugApplication)*1+(time >= tDrugApplication)*IKsRedMed)*g_Ks*(y[0]-E_Ks)*pow((float)y[10],2.0)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/y[2]),1.4f)));
    
      Xs_infinity = 1.0/(1.0+exp((-y[0]*1000.0-20.0)/16.0));
      alpha_Xs    = 1100.0/sqrt(1.0+exp((-10.0-y[0]*1000.0)/6.0));
      beta_Xs     = 1.0/(1.0+exp((-60.0+y[0]*1000.0)/20.0));
      tau_Xs      = 1.0*alpha_Xs*beta_Xs/1000.0;
      dydt[10]   = (Xs_infinity-y[10])/tau_Xs;
    
      //// IKr
      i_Kr         = g_Kr*(y[0]-E_K)*y[8]*y[9]*sqrt(Ko/5.4); //((time<tDrugApplication)*1+(time >= tDrugApplication)*IKrRedMed)*g_Kr*(y[0]-E_K)*y[8]*y[9]*sqrt(Ko/5.4);
    
      V_half       = 1000.0*(-R*T/(F*Q)*log(pow((float)(1.0+Cao/2.6),4.0f)/(pow((float)(1.0+Cao/0.58),4.0f)*L0))-0.019);
    
      Xr1_inf      = 1.0/(1.0+exp((V_half-y[0]*1000.0)/4.9));
      alpha_Xr1    = 450.0/(1.0+exp((-45.0-y[0]*1000.0)/10.0));
      beta_Xr1     = 6.0/(1.0+exp((30.0+y[0]*1000.0)/11.5));
      tau_Xr1      = 1.0*alpha_Xr1*beta_Xr1/1000.0;
      dydt[8]     = (Xr1_inf-y[8])/tau_Xr1;
    
      Xr2_infinity = 1.0/(1.0+exp((y[0]*1000.0+88.0)/50.0));
      alpha_Xr2    = 3.0/(1.0+exp((-60.0-y[0]*1000.0)/20.0));
      beta_Xr2     = 1.12/(1.0+exp((-60.0+y[0]*1000.0)/20.0));
      tau_Xr2      = 1.0*alpha_Xr2*beta_Xr2/1000.0;
      dydt[9]    = (Xr2_infinity-y[9])/tau_Xr2;
    
      //// IK1
      alpha_K1    = 3.91/(1.0+exp(0.5942*(y[0]*1000.0-E_K*1000.0-200.0)));
      beta_K1     = (-1.509*exp(0.0002*(y[0]*1000.0-E_K*1000.0+100.0))+exp(0.5886*(y[0]*1000.0-E_K*1000.0-10.0)))/(1.0+exp(0.4547*(y[0]*1000.0-E_K*1000.0)));
      XK1_inf     = alpha_K1/(alpha_K1+beta_K1);
      i_K1        = g_K1*XK1_inf*(y[0]-E_K)*sqrt(Ko/5.4);
    
      //// INaCa
      i_NaCa      = kNaCa*(exp(gamma*y[0]*F/(R*T))*pow((float)y[17],3.0f)*Cao-exp((gamma-1.0)*y[0]*F/(R*T))*pow((float)Nao,3.0f)*y[2]*alpha)/((pow((float)KmNai,3.0f)+pow((float)Nao,3.0f))*(KmCa+Cao)*(1.0+Ksat*exp((gamma-1.0)*y[0]*F/(R*T))));
    
      //// INaK
      i_NaK       = PNaK*Ko/(Ko+Km_K)*y[17]/(y[17]+Km_Na)/(1.0+0.1245*exp(-0.1*y[0]*F/(R*T))+0.0353*exp(-y[0]*F/(R*T)));
    
      //// IpCa
      i_PCa       = g_PCa*y[2]/(y[2]+KPCa);
    
      //// Background currents
      i_b_Na      = g_b_Na*(y[0]-E_Na);
    
      i_b_Ca      = g_b_Ca*(y[0]-E_Ca);
    
      //// Sarcoplasmic reticulum
      i_up        = VmaxUp/(1.0+pow((float)Kup,2.0f)/pow((float)y[2],2.0f));
    
      i_leak      = (y[1]-y[2])*V_leak;
    
      dydt[3]    = 0;
    
      // RyR
    
      RyRSRCass   = (1 - 1/(1 +  exp((y[1]-0.3)/0.1)));
      i_rel       = g_irel_max*RyRSRCass*y[21]*y[22]*(y[1]-y[2]);
    
      RyRainfss   = RyRa1-RyRa2/(1 + exp((1000*y[2]-(RyRahalf))/0.0082));
      dydt[20]   = (RyRainfss- y[20])/RyRtauadapt;
    
      RyRoinfss   = (1 - 1/(1 +  exp((1000*y[2]-(y[20]+ RyRohalf))/0.003)));
      if (RyRoinfss>= y[21])
        RyRtauact = 18.75e-3;       //s
      else
        RyRtauact = 0.1*18.75e-3;   //s
    
      dydt[21]    = (RyRoinfss- y[21])/(RyRtauact);
    
      RyRcinfss   = (1/(1 + exp((1000*y[2]-(y[20]+RyRchalf))/0.001)));
      if (RyRcinfss>= y[22])
        RyRtauinact = 2*87.5e-3;    //s
      else
        RyRtauinact = 87.5e-3;      //s
    
      dydt[22]    = (RyRcinfss- y[22])/(RyRtauinact);
    
    
    
    
      //// Ca2+ buffering
      Cai_bufc    = 1.0/(1.0+Buf_C*Kbuf_C/pow((float)(y[2]+Kbuf_C), 2.0f));
      Ca_SR_bufSR = 1.0/(1.0+Buf_SR*Kbuf_SR/pow((float)(y[1]+Kbuf_SR), 2.0f));
    
      //// Ionic concentrations
      //Nai
      dydt[17]   = -Cm*(i_Na+i_NaL+i_b_Na+3.0*i_NaK+3.0*i_NaCa+i_fNa)/(F*Vc*1.0e-18);
      //Cai
      dydt[2]    = Cai_bufc*(i_leak-i_up+i_rel-(i_CaL+i_b_Ca+i_PCa-2.0*i_NaCa)*Cm/(2.0*Vc*F*1.0e-18));
      //caSR
      dydt[1]    = Ca_SR_bufSR*Vc/V_SR*(i_up-(i_rel+i_leak));
    
      //// Stimulation
    //  PDEFIELD_TYPE i_stim_Amplitude 		= 5.5e-10;//7.5e-10;   // ampere (in stim_mode)
    //  PDEFIELD_TYPE i_stim_End 				= 1000.0;   // second (in stim_mode)
    //  PDEFIELD_TYPE i_stim_PulseDuration	= 0.005;   // second (in stim_mode)
    //  PDEFIELD_TYPE i_stim_Start 			= 0.0;   // second (in stim_mode)
    //  PDEFIELD_TYPE i_stim_frequency        = 60.0;   // per_second (in stim_mode)
      //PDEFIELD_TYPE stim_flag 				= stimFlag;   // dimensionless (in stim_mode)
    //  PDEFIELD_TYPE i_stim_Period 			= 60.0/i_stim_frequency;
    
      //if stim_flag~=0 && stim_flag~=1
      //error('Paci2020: wrong pacing! stimFlag can be only 0 (spontaneous) or 1 (paced)');
      //end
    
      /*
      if ((time >= i_stim_Start) && (time <= i_stim_End) && (time-i_stim_Start-floor((time-i_stim_Start)/i_stim_Period)*i_stim_Period <= i_stim_PulseDuration))
          i_stim = stim_flag*i_stim_Amplitude/Cm;
      else
          i_stim = 0.0;
      */
    
      //// Membrane potential
      dydt[0] = -(i_K1+i_to+i_Kr+i_Ks+i_CaL+i_NaK+i_Na+i_NaL+i_NaCa+i_PCa+i_f+i_b_Na+i_b_Ca-i_stim);

      //-----WRITE NEW VALUES ------------------------------------------------------------------------------------------------------------------------------------------------------------------      
      #pragma unroll
      for (i=0;i<layers;i++) //Accumulate increments with proper weights.
        alt_PDEvars[i*sizex*sizey + id]=y[i]+dydt[i]*dt;
    } 

  }
}




__global__ void CopyAltToOriginalPDEvars(int sizex, int sizey, int layers, PDEFIELD_TYPE* PDEvars, PDEFIELD_TYPE* alt_PDEvars){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < layers*sizex*sizey; id += stride){
    PDEvars[id] = alt_PDEvars[id]; 
  }
}

__global__ void CopyOriginalToAltPDEvars(int sizex, int sizey, int layers, PDEFIELD_TYPE* PDEvars, PDEFIELD_TYPE* alt_PDEvars){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < layers*sizex*sizey; id += stride){
    alt_PDEvars[id] = PDEvars[id]; 
  }
}

__global__ void FlipSigns(int sizex, int sizey, int layers, PDEFIELD_TYPE* PDEvars, PDEFIELD_TYPE* alt_PDEvars){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < layers*sizex*sizey; id += stride){
    alt_PDEvars[id] = -PDEvars[id]; 
  }
}


void PDE::cuPDEsteps(CellularPotts * cpm, int repeat){
  //copy current couplingcoefficient matrix and celltype matrix from host to device
  couplingcoefficient = cpm->getCouplingCoefficient();
  //int** cellnumber = cpm -> getSigma(); 
  hipError_t errSync;
  hipError_t errAsync;
  celltype = cpm->getTau();
  hipMemcpy(d_couplingcoefficient, couplingcoefficient[0], sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyHostToDevice);
  hipMemcpy(d_celltype, celltype[0], sizex*sizey*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_PDEvars, PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyHostToDevice);

  //setup matrices for upperdiagonal, diagonal and lower diagonal for both the horizontal and vertical direction, since these remain the same during once MCS
  InitializeDiagonals<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, 2/dt, dx2, lowerH, upperH, diagH, lowerV, upperV, diagV, d_couplingcoefficient);
  hipDeviceSynchronize();
  errSync  = hipGetLastError();
  errAsync = hipDeviceSynchronize();
  if (errSync != hipSuccess) 
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
  if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

  for (int iteration = 0; iteration < repeat; iteration++){
    //Do an ODE step of size dt/2
    ODEstepFE<<<par.number_of_cores, par.threads_per_core>>>(dt/2, thetime, layers, sizex, sizey, d_PDEvars, d_alt_PDEvars, d_celltype, next_stepsize, min_stepsize, par.eps, pacing_interval, par.pacing_duration, par.pacing_strength, par.FHN_interval_beats, par.FHN_pulse_duration, par.FHN_pulse_strength, par.FHN_a, par.FHN_b, par.FHN_tau);
    errSync  = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
      printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
      printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

    hipMemcpy(alt_PDEvars, d_alt_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyDeviceToHost);
    for (int i=0; i < sizex*sizey; i++){
      if (!(alt_PDEvars[i]>-100000 && alt_PDEvars[i] < 100000)){
        cout << "Error at i = " << i << ". Abort.\n";
        exit(1);
      }
    }
    cout << "After first FE step, alt_PDEvars[23885] = " << alt_PDEvars[23885] << endl;


    //Do a vertical ADI sweep of size dt/2
    InitializeVerticalVectors<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, 2/dt, dx2, BV, d_couplingcoefficient, d_alt_PDEvars);
    errSync  = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
      printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
      printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
    statusV = hipsparseSgtsvInterleavedBatch(handleV, 0, sizey, lowerV, diagV, upperV, BV, sizex, pbufferV);
    if (statusV != HIPSPARSE_STATUS_SUCCESS)
    {
      cout << statusV << endl;
    }
    hipDeviceSynchronize();
    NewPDEfieldV0<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, BV, d_PDEvars); //////
    errSync  = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
      printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
      printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
    NewPDEfieldOthers<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, layers, BV, d_PDEvars, d_alt_PDEvars); //////
    errSync  = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
      printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
      printf("Async kernel error: %s\n", hipGetErrorString(errAsync)); 
    

    hipMemcpy(PDEvars, d_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyDeviceToHost);
    cout << "After first ADI step, PDEvars[23885] = " << PDEvars[23885] << endl;

    //increase time by dt/2
    thetime = thetime + dt/2;  
    //Do an ODE step of size dt/2
    ODEstepFE<<<par.number_of_cores, par.threads_per_core>>>(dt/2, thetime, layers, sizex, sizey, d_PDEvars, d_alt_PDEvars, d_celltype, next_stepsize, min_stepsize, par.eps, pacing_interval, par.pacing_duration, par.pacing_strength, par.FHN_interval_beats, par.FHN_pulse_duration, par.FHN_pulse_strength, par.FHN_a, par.FHN_b, par.FHN_tau);
    errSync  = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
      printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
      printf("Async kernel error: %s\n", hipGetErrorString(errAsync));   
      
      
    hipMemcpy(alt_PDEvars, d_alt_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyDeviceToHost);
    cout << "After second FE step, alt_PDEvars[23885] = " << alt_PDEvars[23885] << endl;

    //Do a horizontal ADI sweep of size dt/2
    InitializeHorizontalVectors<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, 2/dt, dx2, BH, d_couplingcoefficient, d_alt_PDEvars);
    errSync  = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
      printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
      printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
    statusH = hipsparseSgtsvInterleavedBatch(handleH, 0, sizex, lowerH, diagH, upperH, BH, sizey, pbufferH);
    if (statusH != HIPSPARSE_STATUS_SUCCESS)
    {
      cout << statusH << endl;
    }
    NewPDEfieldH0<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, BH, d_PDEvars);    
    errSync  = hipGetLastError();
    errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
      printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
      printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
    NewPDEfieldOthers<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, layers, BV, d_PDEvars, d_alt_PDEvars); //////
    errSync  = hipGetLastError();
      errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) 
      printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
      printf("Async kernel error: %s\n", hipGetErrorString(errAsync));  
    
      //increase time by dt/2
    thetime = thetime + dt/2; 


    hipMemcpy(PDEvars, d_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyDeviceToHost);
    cout << "After first ADI step, PDEvars[23885] = " << PDEvars[23885] << endl;

    
    hipMemcpy(PDEvars, d_PDEvars, layers*sizex*sizey*sizeof(PDEFIELD_TYPE), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    ofstream myfile;
    myfile.open ("data.txt", std::ios_base::app);
    myfile << thetime << ",";
    for (int i = 0; i < layers; i++)
      myfile << PDEvars[23885 + i*sizex*sizey] << ",";
    myfile << endl;
    cout << "PDEvars[" << int(sizex/2*sizey + 0.5 * sizey) << "] = " << PDEvars[int(sizex/2*sizey + 0.5 * sizey)] << endl;
    if (!(PDEvars[int(sizex/2*sizey + 0.5 * sizey) ]>-100000 && PDEvars[int(sizex/2*sizey + 0.5 * sizey)] < 100000)){
      cout << "We encountered a NaN error. Abort the program. \n";
      exit(1);
    }
  }
  
}

// public
void PDE::Diffuse(int repeat) {
  
  // Just diffuse everywhere (cells are transparent), using finite difference
  // (We're ignoring the problem of how to cope with moving cell
  // boundaries right now)
  
  const PDEFIELD_TYPE dt=par.dt;
  const PDEFIELD_TYPE dx2=par.dx*par.dx;

  for (int r=0;r<repeat;r++) {
    //NoFluxBoundaries();
    if (par.periodic_boundaries) {
      PeriodicBoundaries();
    } else {
      AbsorbingBoundaries();
      //NoFluxBoundaries();
    }
    for (int l=0;l<layers;l++) {
      for (int x=1;x<sizex-1;x++)
	for (int y=1;y<sizey-1;y++) {
	  PDEFIELD_TYPE sum=0.;
	  sum+=PDEvars[l*sizex*sizey + (x+1)*sizey+y];
	  sum+=PDEvars[l*sizex*sizey + (x-1)*sizey+y];
	  sum+=PDEvars[l*sizex*sizey + x*sizey+y+1];
	  sum+=PDEvars[l*sizex*sizey + x*sizey+y-1];
	  sum-=4*PDEvars[l*sizex*sizey + x*sizey+y];
	  alt_PDEvars[l*sizex*sizey + x*sizey+y]=PDEvars[l*sizex*sizey + x*sizey+y]+sum*dt*par.diff_coeff[l]/dx2;
      }
    }
    PDEFIELD_TYPE *tmp;
    tmp=PDEvars;
    PDEvars=alt_PDEvars;
    alt_PDEvars=tmp;
  
    thetime+=dt;
  }
}

/*double PDE::GetChemAmount(const int layer) {
  // Sum the total amount of chemical in the lattice
  // in layer l
  // (This is useful to check particle conservation)
  double sum=0.;
  if (layer==-1) { // default argument: sum all chemical species
    for (int l=0;l<layers;l++) {
      for (int x=1;x<sizex-1;x++) {
	for (int y=1;y<sizey-1;y++) {
	  sum+=PDEvars[l][x][y];
	}
      }
    }
  } else {
    for (int x=1;x<sizex-1;x++)
      for (int y=1;y<sizey-1;y++) {
	sum+=PDEvars[layer][x][y];
      }
  } 
  return sum;
}*/

// private
void PDE::NoFluxBoundaries(void) {
  // all gradients at the edges become zero, 
  // so nothing flows out
  // Note that four corners points are not defined (0.)
  // but they aren't used in the calculations
  for (int l=0;l<layers;l++) {
    for (int x=0;x<sizex;x++) {
      PDEvars[l*sizex*sizey + x*sizey+0]=PDEvars[l*sizex*sizey + x*sizey+1];
      PDEvars[l*sizex*sizey + x*sizey+sizey-1]=PDEvars[l*sizex*sizey + x*sizey+sizey-2];
    }
    for (int y=0;y<sizey;y++) {
      PDEvars[l*sizex*sizey + 0*sizey+y]=PDEvars[l*sizex*sizey + 1*sizey+y];
      PDEvars[l*sizex*sizey + (sizex-1)*sizey+y]=PDEvars[l*sizex*sizey + (sizex-2)*sizey+y];
    }
  }
}


// private
void PDE::AbsorbingBoundaries(void) {
  // all boundaries are sinks, 
  for (int l=0;l<layers;l++) {
    for (int x=0;x<sizex;x++) {
      PDEvars[l*sizex*sizey + x*sizey+0]=0.;
      PDEvars[l*sizex*sizey + x*sizey+sizey-1]=0.;
    }
    for (int y=0;y<sizey;y++) {
      PDEvars[l*sizex*sizey + 0*sizey+y]=0.;
      PDEvars[l*sizex*sizey + (sizex-1)*sizey+y]=0.;
    }
  }
}

// private
void PDE::PeriodicBoundaries(void) {
  // periodic...
  for (int l=0;l<layers;l++) {
    for (int x=0;x<sizex;x++) {
      PDEvars[l*sizex*sizey + x*sizey]=PDEvars[l*sizex*sizey + x*sizey+sizey-2];
      PDEvars[l*sizex*sizey + x*sizey+sizey-1]=PDEvars[l*sizex*sizey + x+sizey+1];
    }
    for (int y=0;y<sizey;y++) {
      PDEvars[l*sizex*sizey + y]=PDEvars[l*sizex*sizey + (sizex-2)*sizey+y];
      PDEvars[l*sizex*sizey + (sizex-1)+y]=PDEvars[l*sizex*sizey + 1*sizey+y];
    }
  }
}

void PDE::GradC(int layer, int first_grad_layer) {
  // calculate the first and second order gradients and put
  // them in the next chemical fields
  if (par.n_chem<5) {
    throw("PDE::GradC: Not enough chemical fields");
  }

  // GradX
  for (int y=0;y<sizey;y++) {
    for (int x=1;x<sizex-1;x++) {
      PDEvars[first_grad_layer*sizex*sizey + x*sizey+y]=(PDEvars[layer*sizex*sizey+(x+1)*sizey+y]-PDEvars[layer*sizex*sizey + (x-1)*sizey+y])/2.;
    } 
  }
  // GradY
  for (int x=0;x<sizex;x++) {
    for (int y=1;y<sizey-1;y++) {
      PDEvars[(first_grad_layer+1)*sizex*sizey + x*sizey+y]=(PDEvars[layer*sizex*sizey+x*sizey+y+1]-PDEvars[layer*sizex*sizey+x*sizey+y-1])/2.;
    } 
  }
  // GradXX
  for (int y=0;y<sizey;y++) {
    for (int x=1;x<sizex-1;x++) {
      PDEvars[(first_grad_layer+2)*sizex*sizey + x*sizey+y]=PDEvars[layer*sizex*sizey+ (x+1)*sizey+y]-PDEvars[layer*sizex*sizey + (x-1)*sizey+y]-2*PDEvars[layer*sizex*sizey+x*sizey+y];
    } 
  }

  // GradYY
  for (int x=0;x<sizex;x++) {
    for (int y=1;y<sizey-1;y++) {
      PDEvars[(first_grad_layer+3)*sizex*sizey + x*sizey+y]=PDEvars[layer*sizex*sizey + x*sizey-1]-PDEvars[layer*sizex*sizey+x*sizey+y+1]-2*PDEvars[layer*sizex*sizey + x*sizey+y];
    } 
  }
}

void PDE::PlotVectorField(Graphics &g, int stride, int linelength, int first_grad_layer) {
  // Plot vector field assuming it's in layer 1 and 2
  for (int x=1;x<sizex-1;x+=stride) {
    for (int y=1;y<sizey-1;y+=stride) {
      
      // calculate line
      int x1,y1,x2,y2;
      
      x1=(int)(x-linelength*PDEvars[(first_grad_layer)*sizex*sizey + x*sizey + y]);
      y1=(int)(y-linelength*PDEvars[(first_grad_layer+1)*sizex*sizey + x*sizey + y]);
      x2=(int)(x+linelength*PDEvars[(first_grad_layer)*sizex*sizey + x*sizey + y]);
      y2=(int)(y+linelength*PDEvars[(first_grad_layer+1)*sizex*sizey + x*sizey + y]);
      if (x1<0) x1=0;
      if (x1>sizex-1) x1=sizex-1;
      if (y1<0) y1=0;
      if (y1>sizey-1) y1=sizey-1;
      if (x2<0) x2=0;
      if (x2>sizex-1) x2=sizex-1;
      if (y2<0) y2=0;
      if (y2>sizey-1) y2=sizey-1;

      // And draw it :-)
      // perhaps I can add arrowheads later to make it even nicer :-)
      g.Line(x1,y1,x2,y2,1);
    }
  }
}

bool PDE::plotPos(int x, int y, Graphics * graphics, int layer) {
  layer = 0;
  double val = PDEvars[layer*sizex*sizey+x*sizey+y];
  if (val > -100){
    graphics->Rectangle(MapColour(val), x, y);
    return false;
  }
  return true;
}


void PDE::SetSpeciesName(int l, const char *name) {
    species_names[l]=string(name);
}


/*void PDE::InitLinearYGradient(int spec, double conc_top, double conc_bottom) {
    for (int y=0;y<sizey;y++) {
      double val=(double)conc_top+y*((double)(conc_bottom-conc_top)/(double)sizey);
    for (int x=0;x<sizex;x++) {
      PDEvars[spec][x][y]=val;
    }
    cerr << y << " " << val << endl;
  }
}*/
