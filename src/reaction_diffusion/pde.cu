#include "hip/hip_runtime.h"
/*

Copyright 1996-2006 Roeland Merks

This file is part of Tissue Simulation Toolkit.

Tissue Simulation Toolkit is free software; you can redistribute
it and/or modify it under the terms of the GNU General Public
License as published by the Free Software Foundation; either
version 2 of the License, or (at your option) any later version.

Tissue Simulation Toolkit is distributed in the hope that it will
be useful, but WITHOUT ANY WARRANTY; without even the implied
warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with Tissue Simulation Toolkit; if not, write to the Free
Software Foundation, Inc., 51 Franklin St, Fifth Floor, Boston, MA
02110-1301 USA

*/
#include <stdio.h>
#include <math.h>
#include <cstdlib>
#include <fstream>
#include <sstream>
#include <chrono>

#include "crash.hpp"
#include "parameter.hpp"
#include "ca.hpp"
#include "pde.hpp"
#include "conrec.hpp"
#include "graph.hpp"
#include <hipsparse.h>

/* STATIC DATA MEMBER INITIALISATION */
const int PDE::nx[9] = {0, 1, 1, 1, 0,-1,-1,-1, 0 };
const int PDE::ny[9] = {0, 1, 0,-1,-1,-1, 0, 1, 1 };

extern Parameter par;

/** PRIVATE **/

PDE::PDE(const int l, const int sx, const int sy) {
  PDEvars=0;
  thetime=0;
  PDEsteps=0;
  sizex=sx;
  sizey=sy;
  layers=l;
  PDEvars=AllocatePDEvars(l,sx,sy);
  alt_PDEvars=AllocatePDEvars(l,sx,sy);
  btype = 1;
  dx2 = par.dx*par.dx;
  dt = par.dt;
  usePDEorAltPDE = true;
}


PDE::PDE(void) {

  PDEvars=0;
  alt_PDEvars=0;
  sizex=0; sizey=0; layers=0;
  thetime=0;
  if (par.useopencl){this->SetupOpenCL();}
}

// destructor (virtual)
PDE::~PDE(void) {
  if (PDEvars) {
    free(PDEvars[0][0]);
    free(PDEvars[0]);
    free(PDEvars);
    PDEvars=0;
  }
  if (alt_PDEvars) {
    free(alt_PDEvars[0][0]);
    free(alt_PDEvars[0]);
    free(alt_PDEvars);
    alt_PDEvars=0;
  }
  free(couplingcoefficient);
  hipFree(PDEvars);
  hipFree(alt_PDEvars);
  hipFree(couplingcoefficient);
  hipFree(upperH);
  hipFree(diagH);
  hipFree(lowerH);
  hipFree(BH);
  hipFree(XH);
  hipFree(upperV);
  hipFree(diagV);
  hipFree(lowerV);
  hipFree(BV);
}

PDEFIELD_TYPE ***PDE::AllocatePDEvars(const int layers, const int sx, const int sy) {
  PDEFIELD_TYPE ***mem;
  sizex=sx; sizey=sy;
  mem=(PDEFIELD_TYPE ***)malloc(layers*sizeof(PDEFIELD_TYPE **));
  if (mem==NULL) {
    MemoryWarning();
  }
  mem[0]=(PDEFIELD_TYPE **)malloc(layers*sizex*sizeof(PDEFIELD_TYPE *));
  if (mem[0]==NULL) { 
    MemoryWarning();
  }
  for (int i=1;i<layers;i++) {
    mem[i]=mem[i-1]+sizex;
  }  
  mem[0][0]=(PDEFIELD_TYPE *)malloc(layers*sizex*sizey*sizeof(PDEFIELD_TYPE));
  if (mem[0][0]==NULL) {
    MemoryWarning();
  }
  for (int i=1;i<layers*sizex;i++) {
    mem[0][i]=mem[0][i-1]+sizey;
  }
  /* Clear PDE plane */
  for (int i=0;i<layers*sizex*sizey;i++) {
    mem[0][0][i]=0.; 
  }
  return mem;
}

void PDE::AllocateTridiagonalvars(int sx, int sy){
  
  sizex=sx; sizey=sy;


  //Allocate lowerH
  lowerH=(PDEFIELD_TYPE **)malloc(sizey*sizeof(PDEFIELD_TYPE*));
  if (lowerH==NULL)
    MemoryWarning();

  lowerH[0]=(PDEFIELD_TYPE *)malloc(sizey*sizex*sizeof(PDEFIELD_TYPE));
  if (lowerH[0]==NULL)  
    MemoryWarning();

    {for (int i=1;i<sizey;i++) 
    lowerH[i]=lowerH[i-1]+sizex;}
  /* clear matrix */

  {for (int i=0;i<sizey*sizex;i++) 
    lowerH[0][i]=0; }

  //Allocate upperH
  upperH=(PDEFIELD_TYPE **)malloc(sizey*sizeof(PDEFIELD_TYPE*));
  if (upperH==NULL)
    MemoryWarning();

  upperH[0]=(PDEFIELD_TYPE *)malloc(sizey*sizex*sizeof(PDEFIELD_TYPE));
  if (upperH[0]==NULL)  
    MemoryWarning();

    {for (int i=1;i<sizey;i++) 
    upperH[i]=upperH[i-1]+sizex;}
  /* clear matrix */

  {for (int i=0;i<sizey*sizex;i++) 
    upperH[0][i]=0; }


  //Allocate diagH
  diagH=(PDEFIELD_TYPE **)malloc(sizey*sizeof(PDEFIELD_TYPE*));
  if (diagH==NULL)
    MemoryWarning();

  diagH[0]=(PDEFIELD_TYPE *)malloc(sizey*sizex*sizeof(PDEFIELD_TYPE));
  if (diagH[0]==NULL)  
    MemoryWarning();

    {for (int i=1;i<sizey;i++) 
    diagH[i]=diagH[i-1]+sizex;}
  /* clear matrix */

  {for (int i=0;i<sizey*sizex;i++) 
    diagH[0][i]=0; }


  //Allocate BH
  BH=(PDEFIELD_TYPE **)malloc(sizey*sizeof(PDEFIELD_TYPE*));
  if (BH==NULL)
    MemoryWarning();

  BH[0]=(PDEFIELD_TYPE *)malloc(sizey*sizex*sizeof(PDEFIELD_TYPE));
  if (BH[0]==NULL)  
    MemoryWarning();

    {for (int i=1;i<sizey;i++) 
    BH[i]=BH[i-1]+sizex;}

  /* clear matrix */
  {for (int i=0;i<sizey*sizex;i++) 
    BH[0][i]=0; }

    //Allocate XH
  XH=(PDEFIELD_TYPE **)malloc(sizey*sizeof(PDEFIELD_TYPE*));
  if (BH==NULL)
    MemoryWarning();

  XH[0]=(PDEFIELD_TYPE *)malloc(sizey*sizex*sizeof(PDEFIELD_TYPE));
  if (XH[0]==NULL)  
    MemoryWarning();

    {for (int i=1;i<sizey;i++) 
    XH[i]=XH[i-1]+sizex;}

  /* clear matrix */
  {for (int i=0;i<sizey*sizex;i++) 
    XH[0][i]=0; }


  //Allocate lowerV
  lowerV=(PDEFIELD_TYPE **)malloc(sizex*sizeof(PDEFIELD_TYPE*));
  if (lowerV==NULL)
    MemoryWarning();

  lowerV[0]=(PDEFIELD_TYPE *)malloc(sizex*sizey*sizeof(PDEFIELD_TYPE));
  if (lowerV[0]==NULL)  
    MemoryWarning();

    {for (int i=1;i<sizex;i++) 
    lowerV[i]=lowerV[i-1]+sizey;}
  /* clear matrix */

  {for (int i=0;i<sizex*sizey;i++) 
    lowerV[0][i]=0; }


  //Allocate upperV
  upperV=(PDEFIELD_TYPE **)malloc(sizex*sizeof(PDEFIELD_TYPE*));
  if (upperV==NULL)
    MemoryWarning();

  upperV[0]=(PDEFIELD_TYPE *)malloc(sizex*sizey*sizeof(PDEFIELD_TYPE));
  if (upperV[0]==NULL)  
    MemoryWarning();

    {for (int i=1;i<sizex;i++) 
    upperV[i]=upperV[i-1]+sizey;}
  /* clear matrix */

  {for (int i=0;i<sizex*sizey;i++) 
    upperV[0][i]=0; }


  //Allocate diagV
  diagV=(PDEFIELD_TYPE **)malloc(sizex*sizeof(PDEFIELD_TYPE*));
  if (diagV==NULL)
    MemoryWarning();

  diagV[0]=(PDEFIELD_TYPE *)malloc(sizex*sizey*sizeof(PDEFIELD_TYPE));
  if (diagV[0]==NULL)  
    MemoryWarning();

    {for (int i=1;i<sizex;i++) 
    diagV[i]=diagV[i-1]+sizey;}
  /* clear matrix */

  {for (int i=0;i<sizex*sizey;i++) 
    diagV[0][i]=0; }


  //Allocate BV
  BV=(PDEFIELD_TYPE **)malloc(sizex*sizeof(PDEFIELD_TYPE*));
  if (BV==NULL)
    MemoryWarning();

  BV[0]=(PDEFIELD_TYPE *)malloc(sizex*sizey*sizeof(PDEFIELD_TYPE));
  if (BV[0]==NULL)  
    MemoryWarning();

    {for (int i=1;i<sizex;i++) 
    BV[i]=BV[i-1]+sizey;}
  /* clear matrix */

  {for (int i=0;i<sizex*sizey;i++) 
    BV[0][i]=0; }

}


void PDE::InitializePDEvars(void){
    for (int i = 0; i< sizex* sizey;i++){

      PDEvars[0][0][i] = -0.050;
      PDEvars[1][0][i] = 0.32;
      PDEvars[2][0][i] = 0.0002;
      PDEvars[3][0][i] = 0;
      PDEvars[4][0][i] = 0;
      PDEvars[5][0][i] = 1;
      PDEvars[6][0][i] = 1;
      PDEvars[7][0][i] = 1;
      PDEvars[8][0][i] = 0;
      PDEvars[9][0][i] = 1;
      PDEvars[10][0][i] = 0;
      PDEvars[11][0][i] = 0.75;
      PDEvars[12][0][i] = 0.75; 
      PDEvars[13][0][i] = 0;
      PDEvars[14][0][i] = 0.1;
      PDEvars[15][0][i] = 1;
      PDEvars[16][0][i] = 0;
      PDEvars[17][0][i] = 9.2;
      PDEvars[18][0][i] = 0;
      PDEvars[19][0][i] = 0.75;
      PDEvars[20][0][i] = 0.3;
      PDEvars[21][0][i] = 0.9;
      PDEvars[22][0][i] = 0.1;
    }


}




void PDE::Plot(Graphics *g,const int l) {
  // l=layer: default layer is 0
  for (int x=0;x<sizex;x++) {
    for (int y=0;y<sizey;y++) {
      // Make the pixel four times as large
      // to fit with the CPM plane
      g->Point(MapColour(PDEvars[l][x][y]),x,y);
      g->Point(MapColour(PDEvars[l][x][y]),x+1,y);
      g->Point(MapColour(PDEvars[l][x][y]),x,y+1);
      g->Point(MapColour(PDEvars[l][x][y]),x+1,y+1);
    }
  }
}

// Plot the value of the PDE only in the medium of the CPM
void PDE::Plot(Graphics *g, CellularPotts *cpm, const int l) {
  // suspend=true suspends calling of DrawScene
  for (int x=0;x<sizex;x++) {
    for (int y=0;y<sizey;y++) { 
      if (cpm->Sigma(x,y)==0) {
	// Make the pixel four times as large
	// to fit with the CPM plane
        g->Point(MapColour(PDEvars[l][x][y]),x,y);
	g->Point(MapColour(PDEvars[l][x][y]),x+1,y);
	g->Point(MapColour(PDEvars[l][x][y]),x,y+1);
	g->Point(MapColour(PDEvars[l][x][y]),x+1,y+1);
      }
    }
  }
}

void PDE::ContourPlot(Graphics *g, int l, int colour) {
  // calls "conrec" routine by Paul Bourke, as downloaded from
  // http://astronomy.swin.edu.au/~pbourke/projection/conrec

  // number of contouring levels
  int nc = 10;

  // A one dimensional array z(0:nc-1) that saves as a list of the contour levels in increasing order.   
  double *z=(double *)malloc(nc*sizeof(double));
  double min=Min(l), max=Max(l);
  double step=(max-min)/nc;
  if (min == 0 && max == 0) return;

  for (int i=0;i<nc;i++) {
    z[i]=(i+1)*step;
  }
  double *x=(double *)malloc(sizex*sizeof(double));
  for (int i=0;i<sizex;i++) {
    x[i]=i;
  }
  double *y=(double *)malloc(sizey*sizeof(double));
  for (int i=0;i<sizey;i++) {
    y[i]=i;
  }

  conrec(PDEvars[l],0,sizex-1,0,sizey-1,x,y,nc,z,g,colour);
  
  free(x);
  free(y);
  free(z);
}



int MapColour3(double val, int l) {
	int step=0;
	if (l==2){
	step = (240)/par.max_Act;
  return (int)(256-val*step-1);}
	else if (l==3 && val>1){
	step = (256)/2;
  return (int)(500+val*step);}
	else
	return 0;
}


void PDE::PlotInCells (Graphics *g, CellularPotts *cpm, const int l) {
  for (int x=0;x<sizex;x++) {
    for (int y=0;y<sizey;y++) {
      if( cpm->Sigma(x,y)>0) {
        if (par.lambda_Act>0) {
	   g->Rectangle(MapColour3(cpm->actPixels[{x,y}],l), x, y);
        } else {
	  g->Rectangle(255, x, y);
        }
        if (par.lambda_matrix>0) {
          if (cpm->matrix[x][y]>0) {
            g->Rectangle(256, x, y);
          }
        }
      } else if (cpm->Sigma(x,y)==-2) {
        g->Rectangle(10, x, y);
      }
      if (cpm->Sigma(x,y)==-3) {
	g->Rectangle(256, x, y);
      }
    }
  }
}



void PDE::SetupOpenCL(){
  extern CLManager clm;

  program = clm.make_program(par.opencl_core_path, OPENCL_PDE_TYPE);

  //Secretion and diffusion variables
  PDEFIELD_TYPE dt = (PDEFIELD_TYPE) par.dt;
  PDEFIELD_TYPE dx2 = (PDEFIELD_TYPE) par.dx*par.dx;
  PDEFIELD_TYPE decay_rate = (PDEFIELD_TYPE) * par.decay_rate;
  PDEFIELD_TYPE secr_rate = (PDEFIELD_TYPE) * par.secr_rate;
  
  
  int btype = 1;
  if (par.periodic_boundaries) btype=2;

  //Allocate memory on the GPU
  clm.cpm = cl::Buffer(clm.context, CL_MEM_READ_WRITE, sizeof(int)*sizex*sizey); 
  clm.numberofedges = cl::Buffer(clm.context, CL_MEM_READ_WRITE, sizeof(int)*sizex*sizey); 
  clm.couplingcoefficient = cl::Buffer(clm.context, CL_MEM_READ_WRITE, sizeof(PDEFIELD_TYPE)*sizex*sizey); 
  clm.pdeA = cl::Buffer(clm.context, CL_MEM_READ_WRITE, sizeof(PDEFIELD_TYPE)*sizex*sizey*layers);
  clm.pdeB = cl::Buffer(clm.context, CL_MEM_READ_WRITE, sizeof(PDEFIELD_TYPE)*sizex*sizey*layers); 
  clm.diffco = cl::Buffer(clm.context, CL_MEM_READ_WRITE, sizeof(PDEFIELD_TYPE)*layers);


  //Making kernel and setting arguments
  kernel_ODEstep = cl::Kernel(program,"ODEstep");      

  kernel_ODEstep.setArg(0, clm.cpm);
  kernel_ODEstep.setArg(1, clm.pdeA);
  kernel_ODEstep.setArg(2, clm.pdeB);
  kernel_ODEstep.setArg(3, sizeof(int), &sizex);
  kernel_ODEstep.setArg(4, sizeof(int), &sizey);
  kernel_ODEstep.setArg(5, sizeof(int), &layers);
  kernel_ODEstep.setArg(6, sizeof(PDEFIELD_TYPE), &decay_rate);
  kernel_ODEstep.setArg(7, sizeof(PDEFIELD_TYPE), &dt);
  kernel_ODEstep.setArg(8, sizeof(PDEFIELD_TYPE), &dx2);
  kernel_ODEstep.setArg(9, clm.diffco);
  kernel_ODEstep.setArg(10,sizeof(PDEFIELD_TYPE), &secr_rate);
  kernel_ODEstep.setArg(11, sizeof(int),  &btype);
  kernel_ODEstep.setArg(12, clm.numberofedges);
  kernel_ODEstep.setArg(13, clm.couplingcoefficient);


  PDEFIELD_TYPE diff_coeff[layers];

  for (int index = 0; index < layers; index++){
    diff_coeff[index] = (PDEFIELD_TYPE) par.diff_coeff[index];
  }

  clm.queue.enqueueWriteBuffer(clm.diffco,
    CL_TRUE, 0, sizeof(PDEFIELD_TYPE)*layers, diff_coeff);

  openclsetup = true;
} 


void PDE::ODEstepCL(CellularPotts *cpm, int repeat){
    extern CLManager clm; 
    if (!openclsetup ){this->SetupOpenCL();}
    //A B scheme used to keep arrays on GPU
    int errorcode = 0;

    
    
    //Write the cellSigma array to GPU for secretion
    clm.queue.enqueueWriteBuffer(clm.cpm,
    CL_TRUE, 0, sizeof(int)*sizex*sizey, cpm->getSigma()[0]);
    clm.queue.enqueueWriteBuffer(clm.numberofedges,
    CL_TRUE, 0, sizeof(int)*sizex*sizey, cpm->getNumberofedges()[0]);
    clm.queue.enqueueWriteBuffer(clm.couplingcoefficient,
    CL_TRUE, 0, sizeof(PDEFIELD_TYPE)*sizex*sizey, cpm->getCouplingCoefficient()[0]);
    //Writing pdefield PDEvars is only necessary if modified outside of clm.pdeA)kernel
    if (first_round) {
      clm.queue.enqueueWriteBuffer(clm.pdeA,  CL_TRUE, 0, sizeof(PDEFIELD_TYPE)*sizex*sizey*layers, PDEvars[0][0]);
      first_round = false;
    }
    //Main loop executing kernel and switching between A and B arrays
    for (int index = 0; index < repeat; index ++){
      for (int innerloop = 0; innerloop < 4; innerloop++){

          using std::chrono::high_resolution_clock;
          using std::chrono::duration_cast;
          using std::chrono::duration;
          using std::chrono::milliseconds;

          auto t1 = high_resolution_clock::now();

        kernel_ODEstep.setArg(14, sizeof(int), &PDEsteps);
        if(innerloop == 0 || innerloop == 1){
          kernel_ODEstep.setArg(1, clm.pdeA);
          kernel_ODEstep.setArg(2, clm.pdeB);
        }
        else{
          kernel_ODEstep.setArg(1, clm.pdeB);
          kernel_ODEstep.setArg(2, clm.pdeA);
        }
        if(innerloop == 0 || innerloop == 2){
          errorcode = clm.queue.enqueueNDRangeKernel(kernel_ODEstep,
                      cl::NullRange, cl::NDRange(sizex*sizey), cl::NullRange);
          errorcode = clm.queue.finish();
        }
        else{
          errorcode = clm.queue.enqueueNDRangeKernel(kernel_ODEstep,
                      cl::NullRange, cl::NDRange(1), cl::NullRange);
          errorcode = clm.queue.finish();
        }
        if (errorcode != 0){
          printf("Error during OpenCL secretion and diffusion");
          exit(0);
        }

        auto t2 = high_resolution_clock::now();
        duration<double, std::milli> ms_double = t2 - t1;
        cout << "For PDEsteps = " << PDEsteps << ", " << ms_double.count() << " ms has elapsed" << endl;
        PDEsteps += 1;

      }
    }
    //Reading from correct array containing the output
    if (clm.pde_AB == 0) {
      clm.queue.enqueueReadBuffer(clm.pdeB,CL_TRUE,0,
                            sizeof(PDEFIELD_TYPE)*sizex*sizey*layers, PDEvars[0][0]);
    }
    else {
      clm.queue.enqueueReadBuffer(clm.pdeA,CL_TRUE,0,
                            sizeof(PDEFIELD_TYPE)*sizex*sizey*layers, PDEvars[0][0]);
    }
    if (errorcode != CL_SUCCESS) cout << "error:" << errorcode << endl;
    
}

void PDE::InitializeCuda(CellularPotts *cpm){
  AllocateTridiagonalvars(sizex, sizey);
  cout << "A" << endl;
  couplingcoefficient = cpm->getCouplingCoefficient();
  hipMallocManaged(&PDEvars[0][0], layers*sizex*sizey*sizeof(PDEFIELD_TYPE));

  hipMallocManaged(&alt_PDEvars[0][0], layers*sizex*sizey*sizeof(PDEFIELD_TYPE));
  hipMallocManaged(&couplingcoefficient[0], sizex*sizey*sizeof(PDEFIELD_TYPE));
  cout << "B" << endl;

  //Needed for ADI steps
  hipMallocManaged(&upperH[0], sizex*sizey*sizeof(PDEFIELD_TYPE));
  hipMallocManaged(&diagH[0], sizex*sizey*sizeof(PDEFIELD_TYPE));
  hipMallocManaged(&lowerH[0], sizex*sizey*sizeof(PDEFIELD_TYPE));
  /*cout << "BH[0] = " << BH[0] << endl;
  cout << "BH[0]+sizex =" << BH[0]+sizex << endl;
  cout << "BH[1] = " << BH[1] << endl; */
  hipMallocManaged(&BH[0], sizex*sizey*sizeof(PDEFIELD_TYPE));
  hipMallocManaged(&XH[0], sizex*sizey*sizeof(PDEFIELD_TYPE));
  hipMallocManaged(&upperV[0], sizey*sizex*sizeof(PDEFIELD_TYPE));
  hipMallocManaged(&diagV[0], sizey*sizex*sizeof(PDEFIELD_TYPE));
  hipMallocManaged(&lowerV[0], sizey*sizex*sizeof(PDEFIELD_TYPE));
  hipMallocManaged(&BV[0], sizey*sizex*sizeof(PDEFIELD_TYPE));

  handleH = 0;
  pbuffersizeH = 0;
  pbufferH = NULL;
  statusH=hipsparseCreate(&handleH);
  hipsparseSgtsvInterleavedBatch_bufferSizeExt(handleH, 0, sizex, lowerH[0], diagH[0], upperH[0], BH[0], sizey, &pbuffersizeH); //Compute required buffersize for horizontal sweep

  hipMalloc( &pbufferH, sizeof(char)* pbuffersizeH);

  handleV = 0;
  pbuffersizeV = 0;
  pbufferV = NULL;
  statusV=hipsparseCreate(&handleV);
  hipsparseSgtsvInterleavedBatch_bufferSizeExt(handleV, 0, sizey, lowerV[0], diagV[0], upperV[0], BV[0], sizex, &pbuffersizeV); //Compute required buffersize for vertical sweep
  hipMalloc( &pbufferV, sizeof(char)* pbuffersizeV);
}

void PDE::InitializePDEs(CellularPotts *cpm){
  InitializePDEvars();
  InitializeCuda(cpm);
}


__global__ void InitializeDiagonals(int sizex, int sizey, PDEFIELD_TYPE twooverdt, PDEFIELD_TYPE dx2, PDEFIELD_TYPE* lowerH, PDEFIELD_TYPE* upperH, PDEFIELD_TYPE* diagH, PDEFIELD_TYPE* lowerV, PDEFIELD_TYPE* upperV, PDEFIELD_TYPE* diagV, PDEFIELD_TYPE* couplingcoefficient){
  //This function could in theory be parellelized further, split into 6 (each part only assigning 1 value.), but this is probably slower
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int xloc; //position we currently want to assign to
  int yloc;
  int idcc; //id corresponding to the couplingcoefficient (+sizey to get the value right, +1 to get the value above)
  for (int id = index; id < sizex*sizey; id += stride){
    xloc = id%sizex;
    yloc = id/sizex;
    idcc = xloc*sizey + yloc;
    if(xloc == 0){
      lowerH[id] = 0;
      diagH[id] = couplingcoefficient[idcc+sizey]/dx2 + twooverdt;
      upperH[id] = -couplingcoefficient[idcc+sizey]/dx2;
    }
    else if(xloc == sizex -1){
      lowerH[id] = -couplingcoefficient[idcc-sizey]/dx2;;
      diagH[id] = couplingcoefficient[idcc-sizey]/dx2 + twooverdt;
      upperH[id] = 0;
    }
    else{
      lowerH[id] = -couplingcoefficient[idcc-sizey]/dx2;
      diagH[id] = (couplingcoefficient[idcc+sizey]+couplingcoefficient[idcc-sizey])/dx2 + twooverdt;
      upperH[id] = -couplingcoefficient[idcc+sizey]/dx2;
    }
    xloc = id/sizey;
    yloc = id%sizey;
    idcc = xloc*sizey + yloc;

    if(yloc == 0){
      lowerH[id] = 0;
      diagH[id] = couplingcoefficient[idcc+1]/dx2 + twooverdt;
      upperH[id] = -couplingcoefficient[idcc+1]/dx2;
    }
    else if(yloc == sizey -1){
      lowerH[id] = -couplingcoefficient[idcc-1]/dx2;;
      diagH[id] = couplingcoefficient[idcc-1]/dx2 + twooverdt;
      upperH[id] = 0;
    }
    else{
      lowerH[id] = -couplingcoefficient[idcc-1]/dx2;
      diagH[id] = (couplingcoefficient[idcc+1]+couplingcoefficient[idcc-1])/dx2 + twooverdt;
      upperH[id] = -couplingcoefficient[idcc+1]/dx2;
      
    }  

  
  }

}

__global__ void InitializeHorizontalVectors(int sizex, int sizey, PDEFIELD_TYPE twooverdt, PDEFIELD_TYPE dx2, PDEFIELD_TYPE* BH, PDEFIELD_TYPE* couplingcoefficient, PDEFIELD_TYPE* alt_PDEvars){
  //This function could in theory be parellelized further, split into 6 (each part only assigning 1 value.), but this is probably slower
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int xloc;
  int yloc;
  int idcc; //id corresponding to the couplingcoefficient and alt_PDEvars(+sizey to get the value right, +1 to get the value above)
  for (int id = index; id < sizex*sizey; id += stride){
    xloc = id%sizex;
    yloc = id/sizex;
    idcc = xloc*sizey + yloc;

    if (yloc == 0)
      BH[idcc] = twooverdt*alt_PDEvars[idcc] + (couplingcoefficient[idcc+1]*(alt_PDEvars[idcc+1] - alt_PDEvars[idcc]))/dx2; 
    else if (yloc == sizey-1)
      BH[idcc] = twooverdt*alt_PDEvars[idcc] + (couplingcoefficient[idcc-1]*(alt_PDEvars[idcc-1] - alt_PDEvars[idcc]))/dx2;  
    else 
      BH[idcc] = twooverdt*alt_PDEvars[idcc] + (couplingcoefficient[idcc+1]*(alt_PDEvars[idcc+1] - alt_PDEvars[idcc]) + couplingcoefficient[idcc-1]*(alt_PDEvars[idcc-1] - alt_PDEvars[idcc]))/dx2;
  
  }

}

__global__ void InitializeVerticalVectors(int sizex, int sizey, PDEFIELD_TYPE twooverdt, PDEFIELD_TYPE dx2, PDEFIELD_TYPE* BH, PDEFIELD_TYPE* couplingcoefficient, PDEFIELD_TYPE* alt_PDEvars){

  //This function could in theory be parellelized further, split into 6 (each part only assigning 1 value.), but this is probably slower
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int xloc;
  int yloc;
  int idcc; //id corresponding to the couplingcoefficient and alt_PDEvars(+sizey to get the value right, +1 to get the value above)
  for (int id = index; id < sizex*sizey; id += stride){
    xloc = id/sizey;
    yloc = id%sizey;
    idcc = xloc*sizey + yloc;

    if (xloc == 0)
      BH[idcc] = twooverdt*alt_PDEvars[idcc] + (couplingcoefficient[idcc+sizey]*(alt_PDEvars[idcc+sizey] - alt_PDEvars[idcc]))/dx2; 
    else if (xloc == sizey-1)
      BH[idcc] = twooverdt*alt_PDEvars[idcc] + (couplingcoefficient[idcc-sizey]*(alt_PDEvars[idcc-sizey] - alt_PDEvars[idcc]))/dx2;  
    else 
      BH[idcc] = twooverdt*alt_PDEvars[idcc] + (couplingcoefficient[idcc+sizey]*(alt_PDEvars[idcc+sizey] - alt_PDEvars[idcc]) + couplingcoefficient[idcc-sizey]*(alt_PDEvars[idcc-sizey] - alt_PDEvars[idcc]))/dx2;
  
  }

}

__global__ void NewPDEfieldH(int sizex, int sizey, PDEFIELD_TYPE* BH, PDEFIELD_TYPE* PDEvars){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < sizex*sizey; id += stride){
    PDEvars[(id%sizex)*sizey + id/sizex] = BH[id]; //Conversion is needed because PDEvars iterates over rowas first and then columns, while BH does the opposite
  }
}

__global__ void NewPDEfieldV(int sizex, int sizey, PDEFIELD_TYPE* BV, PDEFIELD_TYPE* PDEvars){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int id = index; id < sizex*sizey; id += stride){
    PDEvars[id] = BV[id];
  }
}

__device__ void ComputeDerivs(PDEFIELD_TYPE t, PDEFIELD_TYPE* y, PDEFIELD_TYPE* dydt, int id){ // computes the derivatives at some time point with forward Euler
    //equations for Paci2020
  
    // Software implementation of the Paci2020 model of the action potential 
    // of human induced pluripotent stem cell-derived cardiomyocytes, 
    // used in 10.1016/j.bpj.2020.03.018
    //
    // This software is provided for NON-COMMERCIAL USE ONLY 
    // (read the license included in the zip file).
  
  
    //-------------------------------------------------------------------------------
    // State variables
    //-------------------------------------------------------------------------------
  
    // 0: Vm (volt) (in Membrane)
    // 1: Ca_SR (millimolar) (in calcium_dynamics)
    // 2: Cai (millimolar) (in calcium_dynamics)
    // NOT USED 3: g (dimensionless) (in calcium_dynamics)
    // 4: d (dimensionless) (in i_CaL_d_gate)
    // 5: f1 (dimensionless) (in i_CaL_f1_gate)
    // 6: f2 (dimensionless) (in i_CaL_f2_gate)
    // 7: fCa (dimensionless) (in i_CaL_fCa_gate)
    // 8: Xr1 (dimensionless) (in i_Kr_Xr1_gate)
    // 9: Xr2 (dimensionless) (in i_Kr_Xr2_gate)
    // 10: Xs (dimensionless) (in i_Ks_Xs_gate)
    // 11: h (dimensionless) (in i_Na_h_gate)
    // 12: j (dimensionless) (in i_Na_j_gate)
    // 13: m (dimensionless) (in i_Na_m_gate)
    // 14: Xf (dimensionless) (in i_f_Xf_gate)
    // 15: q (dimensionless) (in i_to_q_gate)
    // 16: r (dimensionless) (in i_to_r_gate)
    // 17: Nai (millimolar) (in sodium_dynamics)
    // 18: m_L (dimensionless) (in i_NaL_m_gate)
    // 19: h_L (dimensionless) (in i_NaL_h_gate)
    // 20: RyRa (dimensionless) (in calcium_dynamics)
    // 21: RyRo (dimensionless) (in calcium_dynamics)
    // 22: RyRc (dimensionless) (in calcium_dynamics)
  
    //// Constants
    PDEFIELD_TYPE F = 96485.3415;     // coulomb_per_mole (in model_parameters)
    PDEFIELD_TYPE R = 8.314472;       // joule_per_mole_kelvin (in model_parameters)
    PDEFIELD_TYPE T = 310.0;          // kelvin (in model_parameters) //37°C
  
    //// Cell geometry
    PDEFIELD_TYPE V_SR = 583.73;        // micrometre_cube (in model_parameters)
    PDEFIELD_TYPE Vc   = 8800.0;        // micrometre_cube (in model_parameters)
    PDEFIELD_TYPE Cm   = 9.87109e-11;   // farad (in model_parameters)
  
    //// Extracellular concentrations
    PDEFIELD_TYPE Nao = 151.0; // millimolar (in model_parameters)
    PDEFIELD_TYPE Ko  = 5.4;   // millimolar (in model_parameters)
    PDEFIELD_TYPE Cao = 1.8;   // millimolar (in model_parameters)
  
    //// Intracellular concentrations
    // Naio = 10 mM y[17]
    PDEFIELD_TYPE Ki = 150.0;   // millimolar (in model_parameters)
    // Cai  = 0.0002 mM y[2]
    // caSR = 0.3 mM y[1]
  
    // time (second)
  
    //// Nernst potential
    PDEFIELD_TYPE E_Na = R*T/F*log(Nao/y[17]);
    PDEFIELD_TYPE E_Ca = 0.5*R*T/F*log(Cao/y[2]);
    PDEFIELD_TYPE E_K  = R*T/F*log(Ko/Ki);
    PDEFIELD_TYPE PkNa = 0.03;   // dimensionless (in electric_potentials)
    PDEFIELD_TYPE E_Ks = R*T/F*log((Ko+PkNa*Nao)/(Ki+PkNa*y[17]));
  
    //// INa adapted from DOI:10.3389/fphys.2018.00080
    PDEFIELD_TYPE g_Na        = 3671.2302; //((time<tDrugApplication)*1+(time >= tDrugApplication)*INaFRedMed)*6447.1896;
    PDEFIELD_TYPE i_Na        =  g_Na*pow((float)y[13],3.0f)*y[11]*y[12]*(y[0] - E_Na);
  
    PDEFIELD_TYPE m_inf       = 1 / (1 + exp((y[0]*1000 + 39)/-11.2));
    PDEFIELD_TYPE tau_m       = (0.00001 + 0.00013*exp(-pow((float)((y[0]*1000 + 48)/15),2.0f)) + 0.000045 / (1 + exp((y[0]*1000 + 42)/-5)));
    dydt[13]   = (m_inf-y[13])/tau_m;
  
    PDEFIELD_TYPE h_inf       = 1 / (1 + exp((y[0]*1000 + 66.5)/6.8));
    PDEFIELD_TYPE tau_h       = (0.00007 + 0.034 / (1 + exp((y[0]*1000 + 41)/5.5) + exp(-(y[0]*1000 + 41)/14)) + 0.0002 / (1 + exp(-(y[0]*1000 + 79)/14)));
    dydt[11]   = (h_inf-y[11])/tau_h;
  
    PDEFIELD_TYPE j_inf       = h_inf;
    PDEFIELD_TYPE tau_j       = 10*(0.0007 + 0.15 / (1 + exp((y[0]*1000 + 41)/5.5) + exp(-(y[0]*1000 + 41)/14)) + 0.002 / (1 + exp(-(y[0]*1000 + 79)/14)));
    dydt[12]   = (j_inf-y[12])/tau_j;
  
  
    //// INaL
    PDEFIELD_TYPE myCoefTauM  = 1;
    PDEFIELD_TYPE tauINaL     = 200; //ms
    PDEFIELD_TYPE GNaLmax     = 17.25;//((time<tDrugApplication)*1+(time >= tDrugApplication)*INaLRedMed)* 2.3*7.5; //(S/F)
    PDEFIELD_TYPE Vh_hLate    = 87.61;
    PDEFIELD_TYPE i_NaL       = GNaLmax* pow((float)y[18],3.0f)*y[19]*(y[0]-E_Na);
  
    PDEFIELD_TYPE m_inf_L     = 1/(1+exp(-(y[0]*1000+42.85)/(5.264)));
    PDEFIELD_TYPE alpha_m_L   = 1/(1+exp((-60-y[0]*1000)/5));
    PDEFIELD_TYPE beta_m_L    = 0.1/(1+exp((y[0]*1000+35)/5))+0.1/(1+exp((y[0]*1000-50)/200));
    PDEFIELD_TYPE tau_m_L     = 1 * myCoefTauM*alpha_m_L*beta_m_L;
    dydt[18]   = (m_inf_L-y[18])/tau_m_L*1000;
  
    PDEFIELD_TYPE h_inf_L     = 1/(1+exp((y[0]*1000+Vh_hLate)/(7.488)));
    PDEFIELD_TYPE tau_h_L     = 1 * tauINaL;
    dydt[19]   = (h_inf_L-y[19])/tau_h_L*1000;
  
    //// If adapted from DOI:10.3389/fphys.2018.00080
    PDEFIELD_TYPE g_f         = 1; //((time<tDrugApplication)*1+(time >= tDrugApplication)*IfRedMed)*22.2763088;
    PDEFIELD_TYPE fNa         = 0.37;
    PDEFIELD_TYPE fK          = 1 - fNa;
    PDEFIELD_TYPE i_fK        = fK*g_f*y[14]*(y[0] - E_K);
    PDEFIELD_TYPE i_fNa       = fNa*g_f*y[14]*(y[0] - E_Na);
    PDEFIELD_TYPE i_f         = i_fK + i_fNa;
  
    PDEFIELD_TYPE Xf_infinity = 1.0/(1.0 + exp((y[0]*1000 + 69)/8));
    PDEFIELD_TYPE tau_Xf      = 5600 / (1 + exp((y[0]*1000 + 65)/7) + exp(-(y[0]*1000 + 65)/19));
    dydt[14]   = 1000*(Xf_infinity-y[14])/tau_Xf;
    
  
    //// ICaL
    PDEFIELD_TYPE g_CaL       = 8.635702e-5;   // metre_cube_per_F_per_s (in i_CaL)
    PDEFIELD_TYPE i_CaL;  
    PDEFIELD_TYPE precision = 0.0001;     
    if(y[0]< precision && y[0] > -precision) //hopital
      i_CaL =  g_CaL*4.0*y[0]*pow((float)F,2.0f)/(R*T) *y[4]*y[5]*y[6]*y[7] / (2.0*F/(R*T)) * (y[2] - 0.341*Cao);
    else
      i_CaL = g_CaL*4.0*y[0]*pow((float)F,2.0f)/(R*T)*(y[2]*exp(2.0*y[0]*F/(R*T))-0.341*Cao)/(exp(2.0*y[0]*F/(R*T))-1.0)*y[4]*y[5]*y[6]*y[7]; //((time<tDrugApplication)*1+(time >= tDrugApplication)*ICaLRedMed)*g_CaL*4.0*y[0]*pow(F,2.0)/(R*T)*(y[2]*exp(2.0*y[0]*F/(R*T))-0.341*Cao)/(exp(2.0*y[0]*F/(R*T))-1.0)*y[4]*y[5]*y[6]*y[7];
  
    PDEFIELD_TYPE d_infinity  = 1.0/(1.0+exp(-(y[0]*1000.0+9.1)/7.0));
    PDEFIELD_TYPE alpha_d     = 0.25+1.4/(1.0+exp((-y[0]*1000.0-35.0)/13.0));
    PDEFIELD_TYPE beta_d      = 1.4/(1.0+exp((y[0]*1000.0+5.0)/5.0));
    PDEFIELD_TYPE gamma_d     = 1.0/(1.0+exp((-y[0]*1000.0+50.0)/20.0));
    PDEFIELD_TYPE tau_d       = (alpha_d*beta_d+gamma_d)*1.0/1000.0;
    dydt[4]    = (d_infinity-y[4])/tau_d;
  
    PDEFIELD_TYPE f1_inf      = 1.0/(1.0+exp((y[0]*1000.0+26.0)/3.0));
    PDEFIELD_TYPE constf1;
    if (f1_inf-y[5] > 0.0)
        constf1 = 1.0+1433.0*(y[2]-50.0*1.0e-6);
    else
        constf1 = 1.0;
  
    PDEFIELD_TYPE tau_f1      = (20.0+1102.5*exp(-pow((float)((y[0]*1000.0+27.0)/15.0),2.0f))+200.0/(1.0+exp((13.0-y[0]*1000.0)/10.0))+180.0/(1.0+exp((30.0+y[0]*1000.0)/10.0)))*constf1/1000.0;
    dydt[5]    = (f1_inf-y[5])/tau_f1;
  
    PDEFIELD_TYPE f2_inf      = 0.33+0.67/(1.0+exp((y[0]*1000.0+32.0)/4.0));
    PDEFIELD_TYPE constf2     = 1.0;
    PDEFIELD_TYPE tau_f2      = (600.0*exp(-pow((float)(y[0]*1000.0+25.0),2.0f)/170.0)+31.0/(1.0+exp((25.0-y[0]*1000.0)/10.0))+16.0/(1.0+exp((30.0+y[0]*1000.0)/10.0)))*constf2/1000.0;
    dydt[6]    = (f2_inf-y[6])/tau_f2;
  
    PDEFIELD_TYPE alpha_fCa   = 1.0/(1.0+pow((float)(y[2]/0.0006),8.0f));
    PDEFIELD_TYPE beta_fCa    = 0.1/(1.0+exp((y[2]-0.0009)/0.0001));
    PDEFIELD_TYPE gamma_fCa   = 0.3/(1.0+exp((y[2]-0.00075)/0.0008));
    PDEFIELD_TYPE fCa_inf     = (alpha_fCa+beta_fCa+gamma_fCa)/1.3156;
  
    PDEFIELD_TYPE constfCa;
    if ((y[0] > -0.06) && (fCa_inf > y[7]))
        constfCa = 0.0;
    else
        constfCa = 1.0;
  
    PDEFIELD_TYPE tau_fCa     = 0.002;   // second (in i_CaL_fCa_gate)
    dydt[7]    = constfCa*(fCa_inf-y[7])/tau_fCa;
  
    //// Ito
    PDEFIELD_TYPE g_to        = 29.9038;   // S_per_F (in i_to)  
    PDEFIELD_TYPE i_to        = g_to*(y[0]-E_K)*y[15]*y[16];
  
    PDEFIELD_TYPE q_inf       = 1.0/(1.0+exp((y[0]*1000.0+53.0)/13.0));
    PDEFIELD_TYPE tau_q       = (6.06+39.102/(0.57*exp(-0.08*(y[0]*1000.0+44.0))+0.065*exp(0.1*(y[0]*1000.0+45.93))))/1000.0;
    dydt[15]   = (q_inf-y[15])/tau_q;
  
  
    PDEFIELD_TYPE r_inf       = 1.0/(1.0+exp(-(y[0]*1000.0-22.3)/18.75));
    PDEFIELD_TYPE tau_r       = (2.75352+14.40516/(1.037*exp(0.09*(y[0]*1000.0+30.61))+0.369*exp(-0.12*(y[0]*1000.0+23.84))))/1000.0;
    dydt[16]   = (r_inf-y[16])/tau_r;
  
    //// IKs
    PDEFIELD_TYPE g_Ks        = 2.041;   // S_per_F (in i_Ks)
    PDEFIELD_TYPE i_Ks        = g_Ks*(y[0]-E_Ks)*pow((float)y[10],2.0f)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/y[2]),1.4f))); // ((time<tDrugApplication)*1+(time >= tDrugApplication)*IKsRedMed)*g_Ks*(y[0]-E_Ks)*pow((float)y[10],2.0)*(1.0+0.6/(1.0+pow((float)(3.8*0.00001/y[2]),1.4f)));
  
    PDEFIELD_TYPE Xs_infinity = 1.0/(1.0+exp((-y[0]*1000.0-20.0)/16.0));
    PDEFIELD_TYPE alpha_Xs    = 1100.0/sqrt(1.0+exp((-10.0-y[0]*1000.0)/6.0));
    PDEFIELD_TYPE beta_Xs     = 1.0/(1.0+exp((-60.0+y[0]*1000.0)/20.0));
    PDEFIELD_TYPE tau_Xs      = 1.0*alpha_Xs*beta_Xs/1000.0;
    dydt[10]   = (Xs_infinity-y[10])/tau_Xs;
  
    //// IKr
    PDEFIELD_TYPE L0           = 0.025;   // dimensionless (in i_Kr_Xr1_gate)
    PDEFIELD_TYPE Q            = 2.3;     // dimensionless (in i_Kr_Xr1_gate)
    PDEFIELD_TYPE g_Kr         = 29.8667;   // S_per_F (in i_Kr)
    PDEFIELD_TYPE i_Kr         = g_Kr*(y[0]-E_K)*y[8]*y[9]*sqrt(Ko/5.4); //((time<tDrugApplication)*1+(time >= tDrugApplication)*IKrRedMed)*g_Kr*(y[0]-E_K)*y[8]*y[9]*sqrt(Ko/5.4);
  
    PDEFIELD_TYPE V_half       = 1000.0*(-R*T/(F*Q)*log(pow((float)(1.0+Cao/2.6),4.0f)/(L0*pow((float)(1.0+Cao/0.58),4.0f)))-0.019);
  
    PDEFIELD_TYPE Xr1_inf      = 1.0/(1.0+exp((V_half-y[0]*1000.0)/4.9));
    PDEFIELD_TYPE alpha_Xr1    = 450.0/(1.0+exp((-45.0-y[0]*1000.0)/10.0));
    PDEFIELD_TYPE beta_Xr1     = 6.0/(1.0+exp((30.0+y[0]*1000.0)/11.5));
    PDEFIELD_TYPE tau_Xr1      = 1.0*alpha_Xr1*beta_Xr1/1000.0;
    dydt[8]     = (Xr1_inf-y[8])/tau_Xr1;
  
    PDEFIELD_TYPE Xr2_infinity = 1.0/(1.0+exp((y[0]*1000.0+88.0)/50.0));
    PDEFIELD_TYPE alpha_Xr2    = 3.0/(1.0+exp((-60.0-y[0]*1000.0)/20.0));
    PDEFIELD_TYPE beta_Xr2     = 1.12/(1.0+exp((-60.0+y[0]*1000.0)/20.0));
    PDEFIELD_TYPE tau_Xr2      = 1.0*alpha_Xr2*beta_Xr2/1000.0;
    dydt[9]    = (Xr2_infinity-y[9])/tau_Xr2;
  
    //// IK1
    PDEFIELD_TYPE alpha_K1    = 3.91/(1.0+exp(0.5942*(y[0]*1000.0-E_K*1000.0-200.0)));
    PDEFIELD_TYPE beta_K1     = (-1.509*exp(0.0002*(y[0]*1000.0-E_K*1000.0+100.0))+exp(0.5886*(y[0]*1000.0-E_K*1000.0-10.0)))/(1.0+exp(0.4547*(y[0]*1000.0-E_K*1000.0)));
    PDEFIELD_TYPE XK1_inf     = alpha_K1/(alpha_K1+beta_K1);
    PDEFIELD_TYPE g_K1        = 28.1492;   // S_per_F (in i_K1)
    PDEFIELD_TYPE i_K1        = g_K1*XK1_inf*(y[0]-E_K)*sqrt(Ko/5.4);
  
    //// INaCa
    PDEFIELD_TYPE KmCa        = 1.38;   // millimolar (in i_NaCa)
    PDEFIELD_TYPE KmNai       = 87.5;   // millimolar (in i_NaCa)
    PDEFIELD_TYPE Ksat        = 0.1;    // dimensionless (in i_NaCa)
    PDEFIELD_TYPE gamma       = 0.35;   // dimensionless (in i_NaCa)
    PDEFIELD_TYPE alpha       = 2.16659;
    PDEFIELD_TYPE kNaCa      = 3917.0463; //((time<tDrugApplication)*1+(time >= tDrugApplication)*INaCaRedMed) * 6514.47574;   // A_per_F (in i_NaCa)
    PDEFIELD_TYPE i_NaCa      = kNaCa*(exp(gamma*y[0]*F/(R*T))*pow((float)y[17],3.0f)*Cao-exp((gamma-1.0)*y[0]*F/(R*T))*pow((float)Nao,3.0f)*y[2]*alpha)/((pow((float)KmNai,3.0f)+pow((float)Nao,3.0f))*(KmCa+Cao)*(1.0+Ksat*exp((gamma-1.0)*y[0]*F/(R*T))));
  
    //// INaK
    PDEFIELD_TYPE Km_K        = 1.0;    // millimolar (in i_NaK)
    PDEFIELD_TYPE Km_Na       = 40.0;   // millimolar (in i_NaK)
    PDEFIELD_TYPE PNaK        = 2.74240;// A_per_F (in i_NaK)
    PDEFIELD_TYPE i_NaK       = PNaK*Ko/(Ko+Km_K)*y[17]/(y[17]+Km_Na)/(1.0+0.1245*exp(-0.1*y[0]*F/(R*T))+0.0353*exp(-y[0]*F/(R*T)));
  
    //// IpCa
    PDEFIELD_TYPE KPCa        = 0.0005;   // millimolar (in i_PCa)
    PDEFIELD_TYPE g_PCa       = 0.4125;   // A_per_F (in i_PCa)
    PDEFIELD_TYPE i_PCa       = g_PCa*y[2]/(y[2]+KPCa);
  
    //// Background currents
    PDEFIELD_TYPE g_b_Na      = 1.14;         // S_per_F (in i_b_Na)
    PDEFIELD_TYPE i_b_Na      = g_b_Na*(y[0]-E_Na);
  
    PDEFIELD_TYPE g_b_Ca      = 0.8727264;    // S_per_F (in i_b_Ca)
    PDEFIELD_TYPE i_b_Ca      = g_b_Ca*(y[0]-E_Ca);
  
    //// Sarcoplasmic reticulum
    PDEFIELD_TYPE VmaxUp		= 0.82205;
    PDEFIELD_TYPE Kup			=  4.40435e-4;
    PDEFIELD_TYPE i_up        = VmaxUp/(1.0+pow((float)Kup,2.0f)/pow((float)y[2],2.0f));
  
    PDEFIELD_TYPE V_leak		= 4.48209e-4;
    PDEFIELD_TYPE i_leak      = (y[1]-y[2])*V_leak;
  
    dydt[3]    = 0;
  
    // RyR
    PDEFIELD_TYPE g_irel_max	= 55.808061;
    PDEFIELD_TYPE RyRa1       = 0.05169;
    PDEFIELD_TYPE RyRa2       = 0.050001;
    PDEFIELD_TYPE RyRahalf    = 0.02632;
    PDEFIELD_TYPE RyRohalf    = 0.00944;
    PDEFIELD_TYPE RyRchalf    = 0.00167;
  
    PDEFIELD_TYPE RyRSRCass   = (1 - 1/(1 +  exp((y[1]-0.3)/0.1)));
    PDEFIELD_TYPE i_rel       = g_irel_max*RyRSRCass*y[21]*y[22]*(y[1]-y[2]);
  
    PDEFIELD_TYPE RyRainfss   = RyRa1-RyRa2/(1 + exp((1000*y[2]-(RyRahalf))/0.0082));
    PDEFIELD_TYPE RyRtauadapt = 1; //s
    dydt[20]   = (RyRainfss- y[20])/RyRtauadapt;
  
    PDEFIELD_TYPE RyRoinfss   = (1 - 1/(1 +  exp((1000*y[2]-(y[20]+ RyRohalf))/0.003)));
    PDEFIELD_TYPE RyRtauact;
    if (RyRoinfss>= y[21])
      RyRtauact = 18.75e-3;       //s
    else
      RyRtauact = 0.1*18.75e-3;   //s
  
    dydt[21]    = (RyRoinfss- y[21])/(RyRtauact);
  
    PDEFIELD_TYPE RyRcinfss   = (1/(1 + exp((1000*y[2]-(y[20]+RyRchalf))/0.001)));
    PDEFIELD_TYPE RyRtauinact;
    if (RyRcinfss>= y[22])
      RyRtauinact = 2*87.5e-3;    //s
    else
      RyRtauinact = 87.5e-3;      //s
  
    dydt[22]    = (RyRcinfss- y[22])/(RyRtauinact);
  
  
  
  
    //// Ca2+ buffering
    PDEFIELD_TYPE Buf_C       = 0.25;   // millimolar (in calcium_dynamics)
    PDEFIELD_TYPE Buf_SR      = 10.0;   // millimolar (in calcium_dynamics)
    PDEFIELD_TYPE Kbuf_C      = 0.001;   // millimolar (in calcium_dynamics)
    PDEFIELD_TYPE Kbuf_SR     = 0.3;   // millimolar (in calcium_dynamics)
    PDEFIELD_TYPE Cai_bufc    = 1.0/(1.0+Buf_C*Kbuf_C/pow((float)(y[2]+Kbuf_C), 2.0f));
    PDEFIELD_TYPE Ca_SR_bufSR = 1.0/(1.0+Buf_SR*Kbuf_SR/pow((float)(y[1]+Kbuf_SR), 2.0f));
  
    //// Ionic concentrations
    //Nai
    dydt[17]   = -Cm*(i_Na+i_NaL+i_b_Na+3.0*i_NaK+3.0*i_NaCa+i_fNa)/(F*Vc*1.0e-18);
    //Cai
    dydt[2]    = Cai_bufc*(i_leak-i_up+i_rel-(i_CaL+i_b_Ca+i_PCa-2.0*i_NaCa)*Cm/(2.0*Vc*F*1.0e-18));
     //caSR
    dydt[1]    = Ca_SR_bufSR*Vc/V_SR*(i_up-(i_rel+i_leak));
  
    //// Stimulation
  //  PDEFIELD_TYPE i_stim_Amplitude 		= 5.5e-10;//7.5e-10;   // ampere (in stim_mode)
  //  PDEFIELD_TYPE i_stim_End 				= 1000.0;   // second (in stim_mode)
  //  PDEFIELD_TYPE i_stim_PulseDuration	= 0.005;   // second (in stim_mode)
  //  PDEFIELD_TYPE i_stim_Start 			= 0.0;   // second (in stim_mode)
  //  PDEFIELD_TYPE i_stim_frequency        = 60.0;   // per_second (in stim_mode)
    //PDEFIELD_TYPE stim_flag 				= stimFlag;   // dimensionless (in stim_mode)
  //  PDEFIELD_TYPE i_stim_Period 			= 60.0/i_stim_frequency;
  
    //if stim_flag~=0 && stim_flag~=1
    //error('Paci2020: wrong pacing! stimFlag can be only 0 (spontaneous) or 1 (paced)');
    //end
  
    /*
    if ((time >= i_stim_Start) && (time <= i_stim_End) && (time-i_stim_Start-floor((time-i_stim_Start)/i_stim_Period)*i_stim_Period <= i_stim_PulseDuration))
        i_stim = stim_flag*i_stim_Amplitude/Cm;
    else
        i_stim = 0.0;
    */
    PDEFIELD_TYPE i_stim = 0;
  
    //// Membrane potential
    dydt[0] = -(i_K1+i_to+i_Kr+i_Ks+i_CaL+i_NaK+i_Na+i_NaL+i_NaCa+i_PCa+i_f+i_b_Na+i_b_Ca-i_stim);
}
__global__ void RungeKuttaStep(PDEFIELD_TYPE dt, PDEFIELD_TYPE thetime, int layers, int sizex, int sizey, PDEFIELD_TYPE* PDEvars, PDEFIELD_TYPE* alt_PDEvars){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int i;

  PDEFIELD_TYPE dydt[23];
  PDEFIELD_TYPE ak2[23];
  PDEFIELD_TYPE ak3[23];
  PDEFIELD_TYPE ak4[23];
  PDEFIELD_TYPE ak5[23];
  PDEFIELD_TYPE ak6[23];
  PDEFIELD_TYPE ytemp[23]; 

  PDEFIELD_TYPE y[23];

  PDEFIELD_TYPE a2=0.2,a3=0.3,a4=0.6,a5=1.0,a6=0.875,b21=0.2,
  b31=3.0/40.0,b32=9.0/40.0,b41=0.3,b42 = -0.9,b43=1.2,
  b51 = -11.0/54.0, b52=2.5,b53 = -70.0/27.0,b54=35.0/27.0,
  b61=1631.0/55296.0,b62=175.0/512.0,b63=575.0/13824.0,
  b64=44275.0/110592.0,b65=253.0/4096.0,c1=37.0/378.0,
  c3=250.0/621.0,c4=125.0/594.0,c6=512.0/1771.0;
  for (int id = index; id < sizex*sizey; id += stride){
    for (int l = 0; l < layers; l++) //fill with current PDE values
      y[l] = PDEvars[l*sizex*sizey + l];
  


    //Paci2020
 
    ComputeDerivs(thetime,y,dydt,id);
    for (i=0;i<layers;i++) //First step.
      ytemp[i]=y[i]+b21*dt*dydt[i];
    ComputeDerivs(thetime+a2*dt,ytemp,ak2,id);// Second step.

    for (i=0;i<layers;i++)
      ytemp[i]=y[i]+dt*(b31*dydt[i]+b32*ak2[i]);
    ComputeDerivs(thetime+a3*dt,ytemp,ak3,id); //Third step.
    for (i=0;i<layers;i++)
      ytemp[i]=y[i]+dt*(b41*dydt[i]+b42*ak2[i]+b43*ak3[i]);
    ComputeDerivs(thetime+a4*dt,ytemp,ak4,id); //Fourth step.
    for (i=0;i<layers;i++)
      ytemp[i]=y[i]+dt*(b51*dydt[i]+b52*ak2[i]+b53*ak3[i]+b54*ak4[i]);
    ComputeDerivs(thetime+a5*dt,ytemp,ak5,id); //Fifth step.
    for (i=0;i<layers;i++)
      ytemp[i]=y[i]+dt*(b61*dydt[i]+b62*ak2[i]+b63*ak3[i]+b64*ak4[i]+b65*ak5[i]);
    ComputeDerivs(thetime+a6*dt,ytemp,ak6,id); //Sixth step.
    for (i=0;i<layers;i++) //Accumulate increments with proper weights.
      alt_PDEvars[i*sizex*sizey + id]=PDEvars[i*sizex*sizey + id]+(c1*dydt[i]+c3*ak3[i]+c4*ak4[i]+c6*ak6[i])*dt;
  }
}

void PDE::cuPDEsteps(CellularPotts * cpm, int repeat){
  //setup matrices for upperdiagonal, diagonal and lower diagonal for both the horizontal and vertical direction, since these remain the same during once MCS
  InitializeDiagonals<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, 2/dt, dx2, lowerH[0], upperH[0], diagH[0], lowerV[0], upperV[0], diagV[0], couplingcoefficient[0]);
  for (int iteration = 0; iteration < repeat; iteration++){
    //Do an ODE step of size dt/2
    RungeKuttaStep<<<par.number_of_cores, par.threads_per_core>>>(dt, thetime, layers, sizex, sizey, PDEvars[0][0], alt_PDEvars[0][0]);
    
    //Do a horizontal ADI sweep of size dt/2
    InitializeHorizontalVectors<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, 2/dt, dx2, BH[0], couplingcoefficient[0], alt_PDEvars[0][0]);
    statusH = hipsparseSgtsvInterleavedBatch(handleH, 0, sizex, lowerH[0], diagH[0], upperH[0], BH[0], sizey, &pbuffersizeH);
    if (statusH != HIPSPARSE_STATUS_SUCCESS)
    {
      cout << statusH << endl;
    }
    NewPDEfieldH<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, BH[0], PDEvars[0][0]);

    //Do an ODE step of size dt/2
    RungeKuttaStep<<<par.number_of_cores, par.threads_per_core>>>(dt, thetime, layers, sizex, sizey, PDEvars[0][0], alt_PDEvars[0][0]);

    //Do a vertical ADI sweep of size dt/2
    InitializeVerticalVectors<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, 2/dt, dx2, BV[0], couplingcoefficient[0], alt_PDEvars[0][0]);
    statusV = hipsparseSgtsvInterleavedBatch(handleV, 0, sizey, lowerV[0], diagV[0], upperV[0], BV[0], sizex, &pbuffersizeV);
    if (statusV != HIPSPARSE_STATUS_SUCCESS)
    {
      cout << statusV << endl;
    }
    NewPDEfieldV<<<par.number_of_cores, par.threads_per_core>>>(sizex, sizey, BV[0], PDEvars[0][0]); //////

    thetime = thetime + dt;
  }
}

// public
void PDE::Diffuse(int repeat) {
  
  // Just diffuse everywhere (cells are transparent), using finite difference
  // (We're ignoring the problem of how to cope with moving cell
  // boundaries right now)
  
  const PDEFIELD_TYPE dt=par.dt;
  const PDEFIELD_TYPE dx2=par.dx*par.dx;

  for (int r=0;r<repeat;r++) {
    //NoFluxBoundaries();
    if (par.periodic_boundaries) {
      PeriodicBoundaries();
    } else {
      AbsorbingBoundaries();
      //NoFluxBoundaries();
    }
    for (int l=0;l<layers;l++) {
      for (int x=1;x<sizex-1;x++)
	for (int y=1;y<sizey-1;y++) {
	  PDEFIELD_TYPE sum=0.;
	  sum+=PDEvars[l][x+1][y];
	  sum+=PDEvars[l][x-1][y];
	  sum+=PDEvars[l][x][y+1];
	  sum+=PDEvars[l][x][y-1];
	  sum-=4*PDEvars[l][x][y];
	  alt_PDEvars[l][x][y]=PDEvars[l][x][y]+sum*dt*par.diff_coeff[l]/dx2;
      }
    }
    PDEFIELD_TYPE ***tmp;
    tmp=PDEvars;
    PDEvars=alt_PDEvars;
    alt_PDEvars=tmp;
  
    thetime+=dt;
  }
}

double PDE::GetChemAmount(const int layer) {
  // Sum the total amount of chemical in the lattice
  // in layer l
  // (This is useful to check particle conservation)
  double sum=0.;
  if (layer==-1) { // default argument: sum all chemical species
    for (int l=0;l<layers;l++) {
      for (int x=1;x<sizex-1;x++) {
	for (int y=1;y<sizey-1;y++) {
	  sum+=PDEvars[l][x][y];
	}
      }
    }
  } else {
    for (int x=1;x<sizex-1;x++)
      for (int y=1;y<sizey-1;y++) {
	sum+=PDEvars[layer][x][y];
      }
  } 
  return sum;
}

// private
void PDE::NoFluxBoundaries(void) {
  // all gradients at the edges become zero, 
  // so nothing flows out
  // Note that four corners points are not defined (0.)
  // but they aren't used in the calculations
  for (int l=0;l<layers;l++) {
    for (int x=0;x<sizex;x++) {
      PDEvars[l][x][0]=PDEvars[l][x][1];
      PDEvars[l][x][sizey-1]=PDEvars[l][x][sizey-2];
    }
    for (int y=0;y<sizey;y++) {
      PDEvars[l][0][y]=PDEvars[l][1][y];
      PDEvars[l][sizex-1][y]=PDEvars[l][sizex-2][y];
    }
  }
}


// private
void PDE::AbsorbingBoundaries(void) {
  // all boundaries are sinks, 
  for (int l=0;l<layers;l++) {
    for (int x=0;x<sizex;x++) {
      PDEvars[l][x][0]=0.;
      PDEvars[l][x][sizey-1]=0.;
    }
    for (int y=0;y<sizey;y++) {
      PDEvars[l][0][y]=0.;
      PDEvars[l][sizex-1][y]=0.;
    }
  }
}

// private
void PDE::PeriodicBoundaries(void) {
  // periodic...
  for (int l=0;l<layers;l++) {
    for (int x=0;x<sizex;x++) {
      PDEvars[l][x][0]=PDEvars[l][x][sizey-2];
      PDEvars[l][x][sizey-1]=PDEvars[l][x][1];
    }
    for (int y=0;y<sizey;y++) {
      PDEvars[l][0][y]=PDEvars[l][sizex-2][y];
      PDEvars[l][sizex-1][y]=PDEvars[l][1][y];
    }
  }
}

void PDE::GradC(int layer, int first_grad_layer) {
  // calculate the first and second order gradients and put
  // them in the next chemical fields
  if (par.n_chem<5) {
    throw("PDE::GradC: Not enough chemical fields");
  }

  // GradX
  for (int y=0;y<sizey;y++) {
    for (int x=1;x<sizex-1;x++) {
      PDEvars[first_grad_layer][x][y]=(PDEvars[layer][x+1][y]-PDEvars[layer][x-1][y])/2.;
    } 
  }
  // GradY
  for (int x=0;x<sizex;x++) {
    for (int y=1;y<sizey-1;y++) {
      PDEvars[first_grad_layer+1][x][y]=(PDEvars[layer][x][y+1]-PDEvars[layer][x][y-1])/2.;
    } 
  }
  // GradXX
  for (int y=0;y<sizey;y++) {
    for (int x=1;x<sizex-1;x++) {
      PDEvars[first_grad_layer+2][x][y]=PDEvars[layer][x+1][y]-PDEvars[layer][x-1][y]-2*PDEvars[layer][x][y];
    } 
  }

  // GradYY
  for (int x=0;x<sizex;x++) {
    for (int y=1;y<sizey-1;y++) {
      PDEvars[first_grad_layer+3][x][y]=PDEvars[layer][x][y-1]-PDEvars[layer][x][y+1]-2*PDEvars[layer][x][y];
    } 
  }
}

void PDE::PlotVectorField(Graphics &g, int stride, int linelength, int first_grad_layer) {
  // Plot vector field assuming it's in layer 1 and 2
  for (int x=1;x<sizex-1;x+=stride) {
    for (int y=1;y<sizey-1;y+=stride) {
      
      // calculate line
      int x1,y1,x2,y2;
      
      x1=(int)(x-linelength*PDEvars[first_grad_layer][x][y]);
      y1=(int)(y-linelength*PDEvars[first_grad_layer+1][x][y]);
      x2=(int)(x+linelength*PDEvars[first_grad_layer][x][y]);
      y2=(int)(y+linelength*PDEvars[first_grad_layer+1][x][y]);
      if (x1<0) x1=0;
      if (x1>sizex-1) x1=sizex-1;
      if (y1<0) y1=0;
      if (y1>sizey-1) y1=sizey-1;
      if (x2<0) x2=0;
      if (x2>sizex-1) x2=sizex-1;
      if (y2<0) y2=0;
      if (y2>sizey-1) y2=sizey-1;

      // And draw it :-)
      // perhaps I can add arrowheads later to make it even nicer :-)
      g.Line(x1,y1,x2,y2,1);
    }
  }
}

bool PDE::plotPos(int x, int y, Graphics * graphics, int layer) {
  layer = 0;
  double val = PDEvars[layer][x][y];
  if (val > -100){
    graphics->Rectangle(MapColour(val), x, y); 
    return false;
  }
  return true;
}


void PDE::SetSpeciesName(int l, const char *name) {
    species_names[l]=string(name);
}


void PDE::InitLinearYGradient(int spec, double conc_top, double conc_bottom) {
    for (int y=0;y<sizey;y++) {
      double val=(double)conc_top+y*((double)(conc_bottom-conc_top)/(double)sizey);
    for (int x=0;x<sizex;x++) {
      PDEvars[spec][x][y]=val;
    }
    cerr << y << " " << val << endl;
  }
}
